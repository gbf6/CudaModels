#include "hip/hip_runtime.h"
#include <math.h>
#include <hip/hip_runtime.h>

#include "hip/hip_runtime.h"
#include ""

#include "./common/CudaSafeCall.h"

#include "typedefSparse.h"
#include "sparsePrototypes.cuh"
#include "parseInput.cuh"
#include "VectorOps.cuh"

#include "mainwrapper.cuh"

#include "inputs.cuh"

#include "./CRN/CRNMethods.cu"

#include "./Stim/stimSimple.cu"


void main_wrapper(int pitch, real beta, real Cm, real t, real dt, int totpoints, real rx, gateType g_dev, gateType g_devF, stimulus stim, cudasparse cudaMatrixINT) {
	int grid = (int)ceil(totpoints / (real)BLOCK_DIM_1D);
	int block = BLOCK_DIM_1D;

	TstepEM<<<grid, block>>>(pitch, beta, Cm, t, dt, totpoints, rx, g_dev, g_devF, cudaMatrixINT);
	hipDeviceSynchronize();

	stimSimpleDev<<<1, 1>>>(stim, t, g_devF);
	hipDeviceSynchronize();

	TstepEMExit<<<grid, block>>>(Cm, dt, totpoints, g_dev, g_devF);
	hipDeviceSynchronize();
	
}

void __global__ TstepEM(int pitch, real beta, real Cm, real t, real dt, int totpoints, real rx, gateType g_dev, gateType g_devF, cudasparse cudaMatrixINT) {
	const int i2d = blockIdx.x*BLOCK_DIM_1D + threadIdx.x;
	if(i2d < totpoints) {
		g_devF.vm[i2d] = 0;
		sprDiff(g_devF.vm, &cudaMatrixINT, i2d, totpoints, g_dev.vm, beta);

		GetFDev_CRN(i2d, pitch, beta, Cm, t, dt, totpoints, rx, g_dev, g_devF);

	}
}
void __global__ TstepEMExit(real Cm, real dt, int totpoints, gateType g_dev, gateType g_devF) {
	const int i2d = blockIdx.x*BLOCK_DIM_1D + threadIdx.x;
	if(i2d < totpoints){
		vvaddDev(i2d, dt / Cm, g_devF.vm, g_dev.vm, totpoints);
		for (int i = 0; i < g_dev.qpl; i++) vvaddDev(i2d, dt, g_devF.qp[i], g_dev.qp[i], totpoints);
	}
}