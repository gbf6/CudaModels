#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include "hip/hip_runtime.h"
#include ""


#include "../common/CudaSafeCall.h"

#include "TNNPglobalVariables.cuh"
#include "typedefSparse.h"
#include "sparsePrototypes.cuh"
#include "typedefTNNP.h"

#include "TNNPhostPrototypes.h"
#include "TNNPdevicePrototypes.cuh"

void TNNP_gateinit(int memSize, size_t* pitch, gateType* gate_h, gateType* gate_dev, gateType* gate_devF) {

	hipHostAlloc((void**)&(gate_h->vm), memSize, 0);

	hipHostAlloc((void**)&(gate_h->Cai), memSize, 0);
	hipHostAlloc((void**)&(gate_h->CaSR), memSize, 0);
	hipHostAlloc((void**)&(gate_h->Nai), memSize, 0);
	hipHostAlloc((void**)&(gate_h->Ki), memSize, 0);
	hipHostAlloc((void**)&(gate_h->m), memSize, 0);
	hipHostAlloc((void**)&(gate_h->h), memSize, 0);
	hipHostAlloc((void**)&(gate_h->j), memSize, 0);
	hipHostAlloc((void**)&(gate_h->xr1), memSize, 0);
	hipHostAlloc((void**)&(gate_h->xr2), memSize, 0);
	hipHostAlloc((void**)&(gate_h->xs), memSize, 0);
	hipHostAlloc((void**)&(gate_h->r), memSize, 0);
	hipHostAlloc((void**)&(gate_h->s), memSize, 0);
	hipHostAlloc((void**)&(gate_h->d), memSize, 0);
	hipHostAlloc((void**)&(gate_h->f), memSize, 0);
	hipHostAlloc((void**)&(gate_h->fca), memSize, 0);
	hipHostAlloc((void**)&(gate_h->g), memSize, 0);
	

	// Allocate device memory arrays
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->vm, pitch,
		memSize, 1));

	CudaSafeCall(hipMallocPitch((void **)&gate_dev->Cai, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->CaSR, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->Nai, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->Ki, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->m, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->h, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->j, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->xr1, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->xr2, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->xs, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->r, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->s, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->d, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->f, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->fca, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->g, pitch,
		memSize, 1));

	// Allocate device forward memory arrays
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->vm, pitch,
		memSize, 1));

	CudaSafeCall(hipMallocPitch((void **)&gate_devF->Cai, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->CaSR, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->Nai, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->Ki, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->m, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->h, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->j, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->xr1, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->xr2, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->xs, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->r, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->s, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->d, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->f, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->fca, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->g, pitch,
		memSize, 1));

	puts("\nFinished allocating device arrays\n");

	int totpoints = (int)memSize / sizeof(real);

	for (int idx = 0; idx < totpoints; idx++) {
		gate_h->vm[idx] = TNNP_RestVoltage;

		gate_h->Cai[idx] = 0.0002;
		gate_h->CaSR[idx] = 0.2;
		gate_h->Nai[idx] = 11.6;
		gate_h->Ki[idx] = 138.3;
		gate_h->m[idx] = 0.0;
		gate_h->h[idx] = 0.75;
		gate_h->j[idx] = 0.75;
		gate_h->xr1[idx] = 0.0;
		gate_h->xr2[idx] = 1.0;
		gate_h->xs[idx] = 0.0;
		gate_h->r[idx] = 0.0;
		gate_h->s[idx] = 1.0;
		gate_h->d[idx] = 0.0;
		gate_h->f[idx] = 1.0;
		gate_h->fca[idx] = 1.0;
		gate_h->g)[idx] = 1.0;
	}

	CudaSafeCall(hipMemcpy2D((void *)gate_dev->vm, *pitch, (void *)gate_h->vm,
		memSize, memSize, 1, hipMemcpyHostToDevice));

	CudaSafeCall(hipMemcpy2D((void *)gate_dev->Cai, *pitch, (void *)gate_h->Cai,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->CaSR, *pitch, (void *)gate_h->CaSR,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->Nai, *pitch, (void *)gate_h->Nai,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->Ki, *pitch, (void *)gate_h->Ki,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->m, *pitch, (void *)gate_h->m,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->h, *pitch, (void *)gate_h->h,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->j, *pitch, (void *)gate_h->j,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->xr1, *pitch, (void *)gate_h->xr1,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->xr2, *pitch, (void *)gate_h->xr2,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->xs, *pitch, (void *)gate_h->xs,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->r, *pitch, (void *)gate_h->r,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->s, *pitch, (void *)gate_h->s,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->d, *pitch, (void *)gate_h->d,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->,f *pitch, (void *)gate_h->f,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->fca, *pitch, (void *)gate_h->fca,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->g, *pitch, (void *)gate_h->g,
		memSize, memSize, 1, hipMemcpyHostToDevice));


	CudaSafeCall(hipMemcpy2D((void *)gate_devF->vm, *pitch, (void *)gate_h->vm,
		memSize, memSize, 1, hipMemcpyHostToDevice));

	CudaSafeCall(hipMemcpy2D((void *)gate_devF->Cai, *pitch, (void *)gate_h->Cai,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->CaSR, *pitch, (void *)gate_h->CaSR,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->Nai, *pitch, (void *)gate_h->Nai,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->Ki, *pitch, (void *)gate_h->Ki,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->m, *pitch, (void *)gate_h->m,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->h, *pitch, (void *)gate_h->h,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->j, *pitch, (void *)gate_h->j,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->xr1, *pitch, (void *)gate_h->xr1,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->xr2, *pitch, (void *)gate_h->xr2,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->xs, *pitch, (void *)gate_h->xs,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->r, *pitch, (void *)gate_h->r,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->s, *pitch, (void *)gate_h->s,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->d, *pitch, (void *)gate_h->d,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->,f *pitch, (void *)gate_h->f,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->fca, *pitch, (void *)gate_h->fca,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->g, *pitch, (void *)gate_h->g,
		memSize, memSize, 1, hipMemcpyHostToDevice));

	real** qpH = (real**)malloc(sizeof(real *)*gate_h->qpl);
	int i = 0;
	qpH[i++] = gate_devF->Cai;
	qpH[i++] = gate_devF->CaSR;
	qpH[i++] = gate_devF->Nai;
	qpH[i++] = gate_devF->Ki;
	qpH[i++] = gate_devF->m;
	qpH[i++] = gate_devF->h;
	qpH[i++] = gate_devF->j;
	qpH[i++] = gate_devF->xr1;
	qpH[i++] = gate_devF->xr2;
	qpH[i++] = gate_devF->xs;
	qpH[i++] = gate_devF->r;
	qpH[i++] = gate_devF->s;
	qpH[i++] = gate_devF->d;
	qpH[i++] = gate_devF->f;
	qpH[i++] = gate_devF->fca;
	qpH[i++] = gate_devF->g;

	CudaSafeCall(hipMemcpy((void *)gate_devF->qp, (void*)qpH, sizeof(real *)*gate_h->qpl, hipMemcpyHostToDevice));

	i = 0;
	qpH[i++] = gate_dev->Cai;
	qpH[i++] = gate_dev->CaSR;
	qpH[i++] = gate_dev->Nai;
	qpH[i++] = gate_dev->Ki;
	qpH[i++] = gate_dev->m;
	qpH[i++] = gate_dev->h;
	qpH[i++] = gate_dev->j;
	qpH[i++] = gate_dev->xr1;
	qpH[i++] = gate_dev->xr2;
	qpH[i++] = gate_dev->xs;
	qpH[i++] = gate_dev->r;
	qpH[i++] = gate_dev->s;
	qpH[i++] = gate_dev->d;
	qpH[i++] = gate_dev->f;
	qpH[i++] = gate_dev->fca;
	qpH[i++] = gate_dev->g;

	CudaSafeCall(hipMemcpy((void *)gate_dev->qp, (void*)qpH, sizeof(real *)*gate_h->qpl, hipMemcpyHostToDevice));
	

	CudaCheckError();

	puts("\nFinished initializing device arrays\n");

}

void TNNP_sync(int memSize, size_t pitch, gateType* gate_h, gateType* gate_dev) {

	CudaSafeCall(hipMemcpy2D((void *)gate_h->vm, *pitch, (void *)gate_dev->vm,
		memSize, memSize, 1, hipMemcpyDeviceToHost));

	CudaSafeCall(hipMemcpy2D((void *)gate_h->Cai, *pitch, (void *)gate_dev->Cai,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->CaSR, *pitch, (void *)gate_dev->CaSR,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->Nai, *pitch, (void *)gate_dev->Nai,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->Ki, *pitch, (void *)gate_dev->Ki,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->m, *pitch, (void *)gate_dev->m,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->h, *pitch, (void *)gate_dev->h,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->j, *pitch, (void *)gate_dev->j,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->xr1, *pitch, (void *)gate_dev->xr1,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->xr2, *pitch, (void *)gate_dev->xr2,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->xs, *pitch, (void *)gate_dev->xs,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->r, *pitch, (void *)gate_dev->r,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->s, *pitch, (void *)gate_dev->s,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->d, *pitch, (void *)gate_dev->d,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->,f *pitch, (void *)gate_dev->f,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->fca, *pitch, (void *)gate_dev->fca,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->g, *pitch, (void *)gate_dev->g,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
}

void TNNP_exit(int memSize, size_t pitch, gateType* gate_h, gateType* gate_dev, gateType* gate_devF, sparse* MatrixINT, cudasparse* cudaMatrixINT){
	// Free gate host and device memory
	hipHostFree(gate_h->vm); hipHostFree(gate_h->Cai); hipHostFree(gate_h->CaSR); hipHostFree(gate_h->Nai); hipHostFree(gate_h->Ki); hipHostFree(gate_h->m); hipHostFree(gate_h->h); hipHostFree(gate_h->j); hipHostFree(gate_h->xr1); hipHostFree(gate_h->xr2); hipHostFree(gate_h->xs); hipHostFree(gate_h->r); hipHostFree(gate_h->s); hipHostFree(gate_h->d); hipHostFree(gate_h->f); hipHostFree(gate_h->fca); hipHostFree(gate_h->g); hipHostFree(gate_h->qp);

	hipFree(gate_dev->vm); hipFree(gate_dev->Cai); hipFree(gate_dev->CaSR); hipFree(gate_dev->Nai); hipFree(gate_dev->Ki); hipFree(gate_dev->m); hipFree(gate_dev->h); hipFree(gate_dev->j); hipFree(gate_dev->xr1); hipFree(gate_dev->xr2); hipFree(gate_dev->xs); hipFree(gate_dev->r); hipFree(gate_dev->s); hipFree(gate_dev->d); hipFree(gate_dev->f); hipFree(gate_dev->fca); hipFree(gate_dev->g); hipFree(gate_dev->qp);

	hipFree(gate_devF->vm); hipFree(gate_devF->Cai); hipFree(gate_devF->CaSR); hipFree(gate_devF->Nai); hipFree(gate_devF->Ki); hipFree(gate_devF->m); hipFree(gate_devF->h); hipFree(gate_devF->j); hipFree(gate_devF->xr1); hipFree(gate_devF->xr2); hipFree(gate_devF->xs); hipFree(gate_devF->r); hipFree(gate_devF->s); hipFree(gate_devF->d); hipFree(gate_devF->f); hipFree(gate_devF->fca); hipFree(gate_devF->g); hipFree(gate_devF->qp);

	hipFree(cudaMatrixINT->type);
	hipFree(cudaMatrixINT->rows);
	hipFree(cudaMatrixINT->maxnz);
	hipFree(cudaMatrixINT->csep);
	hipFree(cudaMatrixINT->jcoef);
	hipFree(cudaMatrixINT->coef);
}


void __device__ GetFDev_TNNP(int i2d, int pitch, real beta, real Cm, real t, real dt, int totpoints, real rx, gateType g_dev, gateType g_devF) {

	/*------------------------------------------------------------------------
	* return if outside domain
	*------------------------------------------------------------------------
	*/
	if (i2d >= totpoints) {
		return;
	}

	real V,Ek,Ena,Eks,Eca;
	real Iion,INa,ICaL,Ito,IKr,IKs,IK1,INaCa,INaK,IpCa,IpK,IbNa,IbCa;
	real Nai,Ki,m,h,j,xr1,xr2,xs,r,s,d,f,fca,g;
	real Caisquare,Cai,CaSRsquare,CaCurrent,A,Irel,Ileak,SERCA,CaSRCurrent;
	real CaSR,CaCSQN,dCaSR,bjsr,cjsr,CaBuf,dCai,bc,cc;
	real AM,BM,TAU_M,M_INF,AH_1,BH_1,TAU_H,AH_2,BH_2,H_INF,AJ_1,BJ_1;
	real TAU_J,AJ_2,BJ_2,J_INF,Xr1_INF,axr1,bxr1,TAU_Xr1,Xr2_INF,axr2,bxr2;
	real TAU_Xr2,Xs_INF,Axs,Bxs,TAU_Xs,R_INF,S_INF,TAU_R,TAU_S,D_INF,Ad,Bd,Cd;
	real TAU_D,F_INF,TAU_F,FCa_INF,G_INF,fcaold,gold;
	real rec_iK1,rec_ipK,rec_iNaK,Ak1,Bk1;
	real CaiNew,CaSRNew,fcatemp,gtemp;

	real V    = g_dev.vm[i2d];
	real Cai  = g_dev.Cai[i2d];
	real CaSR = g_dev.CaSR[i2d];
	real Nai  = g_dev.Nai[i2d];
	real Ki   = g_dev.Ki[i2d];
	real m    = g_dev.m[i2d];
	real h    = g_dev.h[i2d];
	real j    = g_dev.j[i2d];
	real xr1  = g_dev.xr1[i2d];
	real xr2  = g_dev.xr2[i2d];
	real xs   = g_dev.xs[i2d];
	real r    = g_dev.r[i2d];
	real s    = g_dev.s[i2d];
	real d    = g_dev.d[i2d];
	real f    = g_dev.f[i2d];
	real fca  = g_dev.fca[i2d];
	real g    = g_dev.g[i2d];

	/*------------------------------------------------------------------------
	* setting local variables
	*------------------------------------------------------------------------
	*/
	real fv = g_devF.vm[i2d];


	Ek=RTONF*(log((Ko/Ki)));
	Ena=RTONF*(log((Nao/Nai)));
	Eks=RTONF*(log((Ko+pKNa*Nao)/(Ki+pKNa*Nai)));
	Eca=0.5*RTONF*(log((Cao/Cai)));
	Ak1=0.1/(1.+exp(0.06*(V-Ek-200)));
	Bk1=(3.*exp(0.0002*(V-Ek+100))+exp(0.1*(V-Ek-10)))/(1.+exp(-0.5*(V-Ek)));
	rec_iK1=Ak1/(Ak1+Bk1);
	rec_iNaK=(1./(1.+0.1245*exp(-0.1*V*F/(R*T))+0.0353*exp(-V*F/(R*T))));
	rec_ipK=1./(1.+exp((25-V)/5.98));


	INa=TNNP_GNa*m*m*m*h*j*(V-Ena);
	ICaL=TNNP_GCaL*d*f*fca*4*V*(F*F/(R*T))*
	   (exp(2*V*F/(R*T))*Cai-0.341*Cao)/(exp(2*V*F/(R*T))-1.);
	Ito=TNNP_Gto*r*s*(V-Ek);
	IKr=TNNP_Gkr*sqrt(Ko/5.4)*xr1*xr2*(V-Ek);
	IKs=TNNP_Gks*xs*xs*(V-Eks);
	IK1=TNNP_GK1*rec_iK1*(V-Ek);
	INaCa=knaca*(1./(KmNai*KmNai*KmNai+Nao*Nao*Nao))*(1./(KmCa+Cao))*
	      (1./(1+ksat*exp((n-1)*V*F/(R*T))))*
	      (exp(n*V*F/(R*T))*Nai*Nai*Nai*Cao-
	       exp((n-1)*V*F/(R*T))*Nao*Nao*Nao*Cai*2.5);
	INaK=knak*(Ko/(Ko+KmK))*(Nai/(Nai+KmNa))*rec_iNaK;
	IpCa=TNNP_GpCa*Cai/(KpCa+Cai);
	IpK=TNNP_GpK*rec_ipK*(V-Ek);
	IbNa=TNNP_GbNa*(V-Ena);
	IbCa=TNNP_GbCa*(V-Eca);
	Iion = IKr+IKs+IK1+Ito+INa+IbNa+ICaL+IbCa+INaK+INaCa+IpCa+IpK;

	/* Update Auxilliary Variables */
    Caisquare=Cai*Cai;
    CaSRsquare=CaSR*CaSR;
    CaCurrent=-(ICaL+IbCa+IpCa-2*INaCa)*inverseVcF2*CAPACITANCE;
    A=0.016464*CaSRsquare/(0.0625+CaSRsquare)+0.008232;
    Irel=A*d*g;
    Ileak=0.00008*(CaSR-Cai);
    SERCA=Vmaxup/(1.+(Kupsquare/Caisquare));
    CaSRCurrent=SERCA-Irel-Ileak;
    CaCSQN=Bufsr*CaSR/(CaSR+Kbufsr);
    dCaSR=dt*(Vc/Vsr)*CaSRCurrent;
    bjsr=Bufsr-CaCSQN-dCaSR-CaSR+Kbufsr;
    cjsr=Kbufsr*(CaCSQN+dCaSR+CaSR);
    CaSRNew=(sqrt(bjsr*bjsr+4*cjsr)-bjsr)/2;
    CaBuf=Bufc*Cai/(Cai+Kbufc);
    dCai=dt*(CaCurrent-CaSRCurrent);
    bc=Bufc-CaBuf-dCai-Cai+Kbufc;
    cc=Kbufc*(CaBuf+dCai+Cai);
    CaiNew=(sqrt(bc*bc+4*cc)-bc)/2;

   	AM=1./(1.+exp((-60.-V)/5.));
    BM=0.1/(1.+exp((V+35.)/5.))+0.10/(1.+exp((V-50.)/200.));
    TAU_M=AM*BM;
    M_INF=1./((1.+exp((-56.86-V)/9.03))*(1.+exp((-56.86-V)/9.03)));
    if (V>=-40.){
		AH_1=0.; 
		BH_1=(0.77/(0.13*(1.+exp(-(V+10.66)/11.1))));
		TAU_H= 1.0/(AH_1+BH_1);
    }
    else{
		AH_2=(0.057*exp(-(V+80.)/6.8));
		BH_2=(2.7*exp(0.079*V)+(3.1e5)*exp(0.3485*V));
		TAU_H=1.0/(AH_2+BH_2);
    }
    H_INF=1./((1.+exp((V+71.55)/7.43))*(1.+exp((V+71.55)/7.43)));
    if(V>=-40.){
		AJ_1=0.;      
		BJ_1=(0.6*exp((0.057)*V)/(1.+exp(-0.1*(V+32.))));
		TAU_J= 1.0/(AJ_1+BJ_1);
    }
    else{
		AJ_2=(((-2.5428e4)*exp(0.2444*V)-(6.948e-6)*
		exp(-0.04391*V))*(V+37.78)/
		   (1.+exp(0.311*(V+79.23))));    
		BJ_2=(0.02424*exp(-0.01052*V)/(1.+exp(-0.1378*(V+40.14))));
		TAU_J= 1.0/(AJ_2+BJ_2);
    }
    J_INF=H_INF;

    Xr1_INF=1./(1.+exp((-26.-V)/7.));
    axr1=450./(1.+exp((-45.-V)/10.));
    bxr1=6./(1.+exp((V-(-30.))/11.5));
    TAU_Xr1=axr1*bxr1;
    Xr2_INF=1./(1.+exp((V-(-88.))/24.));
    axr2=3./(1.+exp((-60.-V)/20.));
    bxr2=1.12/(1.+exp((V-60.)/20.));
    TAU_Xr2=axr2*bxr2;

    Xs_INF=1./(1.+exp((-5.-V)/14.));
    Axs=1100./(sqrt(1.+exp((-10.-V)/6)));
    Bxs=1./(1.+exp((V-60.)/20.));
    TAU_Xs=Axs*Bxs;
    
    R_INF=1./(1.+exp((20-V)/6.));
    S_INF=1./(1.+exp((V+20)/5.));
    TAU_R=9.5*exp(-(V+40.)*(V+40.)/1800.)+0.8;
    TAU_S=85.*exp(-(V+45.)*(V+45.)/320.)+5./(1.+exp((V-20.)/5.))+3.;

	D_INF=1./(1.+exp((-5-V)/7.5));
	Ad=1.4/(1.+exp((-35-V)/13))+0.25;
	Bd=1.4/(1.+exp((V+5)/5));
	Cd=1./(1.+exp((50-V)/20));
	TAU_D=Ad*Bd+Cd;
	F_INF=1./(1.+exp((V+20)/7));
	TAU_F=1125*exp(-(V+27)*(V+27)/240)+80+165/(1.+exp((25-V)/10));

	FCa_INF=(1./(1.+pow((CaiNew/0.000325),8))+
	 0.1/(1.+exp((CaiNew-0.0005)/0.0001))+
	 0.20/(1.+exp((CaiNew-0.00075)/0.0008))+
	 0.23 )/1.46;
	if(CaiNew<0.00035) G_INF=1./(1.+pow((CaiNew/0.00035),6));
	else G_INF=1./(1.+pow((CaiNew/0.00035),16));


	fv -= Iion;
	g_devF.vm[i2d] = fv;

	g_devF.Cai[i2d] = (CaiNew-Cai)/dt;
	g_devF.CaSR[i2d] = (CaSRNew-CaSR)/dt;
	g_devF.Nai[i2d] = -(INa+IbNa+3*INaK+3*INaCa)*inverseVcF*CAPACITANCE;
	g_devF.Ki[i2d] =  -(IK1+Ito+IKr+IKs-2*INaK+IpK)*inverseVcF*CAPACITANCE;
	g_devF.m[i2d] = (M_INF-(M_INF-m)*exp(-dt/TAU_M)-m)/dt;
	g_devF.h[i2d] = (H_INF-(H_INF-h)*exp(-dt/TAU_H)-h)/dt;
	g_devF.j[i2d] = (J_INF-(J_INF-j)*exp(-dt/TAU_J)-j)/dt;
	g_devF.xr1[i2d] = (Xr1_INF-(Xr1_INF-xr1)*exp(-dt/TAU_Xr1)-xr1)/dt;
	g_devF.xr2[i2d] = (Xr2_INF-(Xr2_INF-xr2)*exp(-dt/TAU_Xr2)-xr2)/dt;
	g_devF.xs[i2d] = (Xs_INF-(Xs_INF-xs)*exp(-dt/TAU_Xs)-xs)/dt;
	g_devF.s[i2d] = (S_INF-(S_INF-s)*exp(-dt/TAU_S)-s)/dt;
	g_devF.r[i2d] = (R_INF-(R_INF-r)*exp(-dt/TAU_R)-r)/dt;
	g_devF.d[i2d] = (D_INF-(D_INF-d)*exp(-dt/TAU_D)-d)/dt;
	g_devF.f[i2d] = (F_INF-(F_INF-f)*exp(-dt/TAU_F)-f)/dt;

	fcaold = fca;
	fcatemp = FCa_INF-(FCa_INF-fca)*exp(-dt/taufca);
    if(fcatemp>fcaold && (V)>-60){
		g_devF.fca[i2d] =(fcaold-fca)/dt;
   	} 
   	else{
		g_devF.fca[i2d] =(fcatemp-fca)/dt;
	}


	gold = g;
	gtemp = G_INF-(G_INF-g)*exp(-dt/taug);
    if(gtemp>gold && (V)>-60){
   		g_devF.g[i2d] = (gold-g)/dt;
    }
    else{
		g_devF.g[i2d]= (gtemp-g)/dt;
	}
}