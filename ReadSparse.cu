
#include <stdio.h>
#include <stdlib.h>
#include <ctype.h>

#include "typedefSparse.h"
#include "sparsePrototypes.cuh"
#include "./common/CudaSafeCall.h"

int bvecreadinfo(char* filename, int* ssz) {
	int   sz;
	FILE* fp;
	char  buffer[128];

	fp = fopen(filename, "r");
	if (fp == NULL) {
		return(-1);
	}
	fgets(buffer, 128, fp);
	fclose(fp);

	/* check text IDs */
	if (toupper(buffer[0]) != 'B') {
		/* not a vector file */
		return(-2);
	}
	if ((buffer[2] - '0') != sizeof(byte)) {
		/* wrong data-type size - could convert! */
		return(-4);
	}
	sz = atoi(buffer + 4);

	*ssz = sz;

	return(0);
}


/* allocate a sparse matrix of the given type */
int spralloc(sparse* M, domain_t dtp, int type, int rows,
	int cols, int maxnz) {
	int m, n;
	int* ip;

	M->type = type;
	M->rows = rows;
	M->cols = cols;
	M->maxnz = maxnz;
	M->csep = rows;
	M->dtype = dtp;
	/* grab 2 message tags */
	//M->msgtag = NextMsgTag();
	//NextMsgTag();

	/*if (type == _ZERO) {
		return(0);
	}*/

	/* jcoef array is of size = rows*maxnz */
	M->jcoef = (int*)malloc(rows*maxnz * sizeof(int));
	if (M->jcoef == NULL) {
		return(-1);
	}
	/* set up jcoef area for SPARSE type */
	ip = M->jcoef;
	for (m = 0; m<maxnz; m++) {
		for (n = 0; n<rows; n++) {
			ip[n] = n + 1;
		}
		ip += rows;
	}

	/* coef array is of size = rows*maxnz */
	M->coef = (real*)malloc(rows*maxnz * sizeof(real));
	if (M->coef == NULL) {
		return(-2);
	}
	memset((void*)(M->coef), 0, rows*maxnz * sizeof(real));

	return(0);
}


int sprreadinfo( char* filename, int* type, int* rows, int* cols, 
		int* bwp, int* bwm, int* maxnz ) {
	int   tp,rrr,ccc,bbp,bbm,mmm,m;
	FILE* fp;
	char  buffer[64];
	char* ctmp;

	fp = fopen( filename, "r" );
	if( fp == NULL ) {
		return( -1 );
	}

	fgets( buffer, 64, fp );

	/* check text IDs */
	if( (buffer[0]!='S') && (buffer[0]!='s') ) {
		/* not a sparse matrix file */
		return( -2 );
	}
	if( (buffer[2]-'0') != sizeof(real) ) {
		/* wrong real data-type size - could convert! */
		return( -4 );
	}
	ctmp  = strtok( buffer, tokensep );
	ctmp  = strtok( NULL, tokensep );
	rrr   = atoi( ctmp );
	ctmp  = strtok( NULL, tokensep );
	ccc   = atoi( ctmp );
	ctmp  = strtok( NULL, tokensep );
	bbp   = atoi( ctmp );
	ctmp  = strtok( NULL, tokensep );
	bbm   = atoi( ctmp );
	ctmp  = strtok( NULL, tokensep );
	tp    = atoi( ctmp );
	ctmp  = strtok( NULL, tokensep );
	mmm   = atoi( ctmp );

	fclose( fp );

	*type  = tp;
	*rows  = rrr;
	*cols  = ccc;
	*bwp   = bbp;
	*bwm   = bbm;
	*maxnz = mmm;

	return( 0 );
}

int sprfree(sparse* M) {
	if (M->jcoef != NULL) {
		free(M->jcoef);
	}
	if (M->coef != NULL) {
		free(M->coef);
	}

	M->type = 0;
	M->rows = 0;
	M->cols = 0;
	M->maxnz = 0;
	M->csep = 0;
	M->jcoef = NULL;
	M->coef = NULL;
	M->dtype = Undefd;
	//M->msgtag = MPI_ANY_TAG;

	return(0);
}

/* works for sequential programs only!! */
int sprread( char* filename, sparse* m1 ) {
	int   tp,rrr,ccc,bbp,bbm,mnz,r,c,q,i;
	FILE* fp;
	char  buffer[64];
	char* ctmp;
	real* rtmp;
	int*  itmp;
	real* rptr;
	int*  iptr;

	sprfree( m1 );

	fp = fopen( filename, "r" );
	if( fp == NULL ) {
		return( -1 );
	}

	fgets( buffer, 64, fp );

	/* check text IDs */
	if( (buffer[0]!='S') && (buffer[0]!='s') ) {
		/* not a sparse matrix file */
		return( -2 );
	}

	q = 0x1234;
	ctmp = (char*)(&q);
	if( ctmp[0] == 0x34 ) {
		/* this machine is little endian */
		if( (buffer[1]!='L') && (buffer[1]!='l') ) {
			/* wrong endian-ness - could convert! */
			return( -3 );
		}
	} else {
		/* this machine is big endian */
		if( (buffer[1]!='B') && (buffer[1]!='b') ) {
			/* wrong endian-ness - could convert! */
			return( -3 );
		}
	}

	if( (buffer[2]-'0') != sizeof(real) ) {
		/* wrong real data-type size - could convert! */
		return( -4 );
	}
	ctmp = strtok( buffer, tokensep );
	ctmp = strtok( NULL, tokensep );
	rrr  = atoi( ctmp );
	ctmp = strtok( NULL, tokensep );
	ccc  = atoi( ctmp );
	ctmp = strtok( NULL, tokensep );
	bbp  = atoi( ctmp );
	ctmp = strtok( NULL, tokensep );
	bbm  = atoi( ctmp );
	ctmp = strtok( NULL, tokensep );
	tp   = atoi( ctmp );
	ctmp = strtok( NULL, tokensep );
	mnz  = atoi( ctmp );

	/* set up memory space */
	r = spralloc( m1, Tissue, _PRIMARY, rrr, ccc, mnz );
	if( r < 0 ) {
		return( r-6 );
	}

	/* get jcoef entries */
	q = buffer[3] - '0';
	if( q == sizeof(int) ) {
		fseek( fp, 128, SEEK_SET );
		itmp = m1->jcoef;
		/*for(c=0;c<mnz;c++) {
			fread( itmp, sizeof(int), rrr, fp );
			itmp += rrr;
		}*/
		fread(itmp, sizeof(int), rrr*mnz, fp);
		fseek( fp, 128+mnz*rrr*sizeof(int), SEEK_SET );
	} else {
		fclose( fp );
		return( -5 );
	}

	/* get coef entries */
	rtmp = m1->coef;
	for(c=0;c<mnz;c++) {
		fread( rtmp, sizeof(real), rrr, fp );
		rtmp += rrr;
	}
	fclose(fp);

	return( 0 );
}

void swapendian(void* ptr, int dsize, int num) {
	char* cp = (char*)ptr;
	char b0, b1, b2, b3, b4, b5, b6, b7;
	int i, j;

	switch (dsize) {
	case 8:
		for (i = 0; i<num; i++) {
			b0 = cp[0];
			b1 = cp[1];
			b2 = cp[2];
			b3 = cp[3];
			b4 = cp[4];
			b5 = cp[5];
			b6 = cp[6];
			b7 = cp[7];
			cp[0] = b7;
			cp[1] = b6;
			cp[2] = b5;
			cp[3] = b4;
			cp[4] = b3;
			cp[5] = b2;
			cp[6] = b1;
			cp[7] = b0;
			cp += 8;
		}
		break;
	case 4:
		for (i = 0; i<num; i++) {
			b0 = cp[0];
			b1 = cp[1];
			b2 = cp[2];
			b3 = cp[3];
			cp[0] = b3;
			cp[1] = b2;
			cp[2] = b1;
			cp[3] = b0;
			cp += 4;
		}
		break;
	case 2:
		for (i = 0; i<num; i++) {
			b0 = cp[0];
			b1 = cp[1];
			cp[0] = b1;
			cp[1] = b0;
			cp += 2;
		}
		break;
	case 1:
		/* no work to do */
		break;
	}

	return;
}

void cudasprinit(cudasparse* C, sparse* M) {
	CudaSafeCall(hipMalloc((void **)&C->type, sizeof(M->type)));
	CudaSafeCall(hipMalloc((void **)&C->rows, sizeof(M->rows)));
	CudaSafeCall(hipMalloc((void **)&C->maxnz, sizeof(M->maxnz)));
	CudaSafeCall(hipMalloc((void **)&C->csep, sizeof(M->csep)));
	CudaSafeCall(hipMalloc((void **)&C->jcoef, sizeof(int)*M->rows*M->maxnz));
	CudaSafeCall(hipMalloc((void **)&C->coef, sizeof(real)*M->rows*M->maxnz));

	CudaSafeCall(hipMemcpy((void *)C->type, &M->type, sizeof(M->type), hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy((void *)C->rows, &M->rows, sizeof(M->rows), hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy((void *)C->maxnz, &M->maxnz, sizeof(M->maxnz), hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy((void *)C->csep, &M->csep, sizeof(M->csep), hipMemcpyHostToDevice));

	CudaSafeCall(hipMemcpy((void *)C->jcoef, (void *)M->jcoef, sizeof(int)*M->rows*M->maxnz, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy((void *)C->coef, (void *)M->coef, sizeof(real)*M->rows*M->maxnz, hipMemcpyHostToDevice));

}

void __device__ sprDiff(real* fv, cudasparse* cudaMatrixINT, int i2d, int totpoints, real* v, real beta) {
	for (int i = 0; i < *cudaMatrixINT->maxnz; i++) {
		fv[i2d] += (1.0 / beta)*cudaMatrixINT->coef[i2d + i * totpoints] * v[cudaMatrixINT->jcoef[i2d + i * totpoints] - 1];
		//if (i2d == printv) printf("%d: %.31f, %.31f, %.31f, %.31f\n", cudaMatrixINT.jcoef[i2d + i * totpoints], cudaMatrixINT.coef[i2d + i * totpoints], g_dev.vmp[cudaMatrixINT.jcoef[i2d + i * totpoints] - 1], (1.0 / beta)*cudaMatrixINT.coef[i2d + i * totpoints] * g_dev.vmp[cudaMatrixINT.jcoef[i2d + i * totpoints] - 1], fv);
	}
	return;
}