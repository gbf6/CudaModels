#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include "hip/hip_runtime.h"
#include ""


#include "../common/CudaSafeCall.h"

#include "LRDglobalVariables.cuh"
#include "typedefSparse.h"
#include "sparsePrototypes.cuh"
#include "typedefLRD.h"

#include "parseInput.h"

#include "LRDhostPrototypes.h"
#include "LRDdevicePrototypes.cuh"

real LRDBacNav_RestVoltage = LRDBacNav_RestVoltage_0;
__device__ real LRDBacNav_cm = LRD_cm_0;
__device__ real LRDBacNav_Gna = LRD_Gna_0;
__device__ real LRDBacNav_Gtca = LRD_Gtca_0;
__device__ real LRDBacNav_Gkp = LRD_Gkp_0;
__device__ real LRDBacNav_Gitodv = LRD_Gitodv_0;
__device__ real LRDBacNav_Gcab = LRD_Gcab_0;
__device__ real LRDBacNav_Gnab = LRD_Gnab_0;
__device__ real LRDBacNav_ito = LRD_ito_0;
__device__ real LRDBacNav_ikna = LRD_ikna_0;
__device__ real LRDBacNav_ikatp = LRD_ikatp_0;
__device__ real LRDBacNav_insna = LRD_insna_0;
__device__ real LRDBacNav_insk = LRD_insk_0;
__device__ real LRDBacNav_cleft = LRD_cleft_0;
__device__ real BacNav_Gna = BacNav_Gna_0;


void LRD_init(char** res) {
	rword resources[] = {
	{ "LRDBacNav_IV",	  1007 },
	{ "LRDBacNav_Node",	  1100 },
	{ "LRDBacNav_Nodetype", 1100 },
	{ "LRDBacNav_Patch",	  1007 },
	{ "LRDBacNav_Type",	  1100 },
	{ "LRDBacNav_Vr",	      1008 },
	{ "LRDBacNav_Vrest",	  1008 },
	{ "LRDBacNav_Cm",       1009 },
    { "LRDBacNav_Gna",      1112 },
    { "LRDBacNav_Gtca",     1113 },
    { "LRDBacNav_Gkp",      1114 },
    { "LRDBacNav_Gitodv",   1115 },
    { "LRDBacNav_Gcab",     1116 },
    { "LRDBacNav_Gnab",     1117 },
    { "LRDBacNav_ito",      1118 },
    { "LRDBacNav_ikna",     1119 },
    { "LRDBacNav_ikatp",    1120 },
    { "LRDBacNav_insna",    1121 }, 
    { "LRDBacNav_insk",     1122 },
    { "LRDBacNav_cleft",    1123 },
    { "LRDBacNav_BacNavFactor", 1124},
	{ NULL, 0 }
	};

	int i, j, c, r;
	int cmd;
	real temp;

	temp = GetRealValue(res[i]);
	hipMemcpyToSymbol(HIP_SYMBOL(LRD_cm), (void *)&temp, sizeof(real), 0, hipMemcpyHostToDevice);
			

	i = 0;
	while( res[i] != NULL ) {
		cmd = FindCommand( resources, res[i] );
		switch( cmd ) {
			case 1007:
				/*iv = GetRealArray( res[i] );
				p = (real*)(&LRDBacNav_RestPatch);
				c  = GetNumValues( res[i] );
				if( c > LRDBacNav_PatchSize ) {
					c = LRDBacNav_PatchSize;
				}
				for(j=0;j<c;j++) {
					p[j] = iv[j];
				}*/
				break;
			case 1008:
				LRDBacNav_RestVoltage = GetRealValue( res[i] );
				break;
            case 1009:
				temp = GetRealValue(res[i]);
				hipMemcpyToSymbol(HIP_SYMBOL(LRDBacNav_cm), (void *)&temp, sizeof(real), 0, hipMemcpyHostToDevice);
                break; 
			case 1100:
				//LRDBacNav_NodeType = GetByteValue( res[i] );
				break;
			case 1112:
				temp = GetRealValue(res[i]);
				hipMemcpyToSymbol(HIP_SYMBOL(LRDBacNav_Gna), (void *)&temp, sizeof(real), 0, hipMemcpyHostToDevice);
				break;
			case 1113:
				temp = GetRealValue(res[i]);
				hipMemcpyToSymbol(HIP_SYMBOL(LRDBacNav_Gtca), (void *)&temp, sizeof(real), 0, hipMemcpyHostToDevice);
				break;
			case 1114:
				temp = GetRealValue(res[i]);
				hipMemcpyToSymbol(HIP_SYMBOL(LRDBacNav_Gkp), (void *)&temp, sizeof(real), 0, hipMemcpyHostToDevice);
				break;
			case 1115:
				temp = GetRealValue(res[i]);
				hipMemcpyToSymbol(HIP_SYMBOL(LRDBacNav_Gitodv), (void *)&temp, sizeof(real), 0, hipMemcpyHostToDevice);
				break;
			case 1116:
				temp = GetRealValue(res[i]);
				hipMemcpyToSymbol(HIP_SYMBOL(LRDBacNav_Gcab), (void *)&temp, sizeof(real), 0, hipMemcpyHostToDevice);
				break;
			case 1117:
				temp = GetRealValue(res[i]);
				hipMemcpyToSymbol(HIP_SYMBOL(LRDBacNav_Gnab), (void *)&temp, sizeof(real), 0, hipMemcpyHostToDevice);
				break;
            case 1118:
				temp = GetRealValue(res[i]);
				hipMemcpyToSymbol(HIP_SYMBOL(LRDBacNav_ito), (void *)&temp, sizeof(real), 0, hipMemcpyHostToDevice);
	            break;
            case 1119:
				temp = GetRealValue(res[i]);
				hipMemcpyToSymbol(HIP_SYMBOL(LRDBacNav_ikna), (void *)&temp, sizeof(real), 0, hipMemcpyHostToDevice);
                break;
            case 1120: 
				temp = GetRealValue(res[i]);
				hipMemcpyToSymbol(HIP_SYMBOL(LRDBacNav_ikatp), (void *)&temp, sizeof(real), 0, hipMemcpyHostToDevice);
                break;
            case 1121:
				temp = GetRealValue(res[i]);
				hipMemcpyToSymbol(HIP_SYMBOL(LRDBacNav_insna), (void *)&temp, sizeof(real), 0, hipMemcpyHostToDevice);
                break;
            case 1122:
				temp = GetRealValue(res[i]);
				hipMemcpyToSymbol(HIP_SYMBOL(LRDBacNav_insk), (void *)&temp, sizeof(real), 0, hipMemcpyHostToDevice);
                break;
            case 1123:
				temp = GetRealValue(res[i]);
				hipMemcpyToSymbol(HIP_SYMBOL(LRDBacNav_cleft), (void *)&temp, sizeof(real), 0, hipMemcpyHostToDevice);
                break;
		    case 1124:
				temp = GetRealValue(res[i]);
				hipMemcpyToSymbol(HIP_SYMBOL(BacNav_Gna), (void *)&temp, sizeof(real), 0, hipMemcpyHostToDevice);
		}
		i++;
	}
}

void LRD_gateinit(int memSize, size_t* pitch, gateType* gate_h, gateType* gate_dev, gateType* gate_devF) {

	hipHostAlloc((void**)&(gate_h->vm), memSize, 0);
	hipHostAlloc((void**)&(gate_h->m), memSize, 0);
	hipHostAlloc((void**)&(gate_h->h), memSize, 0);
	hipHostAlloc((void**)&(gate_h->j), memSize, 0);
	hipHostAlloc((void**)&(gate_h->mb), memSize, 0);
	hipHostAlloc((void**)&(gate_h->hb), memSize, 0);
	hipHostAlloc((void**)&(gate_h->d), memSize, 0);
	hipHostAlloc((void**)&(gate_h->f), memSize, 0);
	hipHostAlloc((void**)&(gate_h->b), memSize, 0);
	hipHostAlloc((void**)&(gate_h->g), memSize, 0);
	hipHostAlloc((void**)&(gate_h->xr), memSize, 0);
	hipHostAlloc((void**)&(gate_h->xs1), memSize, 0);
	hipHostAlloc((void**)&(gate_h->xs2), memSize, 0);
	hipHostAlloc((void**)&(gate_h->zdv), memSize, 0);
	hipHostAlloc((void**)&(gate_h->ydv), memSize, 0);
	hipHostAlloc((void**)&(gate_h->nai), memSize, 0);
	hipHostAlloc((void**)&(gate_h->ki), memSize, 0);
	hipHostAlloc((void**)&(gate_h->nsr), memSize, 0);
	hipHostAlloc((void**)&(gate_h->nao), memSize, 0);
	hipHostAlloc((void**)&(gate_h->ko), memSize, 0);
	hipHostAlloc((void**)&(gate_h->cao), memSize, 0);

	hipHostAlloc((void**)&(gate_h->cai), memSize, 0);
	hipHostAlloc((void**)&(gate_h->jsr), memSize, 0);
	hipHostAlloc((void**)&(gate_h->caiont), memSize, 0);
	hipHostAlloc((void**)&(gate_h->BOOL), memSize, 0);
	hipHostAlloc((void**)&(gate_h->tcicr), memSize, 0);
	hipHostAlloc((void**)&(gate_h->tjsrol), memSize, 0);
	hipHostAlloc((void**)&(gate_h->dcaiont), memSize, 0);

	// Allocate device memory arrays
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->vm, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->m, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->h, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->j, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->mb, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->hb, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->d, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->f, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->b, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->g, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->xr, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->xs1, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->xs2, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->zdv, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->ydv, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->nai, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->ki, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->nsr, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->nao, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->ko, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->cao, pitch,
		memSize, 1));

	CudaSafeCall(hipMallocPitch((void **)&gate_dev->cai, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->jsr, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->caiont, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->BOOL, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->tcicr, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->tjsrol, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->dcaiont, pitch,
		memSize, 1));

	// Allocate device forward memory arrays
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->vm, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->m, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->h, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->j, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->mb, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->hb, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->d, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->f, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->b, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->g, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->xr, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->xs1, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->xs2, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->zdv, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->ydv, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->nai, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->ki, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->nsr, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->nao, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->ko, pitch,
		memSize, 1));

	CudaSafeCall(hipMallocPitch((void **)&gate_devF->cai, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->jsr, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->caiont, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->BOOL, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->tcicr, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->tjsrol, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->dcaiont, pitch,
		memSize, 1));

	puts("\nFinished allocating device arrays\n");

	int totpoints = (int)memSize / sizeof(real);

	for (int idx = 0; idx < totpoints; idx++) {
		gate_h->vm[idx] = LRD_RestVoltage;

		gate_h->m[idx] = 0.0008;
		gate_h->h[idx] = 0.993771;
		gate_h->j[idx] = 0.995727;
		gate_h->mb[idx] = 0.000094;
		gate_h->hb[idx] = 0.8231;
		gate_h->d[idx] = 3.210618e-06;
		gate_h->f[idx] = 0.999837;
		gate_h->b[idx] = 0.000970231;
		gate_h->g[idx] = 0.994305;
		gate_h->xr[idx] = 0.000124042;
		gate_h->xs1[idx] = 0.00445683;
		gate_h->xs2[idx] = 0.00445683;
		gate_h->zdv[idx] = 0.0120892;
		gate_h->ydv[idx] = 0.999978;
		gate_h->nai[idx] = 9.0;
		gate_h->ki[idx] = 141.2;
		gate_h->nsr[idx] = 1.838;
		gate_h->nao[idx] = 140;
		gate_h->ko[idx] = 4.5;
		gate_h->cao[idx] = 1.8;

		gate_h->cai[idx] = 0.00006;
		gate_h->jsr[idx] = 1.838;
		gate_h->caiont[idx] = 0;
		gate_h->BOOL[idx] = 0;
		gate_h->tcicr[idx] = -25;
		gate_h->tjsrol[idx] = -25;
		gate_h->dcaiont[idx] = 0;
	}

	CudaSafeCall(hipMemcpy2D((void *)gate_dev->vm, *pitch, (void *)gate_h->vm,
		memSize, memSize, 1, hipMemcpyHostToDevice));

	CudaSafeCall(hipMemcpy2D((void *)gate_dev->m, *pitch, (void *)gate_h->m,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->h, *pitch, (void *)gate_h->h,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->j, *pitch, (void *)gate_h->j,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->mb, *pitch, (void *)gate_h->mb,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->hb, *pitch, (void *)gate_h->hb,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->d, *pitch, (void *)gate_h->d,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->f, *pitch, (void *)gate_h->f,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->b, *pitch, (void *)gate_h->b,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->g, *pitch, (void *)gate_h->g,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->xr, *pitch, (void *)gate_h->xr,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->xs1, *pitch, (void *)gate_h->xs1,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->xs2, *pitch, (void *)gate_h->xs2,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->zdv, *pitch, (void *)gate_h->zdv,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->ydv, *pitch, (void *)gate_h->ydv,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->nai, *pitch, (void *)gate_h->nai,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->ki, *pitch, (void *)gate_h->ki,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->nsr, *pitch, (void *)gate_h->nsr,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->nao, *pitch, (void *)gate_h->nao,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->ko, *pitch, (void *)gate_h->ko,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->cao, *pitch, (void *)gate_h->cao,
		memSize, memSize, 1, hipMemcpyHostToDevice));

	CudaSafeCall(hipMemcpy2D((void *)gate_dev->cai, *pitch, (void *)gate_h->cai,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->jsr, *pitch, (void *)gate_h->jsr,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->caiont, *pitch, (void *)gate_h->caiont,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->BOOL, *pitch, (void *)gate_h->BOOL,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->tcicr, *pitch, (void *)gate_h->tcicr,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->tjsrol, *pitch, (void *)gate_h->tjsrol,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->dcaiont, *pitch, (void *)gate_h->dcaiont,
		memSize, memSize, 1, hipMemcpyHostToDevice));


	CudaSafeCall(hipMemcpy2D((void *)gate_devF->vm, *pitch, (void *)gate_h->vm,
		memSize, memSize, 1, hipMemcpyHostToDevice));

	CudaSafeCall(hipMemcpy2D((void *)gate_devF->m, *pitch, (void *)gate_h->m,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->h, *pitch, (void *)gate_h->h,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->j, *pitch, (void *)gate_h->j,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->mb, *pitch, (void *)gate_h->mb,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->hb, *pitch, (void *)gate_h->hb,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->d, *pitch, (void *)gate_h->d,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->f, *pitch, (void *)gate_h->f,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->b, *pitch, (void *)gate_h->b,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->g, *pitch, (void *)gate_h->g,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->xr, *pitch, (void *)gate_h->xr,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->xs1, *pitch, (void *)gate_h->xs1,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->xs2, *pitch, (void *)gate_h->xs2,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->zdv, *pitch, (void *)gate_h->zdv,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->ydv, *pitch, (void *)gate_h->ydv,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->nai, *pitch, (void *)gate_h->nai,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->ki, *pitch, (void *)gate_h->ki,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->nsr, *pitch, (void *)gate_h->nsr,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->nao, *pitch, (void *)gate_h->nao,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->ko, *pitch, (void *)gate_h->ko,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->cao, *pitch, (void *)gate_h->cao,
		memSize, memSize, 1, hipMemcpyHostToDevice));

	CudaSafeCall(hipMemcpy2D((void *)gate_devF->cai, *pitch, (void *)gate_h->cai,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->jsr, *pitch, (void *)gate_h->jsr,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->caiont, *pitch, (void *)gate_h->caiont,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->BOOL, *pitch, (void *)gate_h->BOOL,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->tcicr, *pitch, (void *)gate_h->tcicr,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->tjsrol, *pitch, (void *)gate_h->tjsrol,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->dcaiont, *pitch, (void *)gate_h->dcaiont,
		memSize, memSize, 1, hipMemcpyHostToDevice));

	real** qpH = (real**)malloc(sizeof(real *)*gate_h->qpl);
	int i = 0;
	qpH[i++] = gate_devF->m;
	qpH[i++] = gate_devF->h;
	qpH[i++] = gate_devF->j;
	qpH[i++] = gate_devF->mb;
	qpH[i++] = gate_devF->hb;
	qpH[i++] = gate_devF->f;
	qpH[i++] = gate_devF->b;
	qpH[i++] = gate_devF->g;
	qpH[i++] = gate_devF->xr;
	qpH[i++] = gate_devF->xs1;
	qpH[i++] = gate_devF->xs2;
	qpH[i++] = gate_devF->zdv;
	qpH[i++] = gate_devF->ydv;
	qpH[i++] = gate_devF->nai;
	qpH[i++] = gate_devF->ki;
	qpH[i++] = gate_devF->nsr;
	qpH[i++] = gate_devF->nao;
	qpH[i++] = gate_devF->ko;
	qpH[i++] = gate_devF->cao;

	CudaSafeCall(hipMemcpy((void *)gate_devF->qp, (void*)qpH, sizeof(real *)*gate_h->qpl, hipMemcpyHostToDevice));

	i = 0;
	qpH[i++] = gate_dev->m;
	qpH[i++] = gate_dev->h;
	qpH[i++] = gate_dev->j;
	qpH[i++] = gate_dev->mb;
	qpH[i++] = gate_dev->hb;
	qpH[i++] = gate_dev->f;
	qpH[i++] = gate_dev->b;
	qpH[i++] = gate_dev->g;
	qpH[i++] = gate_dev->xr;
	qpH[i++] = gate_dev->xs1;
	qpH[i++] = gate_dev->xs2;
	qpH[i++] = gate_dev->zdv;
	qpH[i++] = gate_dev->ydv;
	qpH[i++] = gate_dev->nai;
	qpH[i++] = gate_dev->ki;
	qpH[i++] = gate_dev->nsr;
	qpH[i++] = gate_dev->nao;
	qpH[i++] = gate_dev->ko;
	qpH[i++] = gate_dev->cao;

	CudaSafeCall(hipMemcpy((void *)gate_dev->qp, (void*)qpH, sizeof(real *)*gate_h->qpl, hipMemcpyHostToDevice));
	

	CudaCheckError();

	puts("\nFinished initializing device arrays\n");

}

void LRD_sync(int memSize, size_t pitch, gateType* gate_h, gateType* gate_dev) {

	CudaSafeCall(hipMemcpy2D((void *)gate_h->vm, *pitch, (void *)gate_dev->vm,
		memSize, memSize, 1, hipMemcpyDeviceToHost));

	CudaSafeCall(hipMemcpy2D((void *)gate_h->m, *pitch, (void *)gate_dev->m,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->h, *pitch, (void *)gate_dev->h,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->j, *pitch, (void *)gate_dev->j,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->mb, *pitch, (void *)gate_dev->mb,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->hb, *pitch, (void *)gate_dev->hb,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->d, *pitch, (void *)gate_dev->d,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->f, *pitch, (void *)gate_dev->f,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->b, *pitch, (void *)gate_dev->b,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->g, *pitch, (void *)gate_dev->g,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->xr, *pitch, (void *)gate_dev->xr,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->xs1, *pitch, (void *)gate_dev->xs1,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->xs2, *pitch, (void *)gate_dev->xs2,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->zdv, *pitch, (void *)gate_dev->zdv,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->ydv, *pitch, (void *)gate_dev->ydv,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->nai, *pitch, (void *)gate_dev->nai,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->ki, *pitch, (void *)gate_dev->ki,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->nsr, *pitch, (void *)gate_dev->nsr,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->nao, *pitch, (void *)gate_dev->nao,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->ko, *pitch, (void *)gate_dev->ko,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->cao, *pitch, (void *)gate_dev->cao,
		memSize, memSize, 1, hipMemcpyDeviceToHost));

	CudaSafeCall(hipMemcpy2D((void *)gate_h->cai, *pitch, (void *)gate_dev->cai,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->jsr, *pitch, (void *)gate_dev->jsr,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->caiont, *pitch, (void *)gate_dev->caiont,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->BOOL, *pitch, (void *)gate_dev->BOOL,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->tcicr, *pitch, (void *)gate_dev->tcicr,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->tjsrol, *pitch, (void *)gate_dev->tjsrol,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->dcaiont, *pitch, (void *)gate_dev->dcaiont,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
}

void LRD_exit(int memSize, size_t pitch, gateType* gate_h, gateType* gate_dev, gateType* gate_devF, sparse* MatrixINT, cudasparse* cudaMatrixINT){
	// Free gate host and device memory
	hipHostFree(gate_h->vm);

	hipHostFree(gate_h->m);
	hipHostFree(gate_h->h);
	hipHostFree(gate_h->j);
	hipHostFree(gate_h->mb);
	hipHostFree(gate_h->hb);
	hipHostFree(gate_h->d);
	hipHostFree(gate_h->f);
	hipHostFree(gate_h->b);
	hipHostFree(gate_h->g);
	hipHostFree(gate_h->xr);
	hipHostFree(gate_h->xs1);
	hipHostFree(gate_h->xs2);
	hipHostFree(gate_h->zdv);
	hipHostFree(gate_h->ydv);
	hipHostFree(gate_h->nai);
	hipHostFree(gate_h->ki);
	hipHostFree(gate_h->nsr);
	hipHostFree(gate_h->nao);
	hipHostFree(gate_h->ko);
	hipHostFree(gate_h->cao);

	hipHostFree(gate_h->cai);
	hipHostFree(gate_h->jsr);
	hipHostFree(gate_h->caiont);
	hipHostFree(gate_h->BOOL);
	hipHostFree(gate_h->tcicr);
	hipHostFree(gate_h->tjsrol);
	hipHostFree(gate_h->dcaiont);
	hipHostFree(gate_h->qp);


	hipFree(gate_dev->vm);

	hipFree(gate_dev->m);
	hipFree(gate_dev->h);
	hipFree(gate_dev->j);
	hipFree(gate_dev->mb);
	hipFree(gate_dev->hb);
	hipFree(gate_dev->d);
	hipFree(gate_dev->f);
	hipFree(gate_dev->b);
	hipFree(gate_dev->g);
	hipFree(gate_dev->xr);
	hipFree(gate_dev->xs1);
	hipFree(gate_dev->xs2);
	hipFree(gate_dev->zdv);
	hipFree(gate_dev->ydv);
	hipFree(gate_dev->nai);
	hipFree(gate_dev->ki);
	hipFree(gate_dev->nsr);
	hipFree(gate_dev->nao);
	hipFree(gate_dev->ko);
	hipFree(gate_dev->cao);

	hipFree(gate_dev->cai);
	hipFree(gate_dev->jsr);
	hipFree(gate_dev->caiont);
	hipFree(gate_dev->BOOL);
	hipFree(gate_dev->tcicr);
	hipFree(gate_dev->tjsrol);
	hipFree(gate_dev->dcaiont);
	hipFree(gate_dev->qp);


	hipFree(gate_devF->vm);

	hipFree(gate_devF->m);
	hipFree(gate_devF->h);
	hipFree(gate_devF->j);
	hipFree(gate_devF->mb);
	hipFree(gate_devF->hb);
	hipFree(gate_devF->d);
	hipFree(gate_devF->f);
	hipFree(gate_devF->b);
	hipFree(gate_devF->g);
	hipFree(gate_devF->xr);
	hipFree(gate_devF->xs1);
	hipFree(gate_devF->xs2);
	hipFree(gate_devF->zdv);
	hipFree(gate_devF->ydv);
	hipFree(gate_devF->nai);
	hipFree(gate_devF->ki);
	hipFree(gate_devF->nsr);
	hipFree(gate_devF->nao);
	hipFree(gate_devF->ko);
	hipFree(gate_devF->cao);

	hipFree(gate_devF->cai);
	hipFree(gate_devF->jsr);
	hipFree(gate_devF->caiont);
	hipFree(gate_devF->BOOL);
	hipFree(gate_devF->tcicr);
	hipFree(gate_devF->tjsrol);
	hipFree(gate_devF->dcaiont);	
	hipFree(gate_devF->qp);



	hipFree(cudaMatrixINT->type);
	hipFree(cudaMatrixINT->rows);
	hipFree(cudaMatrixINT->maxnz);
	hipFree(cudaMatrixINT->csep);
	hipFree(cudaMatrixINT->jcoef);
	hipFree(cudaMatrixINT->coef);
}


void __device__ GetFDev_LRD(int i2d, int pitch, real beta, real Cm, real t, real dt, int totpoints, real rx, gateType g_dev, gateType g_devF) {

	/*------------------------------------------------------------------------
	* return if outside domain
	*------------------------------------------------------------------------
	*/
	if (i2d >= totpoints) {
		return;
	}

	/* declare variables */ 	                                 
    real vm,BOOL,tcicr,tjsrol,csqn;
    real LRDBacNav_RTF;
    real LRDBacNav_Gkr, LRDBacNav_Gks,LRDBacNav_Gki,LRDBacNav_Gkatp;
    real LRDBacNav_Ena,LRDBacNav_Etca,LRDBacNav_Ekr,LRDBacNav_Eks,LRDBacNav_Eki,LRDBacNav_Ekp;
    real LRDBacNav_Ekna,LRDBacNav_Ekatp,LRDBacNav_Ekdv,LRDBacNav_Ecan,LRDBacNav_Enan;
    real m,h,j,aproto,aproto2,am,bm,ah,bh,aj,bj,mb,hb,taumb, tauhb, minfb, hinfb, Ina;  
    real d,f,dss,dss1,taud,fss,tauf,Ibarca,Ibarna,Ibark;
    real fca,Ilca,Ilcana,Ilcak,Ilcatot;
    real b,g,bss,taub,gss,taug,Itca;
    real xr,r,xrss,tauxr,Ikr;
    real xs1,xs2,xs1ss,xs2ss,tauxs1,tauxs2,Iks; 
    real aki,bki,kin,Ikti;
    real kp,Ikp;
    real Inaca;
    real sigma,fnak,Inak;
    real Ipca;
    real Icab;
    real Inab;
    real pona,pov,Ikna;
    real patp,gkbaratp,Ikatp;
    real Ibarnsna,Ibarnsk,Insna,Insk;
    real rvdv,Ito;
    real azdv,bzdv,tauzdv,zdvss,zdv;
    real aydv,bydv,tauydv,ydvss,ydv;
    real naiont,kiont,caiont,Itotal;
    /*ions*/
    real nao,ko,cao;
    real dnao,dko,dcao;
 
	real nai,ki;
	real dnai,dki;

    real itr;
    real nsr,kleak,ileak,iup,dnsr;
    /* JSR CICR */
    real dcaiont,caiontold;
    real magrel,on,off,irelcicr;
    real greljsrol,ireljsrol;
    real trpn,cmdn;
    real jsr,bjsr,cjsr,djsr; 
    /* cai update here */
    real cai,catotal,bmyo,cmyo,dmyo,gpig,dcai;
    real vcell,ageo,acap,vmyo,vnsr,vjsr,vcleft;
    LRDBacNav_RTF = LRDBacNav_R*LRDBacNav_temp/LRDBacNav_frdy;


	vm        = g_dev.vm[i2d];

	m 		  = g_dev.m[i2d];
	h 		  = g_dev.h[i2d];
	j 		  = g_dev.j[i2d];
	mb 		  = g_dev.mb[i2d];
	hb 		  = g_dev.hb[i2d];
	d 		  = g_dev.d[i2d];
	f 		  = g_dev.f[i2d];
	b 		  = g_dev.b[i2d];
	g 	      = g_dev.g[i2d];
	xr 	      = g_dev.xr[i2d];
	xs1 	  = g_dev.xs1[i2d];
	xs2	      = g_dev.xs2[i2d];
	zdv       = g_dev.zdv[i2d];
	ydv	      = g_dev.ydv[i2d];
	nai 	  = g_dev.nai[i2d];
	ki 	      = g_dev.ki[i2d];
	nsr	      = g_dev.nsr[i2d];
	nao	      = g_dev.nao[i2d];
	ko 	      = g_dev.ko[i2d];
	cao       = g_dev.cao[i2d];

	cai       = g_dev.cai[i2d];
	jsr       = g_dev.jsr[i2d];
	caiont    = g_dev.caiont[i2d];
	BOOL      = g_dev.BOOL[i2d];
	tcicr     = g_dev.tcicr[i2d];
	tjsrol    = g_dev.tjsrol[i2d];
	dcaiont   = g_dev.dcaiont[i2d];

	/*------------------------------------------------------------------------
	* setting local variables
	*------------------------------------------------------------------------
	*/

	real fv = g_devF.vm[i2d];

	/* Declare varying G's and E's */
	LRDBacNav_Gkr   = 0.02614*sqrt(ko/5.4); 
	LRDBacNav_Gks   = 0.433*(1+0.6/(1+pow((0.000038/cai),1.4)));
	LRDBacNav_Gki   = 0.75*(sqrt(ko/5.4));
	LRDBacNav_Gkatp = 0.000195/nicholsarea;

	LRDBacNav_Ena   = (LRDBacNav_RTF)*log(nao/nai);
	LRDBacNav_Etca  = 0.5*(LRDBacNav_RTF)*log(cao/cai);  
	LRDBacNav_Ekr   = (LRDBacNav_RTF)*log(ko/ki);
	LRDBacNav_Eks   = (LRDBacNav_RTF)*log((ko+prnak*nao)/(ki+prnak*nai));
	LRDBacNav_Eki   = (LRDBacNav_RTF)*log(ko/ki);
	LRDBacNav_Ekp   = LRDBacNav_Eki;
	LRDBacNav_Ekna  = LRDBacNav_Ekr; 
	LRDBacNav_Ekatp = LRDBacNav_Ekr; 
	LRDBacNav_Ekdv  = LRDBacNav_Ekr;
	LRDBacNav_Ecan  = LRDBacNav_Etca;
	LRDBacNav_Enan  = LRDBacNav_Ena;

	/* Na current [15] */
	aproto  = 1-1.0/(1+exp(-(vm+40)/0.024));
	am = 0.32*(vm+47.13)/(1-exp(-0.1*(vm+47.13)));
	bm = 0.08*exp(-vm/11);
	ah = aproto*0.135*exp((80+vm)/-6.8);
	bh = (1-aproto)/(0.13*(1+exp((vm+10.66)/(-11.1)))) + aproto*(3.56*exp(0.079*vm)+3.1*pow(10,5)*exp(0.35*vm));
	aj = aproto*(-127140*exp(0.2444*vm)-0.00003474*exp(-0.04391*vm))*((vm+37.78)/(1+exp(0.311*(vm+79.23))));
	bj = (1-aproto)*(0.3*exp(-2.535*pow(10,-7)*vm)/(1+exp(-0.1*(vm+32))))+aproto*(0.1212*exp(-0.01052*vm))/(1+exp(-0.1378*(vm+40.14)));        
	/* BacNav component */
	minfb = (1.0/(1.0+exp((vm+28.34)/(-5.33))));
	hinfb = (1.0-1.0/(1.0+exp((-77.21-vm)/8.32)));
	taumb = (86.37/(exp((vm+82.74)/17.64) + exp(-(vm+ 6.008)/3.337)) + .4844);
	tauhb = (96.17-(96.17-10.45)/(1.0+exp((-23.26-vm)/2.529)));
	Ina=(LRDBacNav_Gna*(m*m*m*h*j)+BacNav_Gna*(mb*mb*mb*hb))*(vm-LRDBacNav_Ena);

	/* L-type Calcium current [14,15] */
	dss  = 1/(1+exp(-(vm+10)/6.24));
	dss1 = 1/(1+exp(-(vm+60)/0.024));
	taud = dss*(1-exp(-(vm+10)/6.24))/(0.035*(vm+10));
	dss = dss * dss1;
	fss  = (1/(1+exp((vm+32)/8)))+(0.6/(1+exp((50-vm)/20)));
	tauf = 1/(0.0197*exp(-0.0337*0.0337*(vm+10)*(vm+10))+0.02);

	Ibarca = pca*zca*zca*((vm*LRDBacNav_frdy)/(LRDBacNav_RTF))*((gacai*cai*exp((zca*vm)/(LRDBacNav_RTF))-gacao*cao)/(exp((zca*vm)/(LRDBacNav_RTF))-1));
	Ibarna = pna*zna*zna*((vm*LRDBacNav_frdy)/(LRDBacNav_RTF))*((ganai*nai*exp((zna*vm)/(LRDBacNav_RTF))-ganao*nao)/(exp((zna*vm)/(LRDBacNav_RTF))-1));
	Ibark  = pk*zk*zk*((vm*LRDBacNav_frdy)/(LRDBacNav_RTF))*((gaki*ki*exp((zk*vm)/(LRDBacNav_RTF))-gako*ko)/(exp((zk*vm)/(LRDBacNav_RTF))-1));

	fca = 1/(1+cai/kmca);

	Ilca    = d*f*fca*Ibarca;
	Ilcana  = d*f*fca*Ibarna;
	Ilcak   = d*f*fca*Ibark;	
	Ilcatot = Ilca+Ilcana+Ilcak;

	/* T-type Calcium current [13] */
	bss  = 1/(1+exp(-(vm+14)/10.8));
	taub = 3.7+6.1/(1+exp((vm+25)/4.5));
	gss  = 1/(1+exp((vm+60)/5.6));
	aproto2 = 1-1/(1+exp(-vm/0.0024));
	taug = aproto2*(-0.875*vm+12.0)+12.0*(1-aproto2);
	Itca = LRDBacNav_Gtca*b*b*g*(vm-LRDBacNav_Etca);

	/* K current - Rapid [13] */
	xrss  = 1/(1+exp(-(vm+21.5)/7.5));
	tauxr = 1/(0.00138*(vm+14.2)/(1-exp(-0.123*(vm+14.2)))+0.00061*(vm+38.9)/(exp(0.145*(vm+38.9))-1));

	r = 1/(1+exp((vm+9)/22.4));

	Ikr = LRDBacNav_Gkr*xr*r*(vm-LRDBacNav_Ekr);

	/* K current - Slow [10,13] */
	xs1ss  = 1/(1+exp(-(vm-1.5)/16.7));
	xs2ss  = xs1ss;
	tauxs1 = 1/(0.0000719*(vm+30)/(1-exp(-0.148*(vm+30)))+0.000131*(vm+30)/(exp(0.0687*(vm+30))-1));
	tauxs2 = 4*tauxs1;

	Iks = LRDBacNav_Gks*xs1*xs2*(vm-LRDBacNav_Eks);

	/* K current -  Time independent [15] */
	aki = 1.02/(1+exp(0.2385*(vm-LRDBacNav_Eki-59.215)));
	bki = (0.49124*exp(0.08032*(vm-LRDBacNav_Eki+5.476))+exp(0.06175*(vm-LRDBacNav_Eki-594.31)))/(1+exp(-0.5143*(vm-LRDBacNav_Eki+4.753)));	

	kin = aki/(aki+bki);

	Ikti = LRDBacNav_Gki*kin*(vm-LRDBacNav_Eki);

	/* K current - Plateau [15] */
	kp  = 1/(1+exp((7.488-vm)/5.98));	

	Ikp = LRDBacNav_Gkp*kp*(vm-LRDBacNav_Ekp);

	/* Na-Ca exchanger [6,14,15] */
	Inaca = c1*exp((gammas-1)*vm/(LRDBacNav_RTF))*((exp(vm/(LRDBacNav_RTF))*nai*nai*nai*cao-nao*nao*nao*cai)/(1+c2*exp((gammas-1)*vm/(LRDBacNav_RTF))*(exp(vm/(LRDBacNav_RTF))*nai*nai*nai*cao+nao*nao*nao*cai)));

	/* Na-K pump [15] */
	sigma = (exp(nao/67.3)-1)/7;
	fnak  = 1/(1+0.1245*exp((-0.1*vm)/(LRDBacNav_RTF))+0.0365*sigma*exp((-vm)/(LRDBacNav_RTF)));

	Inak = Ibarnak*fnak*(1/(1+kmnai*kmnai/(nai*nai)))*(ko/(ko+kmko));

	/* Sarcolemmal Ca pump [15] */
	Ipca = (Ibarpca*cai)/(kmpca+cai);

	/* Ca background current [15] */
	Icab = LRDBacNav_Gcab*(vm-LRDBacNav_Ecan);

	/* Na background current [15] */
	Inab = LRDBacNav_Gnab*(vm-LRDBacNav_Enan);          

	/* Na activated K current [6] */
	pona = 0.85/(1+pow((kdkna/nai),2.8));
	pov  = 0.8-(0.65/(1+exp((vm+125)/15)));

	Ikna = LRDBacNav_ikna*LRDBacNav_Gkna*pona*pov*(vm-LRDBacNav_Ekna);

	/* ATP sensitive K current [11] */
	patp = 1/(1+(pow((atpi/katp),hatp)));
	gkbaratp = LRDBacNav_Gkatp*patp*(pow((ko/4),natp));

	Ikatp = LRDBacNav_ikatp*gkbaratp*(vm-LRDBacNav_Ekatp);     

	/* Non-specific Ca-activated current [14,15] */
	Ibarnsna = pnsca*zna*zna*((vm*LRDBacNav_frdy)/(LRDBacNav_RTF))*((ganai*nai*exp((zna*vm)/(LRDBacNav_RTF))-ganao*nao)/(exp((zna*vm)/(LRDBacNav_RTF))-1));
	Ibarnsk  = pnsca*zk*zk*((vm*LRDBacNav_frdy)/(LRDBacNav_RTF))*((gaki*ki*exp((zk*vm)/(LRDBacNav_RTF))-gako*ko)/(exp((zk*vm)/(LRDBacNav_RTF))-1));

	Insna = LRDBacNav_insna*Ibarnsna/(1+kmnsca*kmnsca*kmnsca/(cai*cai*cai)); 
	Insk = LRDBacNav_insk*Ibarnsk/(1+kmnsca*kmnsca*kmnsca/(cai*cai*cai));   

	/* Transient outward current */
	rvdv = exp(vm/100);

	azdv = (10*exp((vm-40)/25))/(1+exp((vm-40)/25));
	bzdv = (10*exp(-(vm+90)/25))/(1+exp(-(vm+90)/25));
	tauzdv = 1/(azdv+bzdv);
	zdvss = azdv/(azdv+bzdv);

	aydv = 0.015/(1+exp((vm+60)/5));
	bydv = (0.1*exp((vm+25)/5))/(1+exp((vm+25)/5));
	tauydv = 1/(aydv+bydv);
	ydvss = aydv/(aydv+bydv);

	Ito = LRDBacNav_ito*LRDBacNav_Gitodv*zdv*zdv*zdv*ydv*rvdv*(vm-LRDBacNav_Ekdv);

	/* Summing currents (inactive currents are set to zero with activation variables) */
	naiont = Ina+Inab+Ilcana+3*Inak+3*Inaca+Insna;
	kiont  = Ikr+Iks+Ikti+Ikp+Ilcak+-2*Inak+Insk+Ito+Ikna+Ikatp;  
	caiont = Ilca+Icab+Ipca-2*Inaca+Itca;

	Itotal = LRDBacNav_cm*(naiont+kiont+caiont);     /* uA/cm2 */

	if (((t-tcicr)>80) && (vm<-30)) {
		BOOL = 0;  
		g_dev.BOOL[i2d] = BOOL;                                                                  
	}

	/* Put voltage update here  */
	fv += -Itotal;
	g_devF.vm[i2d] = fv;

	/* change in cleft concentration */
	dnao = LRDBacNav_cleft*((nabm-nao)/taudiff+naiont*acap*LRDBacNav_cm/(vcleft*LRDBacNav_frdy));
	dko = LRDBacNav_cleft*((kbm-ko)/taudiff+kiont*acap*LRDBacNav_cm/(vcleft*LRDBacNav_frdy));
	dcao = LRDBacNav_cleft*((cabm-cao)/taudiff+caiont*acap*LRDBacNav_cm/(vcleft*LRDBacNav_frdy*2));

	/* change in nai and ki concentration */
	dnai = -LRDBacNav_cm*(naiont*acap)/(vmyo*zna*LRDBacNav_frdy);     /* dnai/dt */
	dki = -LRDBacNav_cm*(kiont*acap)/(vmyo*zk*LRDBacNav_frdy);        /* dki/dt */

	/* change in itr [14] */ 
	itr = (nsr-jsr)/tautr;                           

	/* change in nsr [14] */
	kleak = iupbar/nsrbar;
	ileak = kleak*nsr;
	iup   = iupbar*cai/(cai+kmup);

	dnsr = (iup-ileak-itr*vjsr/vnsr);          /* dnsr/dt */


	/* Calcium-induced-calcium-release (CICR) criteia [6] */
	if ((vm>-35) && (((caiont-caiontold)/dt)<dcaiont) && (BOOL==0)){
		BOOL  = 1;
		tcicr = t;
		g_dev.BOOL[i2d] = BOOL;
		g_dev.tcicr[i2d] = tcicr;   /* changes reference time */
	}   

	on  = 1/(1+exp((-(t-tcicr)+4)/.5));
	off = 1-on;
	magrel = 1/(1+exp(((Ilca+Icab+Ipca-2*Inaca+Itca)+5)/0.9));

	irelcicr = gmaxrel*on*off*magrel*(jsr-cai);

	/* JSR Calciium overload [13] */
	greljsrol = grelbarjsrol*(1-exp(-(t-tjsrol)/tauon))*exp(-(t-tjsrol)/tauoff);
	ireljsrol = greljsrol*(jsr-cai);

	csqn = csqnbar*(jsr/(jsr+kmcsqn));

	djsr = dt*(itr-irelcicr-ireljsrol);

	bjsr = csqnbar-csqn-djsr-jsr+kmcsqn;
	cjsr = kmcsqn*(csqn+djsr+jsr);

	jsr =(sqrt(bjsr*bjsr+4*cjsr)-bjsr)/2; 

	/* Calcium buffers in myoplasm [15] */
	trpn = trpnbar*(cai/(cai+kmtrpn));
	cmdn = cmdnbar*(cai/(cai+kmcmdn));
     
	/* change in cai concentration [13] */
	dcai = -dt*(((LRDBacNav_cm*caiont*acap)/(vmyo*zca*LRDBacNav_frdy))+((iup-ileak)*vnsr/vmyo)-(irelcicr*vjsr/vmyo)-(ireljsrol*vjsr/vmyo));

	catotal = trpn+cmdn+dcai+cai;

	bmyo = cmdnbar+trpnbar-catotal+kmtrpn+kmcmdn;
	cmyo = (kmcmdn*kmtrpn)-(catotal*(kmtrpn+kmcmdn))+(trpnbar*kmcmdn)+(cmdnbar*kmtrpn);
	dmyo = -kmtrpn*kmcmdn*catotal;
	gpig = sqrt(bmyo*bmyo-3*cmyo);

	cai = ((2*gpig/3)*cos(acos((9*bmyo*cmyo-2*bmyo*bmyo*bmyo-27*dmyo)/(2*pow((bmyo*bmyo-3*cmyo),1.5)))/3)-(bmyo/3)); 

	/* Calcium overload criteria [15] */
	if((csqn>=csqnth) && ((t-tjsrol)>50)){
		printf("Spontaneous Release occured at time %lf at node %i\n",t,i);
		tjsrol = t;  	       
		g_dev.tjsrol[i2d] = tjsrol;                /* changes reference time */
	}

    g_devF.m[i2d]    = am*(1.0-m) - bm*m;
    g_devF.h[i2d]    = ah*(1.0-h) - bh*h;
    g_devF.j[i2d]    = aj*(1.0-j) - bj*j;
    g_devF.mb[i2d]   = (minfb - mb)/taumb;
	g_devF.hb[i2d]   = (hinfb - hb)/tauhb;
    g_devF.d[i2d]    = (dss/taud)*(1-d)-(1-dss)*(d/taud);
    g_devF.f[i2d]    = (fss/tauf)*(1-f)-(1-fss)*(f/tauf);
    g_devF.b[i2d]    = (bss/taub)*(1-b)-(1-bss)*(b/taub);
    g_devF.g[i2d]    = (gss/taug)*(1-g)-(1-gss)*(g/taug);
    g_devF.xr[i2d]   = (xrss/tauxr)*(1-xr)-(1-xrss)*(xr/tauxr);
    g_devF.xs1[i2d]  = (xs1ss/tauxs1)*(1-xs1)-(1-xs1ss)*(xs1/tauxs1);
    g_devF.xs2[i2d]  = (xs2ss/tauxs2)*(1-xs2)-(1-xs2ss)*(xs2/tauxs2);
    g_devF.zdv[i2d]  = (zdvss/tauzdv)*(1-zdv)-(1-zdvss)*(zdv/tauzdv);
    g_devF.ydv[i2d]  = (ydvss/tauydv)*(1-ydv)-(1-ydvss)*(ydv/tauydv);
    g_devF.nai[i2d]  = dnai;
    g_devF.ki[i2d]   = dki;
    g_devF.nsr[i2d]  = dnsr;
    g_devF.nao[i2d]  = dnao; 
    g_devF.ko[i2d]   = dko; 
    g_devF.cao[i2d]  = dcao;


  /* assign Temp variables to memory */    
    g_devF.caiont[i2d]  = caiont;
    g_devF.cai[i2d]     = cai; 
    g_devF.jsr[i2d]     = jsr;
    g_devF.dcaiont[i2d] = (caiont-caiontold)/dt;

}