#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include "hip/hip_runtime.h"
#include ""


#include "../common/CudaSafeCall.h"

#include "MaleckarglobalVariables.cuh"
#include "typedefSparse.h"
#include "sparsePrototypes.cuh"
#include "typedefMaleckar.h"

#include "MaleckarhostPrototypes.h"
#include "MaleckardevicePrototypes.cuh"

void Maleckar_gateinit(int memSize, size_t* pitch, gateType* gate_h, gateType* gate_dev, gateType* gate_devF) {

	hipHostAlloc((void**)&(gate_h->vm), memSize, 0);
	hipHostAlloc((void**)&(gate_h->r), memSize, 0);
	hipHostAlloc((void**)&(gate_h->s), memSize, 0);
	

	// Allocate device memory arrays
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->vm, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->r, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->x, pitch,
		memSize, 1));

	// Allocate device forward memory arrays
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->vm, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->r, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->s, pitch,
		memSize, 1));

	puts("\nFinished allocating device arrays\n");

	int totpoints = (int)memSize / sizeof(real);

	for (int idx = 0; idx < totpoints; idx++) {
		gate_h->vm[idx] = Maleckar_RestVoltage;
		gate_h->r[idx] = 5.454e-02;
		gate_h->s[idx] = 9.814e-01;
	}

	CudaSafeCall(hipMemcpy2D((void *)gate_dev->vm, *pitch, (void *)gate_h->vm,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->r, *pitch, (void *)gate_h->r,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->s, *pitch, (void *)gate_h->s,
		memSize, memSize, 1, hipMemcpyHostToDevice));


	CudaSafeCall(hipMemcpy2D((void *)gate_devF->vm, *pitch, (void *)gate_h->vm,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->r, *pitch, (void *)gate_h->r,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->s, *pitch, (void *)gate_h->s,
		memSize, memSize, 1, hipMemcpyHostToDevice));

	real** qpH = (real**)malloc(sizeof(real *)*gate_h->qpl);
	int i = 0;
	qpH[i++] = gate_devF->r;
	qpH[i++] = gate_devF->s;


	CudaSafeCall(hipMemcpy((void *)gate_devF->qp, (void*)qpH, sizeof(real *)*gate_h->qpl, hipMemcpyHostToDevice));

	i = 0;
	qpH[i++] = gate_dev->r;
	qpH[i++] = gate_dev->s;

	CudaSafeCall(hipMemcpy((void *)gate_dev->qp, (void*)qpH, sizeof(real *)*gate_h->qpl, hipMemcpyHostToDevice));
	

	CudaCheckError();

	puts("\nFinished initializing device arrays\n");

}

void Maleckar_sync(int memSize, size_t pitch, gateType* gate_h, gateType* gate_dev) {
	CudaSafeCall(hipMemcpy2D((void *)gate_h->vm, memSize, (void *)gate_dev->vm,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->r, memSize, (void *)gate_dev->r,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->s, memSize, (void *)gate_dev->s,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
}

void Maleckar_exit(int memSize, size_t pitch, gateType* gate_h, gateType* gate_dev, gateType* gate_devF, sparse* MatrixINT, cudasparse* cudaMatrixINT){
	// Free gate host and device memory
	hipHostFree(gate_h->vm); hipHostFree(gate_h->r); hipHostFree(gate_h->s); hipFree(gate_dev->qp); 

	hipFree(gate_dev->vm); hipFree(gate_dev->r); hipFree(gate_dev->s); hipFree(gate_dev->qp); 

	hipFree(gate_devF->vm); hipFree(gate_devF->r); hipFree(gate_devF->s); hipFree(gate_devF->qp); 

	hipFree(cudaMatrixINT->type);
	hipFree(cudaMatrixINT->rows);
	hipFree(cudaMatrixINT->maxnz);
	hipFree(cudaMatrixINT->csep);
	hipFree(cudaMatrixINT->jcoef);
	hipFree(cudaMatrixINT->coef);
}


void __device__ GetFDev_Maleckar(int i2d, int pitch, real beta, real Cm, real t, real dt, int totpoints, real rx, gateType g_dev, gateType g_devF) {

	/*------------------------------------------------------------------------
	* return if outside domain
	*------------------------------------------------------------------------
	*/
	if (i2d >= totpoints) {
		return;
	}

	real   vm;
	real   r, s;
	real   r_inf, s_inf, tau_r, tau_s, aK1, bK1, ENa, EK;
	real   Ikv, Ik1, Inak, Ibna, Iion;

	real vm = g_dev.vm[i2d];
	real r  = g_dev.r[i2d];
	real s  = g_dev.s[i2d];

	/*------------------------------------------------------------------------
	* setting local variables
	*------------------------------------------------------------------------
	*/
	
	real fv = g_devF.vm[i2d];

	/* gating variables */
	r_inf = 1/(1+exp(-(vm)/11));
	tau_r = 20.3 + 138 * exp( -sqr((vm+20)/25.9) );
	s_inf = 1/(1+exp((vm+3)/7));
	tau_s = 1574 + 5268 * exp( -sqr((vm+23)/22.7) );
	
	/* I_Kv */
	EK = R*T/F * log(Ko/Ki);
	Ikv = gkv * r * s * (vm-EK);
	
	/* I_K1 */
	aK1 = 0.1/(1+exp(0.06*(vm-EK-200)));
	bK1 = ( 3*exp(0.0002*(vm-EK+100)) + exp(0.1*(vm-EK-10)) )
	    / ( 1+exp(-0.5*(vm-EK)) );
	Ik1 = gK1 * aK1/(aK1+bK1) * (vm-EK);
	
	/* I_NaK */
	Inak = INaKbar / (1+KmK/Ko) / (1+pow((KmNa/Nai),1.5)) * (vm-Vrev) / (vm-B);
	
	/* I_bNa */
	ENa =  R*T/F * log(Nao/Nai);
	Ibna = GbNa * (vm-ENa);

	/* I_ion */
	Iion = Ikv + Ik1 + Inak + Ibna; 
	
	/* differential equations */
	fv -= Cm*Iion;
	g_devF.r[i2d]  = (r_inf - r) / tau_r;
	g_devF.s[i2d]  = (s_inf - s) / tau_s;


	g_devF.vm[i2d] = fv;
}