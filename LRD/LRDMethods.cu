#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include "hip/hip_runtime.h"
#include ""


#include "../common/CudaSafeCall.h"

#include "LRDglobalVariables.cuh"
#include "typedefSparse.h"
#include "sparsePrototypes.cuh"
#include "typedefLRD.h"

#include "LRDhostPrototypes.h"
#include "LRDdevicePrototypes.cuh"

void LRD_gateinit(int memSize, size_t* pitch, gateType* gate_h, gateType* gate_dev, gateType* gate_devF) {

	hipHostAlloc((void**)&(gate_h->vm), memSize, 0);
	hipHostAlloc((void**)&(gate_h->m), memSize, 0);
	hipHostAlloc((void**)&(gate_h->h), memSize, 0);
	hipHostAlloc((void**)&(gate_h->j), memSize, 0);
	hipHostAlloc((void**)&(gate_h->d), memSize, 0);
	hipHostAlloc((void**)&(gate_h->f), memSize, 0);
	hipHostAlloc((void**)&(gate_h->b), memSize, 0);
	hipHostAlloc((void**)&(gate_h->g), memSize, 0);
	hipHostAlloc((void**)&(gate_h->xr), memSize, 0);
	hipHostAlloc((void**)&(gate_h->xs1), memSize, 0);
	hipHostAlloc((void**)&(gate_h->xs2), memSize, 0);
	hipHostAlloc((void**)&(gate_h->zdv), memSize, 0);
	hipHostAlloc((void**)&(gate_h->ydv), memSize, 0);
	hipHostAlloc((void**)&(gate_h->nai), memSize, 0);
	hipHostAlloc((void**)&(gate_h->ki), memSize, 0);
	hipHostAlloc((void**)&(gate_h->nsr), memSize, 0);
	hipHostAlloc((void**)&(gate_h->nao), memSize, 0);
	hipHostAlloc((void**)&(gate_h->ko), memSize, 0);
	hipHostAlloc((void**)&(gate_h->cao), memSize, 0);

	hipHostAlloc((void**)&(gate_h->cai), memSize, 0);
	hipHostAlloc((void**)&(gate_h->jsr), memSize, 0);
	hipHostAlloc((void**)&(gate_h->caiont), memSize, 0);
	hipHostAlloc((void**)&(gate_h->BOOL), memSize, 0);
	hipHostAlloc((void**)&(gate_h->tcicr), memSize, 0);
	hipHostAlloc((void**)&(gate_h->tjsrol), memSize, 0);
	hipHostAlloc((void**)&(gate_h->dcaiont), memSize, 0);

	// Allocate device memory arrays
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->vm, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->m, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->h, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->j, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->d, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->f, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->b, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->g, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->xr, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->xs1, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->xs2, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->zdv, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->ydv, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->nai, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->ki, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->nsr, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->nao, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->ko, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->cao, pitch,
		memSize, 1));

	CudaSafeCall(hipMallocPitch((void **)&gate_dev->cai, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->jsr, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->caiont, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->BOOL, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->tcicr, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->tjsrol, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->dcaiont, pitch,
		memSize, 1));

	// Allocate device forward memory arrays
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->vm, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->m, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->h, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->j, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->d, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->f, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->b, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->g, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->xr, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->xs1, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->xs2, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->zdv, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->ydv, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->nai, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->ki, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->nsr, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->nao, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->ko, pitch,
		memSize, 1));

	CudaSafeCall(hipMallocPitch((void **)&gate_devF->cai, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->jsr, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->caiont, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->BOOL, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->tcicr, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->tjsrol, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->dcaiont, pitch,
		memSize, 1));

	puts("\nFinished allocating device arrays\n");

	int totpoints = (int)memSize / sizeof(real);

	for (int idx = 0; idx < totpoints; idx++) {
		gate_h->vm[idx] = LRD_RestVoltage;

		gate_h->m[idx] = 0.0008;
		gate_h->h[idx] = 0.993771;
		gate_h->j[idx] = 0.995727;
		gate_h->d[idx] = 3.210618e-06;
		gate_h->f[idx] = 0.999837;
		gate_h->b[idx] = 0.000970231;
		gate_h->g[idx] = 0.994305;
		gate_h->xr[idx] = 0.000124042;
		gate_h->xs1[idx] = 0.00445683;
		gate_h->xs2[idx] = 0.00445683;
		gate_h->zdv[idx] = 0.0120892;
		gate_h->ydv[idx] = 0.999978;
		gate_h->nai[idx] = 9.0;
		gate_h->ki[idx] = 141.2;
		gate_h->nsr[idx] = 1.838;
		gate_h->nao[idx] = 140;
		gate_h->ko[idx] = 4.5;
		gate_h->cao[idx] = 1.8;

		gate_h->cai[idx] = 0.00006;
		gate_h->jsr[idx] = 1.838;
		gate_h->caiont[idx] = 0;
		gate_h->BOOL[idx] = 0;
		gate_h->tcicr[idx] = -25;
		gate_h->tjsrol[idx] = -25;
		gate_h->dcaiont[idx] = 0;
	}

	CudaSafeCall(hipMemcpy2D((void *)gate_dev->vm, *pitch, (void *)gate_h->vm,
		memSize, memSize, 1, hipMemcpyHostToDevice));

	CudaSafeCall(hipMemcpy2D((void *)gate_dev->m, *pitch, (void *)gate_h->m,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->h, *pitch, (void *)gate_h->h,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->j, *pitch, (void *)gate_h->j,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->d, *pitch, (void *)gate_h->d,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->f, *pitch, (void *)gate_h->f,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->b, *pitch, (void *)gate_h->b,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->g, *pitch, (void *)gate_h->g,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->xr, *pitch, (void *)gate_h->xr,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->xs1, *pitch, (void *)gate_h->xs1,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->xs2, *pitch, (void *)gate_h->xs2,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->zdv, *pitch, (void *)gate_h->zdv,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->ydv, *pitch, (void *)gate_h->ydv,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->nai, *pitch, (void *)gate_h->nai,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->ki, *pitch, (void *)gate_h->ki,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->nsr, *pitch, (void *)gate_h->nsr,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->nao, *pitch, (void *)gate_h->nao,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->ko, *pitch, (void *)gate_h->ko,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->cao, *pitch, (void *)gate_h->cao,
		memSize, memSize, 1, hipMemcpyHostToDevice));

	CudaSafeCall(hipMemcpy2D((void *)gate_dev->cai, *pitch, (void *)gate_h->cai,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->jsr, *pitch, (void *)gate_h->jsr,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->caiont, *pitch, (void *)gate_h->caiont,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->BOOL, *pitch, (void *)gate_h->BOOL,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->tcicr, *pitch, (void *)gate_h->tcicr,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->tjsrol, *pitch, (void *)gate_h->tjsrol,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->dcaiont, *pitch, (void *)gate_h->dcaiont,
		memSize, memSize, 1, hipMemcpyHostToDevice));


	CudaSafeCall(hipMemcpy2D((void *)gate_devF->vm, *pitch, (void *)gate_h->vm,
		memSize, memSize, 1, hipMemcpyHostToDevice));

	CudaSafeCall(hipMemcpy2D((void *)gate_devF->m, *pitch, (void *)gate_h->m,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->h, *pitch, (void *)gate_h->h,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->j, *pitch, (void *)gate_h->j,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->d, *pitch, (void *)gate_h->d,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->f, *pitch, (void *)gate_h->f,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->b, *pitch, (void *)gate_h->b,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->g, *pitch, (void *)gate_h->g,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->xr, *pitch, (void *)gate_h->xr,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->xs1, *pitch, (void *)gate_h->xs1,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->xs2, *pitch, (void *)gate_h->xs2,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->zdv, *pitch, (void *)gate_h->zdv,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->ydv, *pitch, (void *)gate_h->ydv,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->nai, *pitch, (void *)gate_h->nai,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->ki, *pitch, (void *)gate_h->ki,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->nsr, *pitch, (void *)gate_h->nsr,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->nao, *pitch, (void *)gate_h->nao,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->ko, *pitch, (void *)gate_h->ko,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->cao, *pitch, (void *)gate_h->cao,
		memSize, memSize, 1, hipMemcpyHostToDevice));

	CudaSafeCall(hipMemcpy2D((void *)gate_devF->cai, *pitch, (void *)gate_h->cai,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->jsr, *pitch, (void *)gate_h->jsr,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->caiont, *pitch, (void *)gate_h->caiont,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->BOOL, *pitch, (void *)gate_h->BOOL,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->tcicr, *pitch, (void *)gate_h->tcicr,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->tjsrol, *pitch, (void *)gate_h->tjsrol,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->dcaiont, *pitch, (void *)gate_h->dcaiont,
		memSize, memSize, 1, hipMemcpyHostToDevice));

	real** qpH = (real**)malloc(sizeof(real *)*gate_h->qpl);
	int i = 0;
	qpH[i++] = gate_devF->m;
	qpH[i++] = gate_devF->h;
	qpH[i++] = gate_devF->j;
	qpH[i++] = gate_devF->;
	qpH[i++] = gate_devF->f;
	qpH[i++] = gate_devF->b;
	qpH[i++] = gate_devF->g;
	qpH[i++] = gate_devF->xr;
	qpH[i++] = gate_devF->xs1;
	qpH[i++] = gate_devF->xs2;
	qpH[i++] = gate_devF->zdv;
	qpH[i++] = gate_devF->ydv;
	qpH[i++] = gate_devF->nai;
	qpH[i++] = gate_devF->ki;
	qpH[i++] = gate_devF->nsr;
	qpH[i++] = gate_devF->nao;
	qpH[i++] = gate_devF->ko;
	qpH[i++] = gate_devF->cao;

	CudaSafeCall(hipMemcpy((void *)gate_devF->qp, (void*)qpH, sizeof(real *)*gate_h->qpl, hipMemcpyHostToDevice));

	i = 0;
	qpH[i++] = gate_dev->m;
	qpH[i++] = gate_dev->h;
	qpH[i++] = gate_dev->j;
	qpH[i++] = gate_dev->;
	qpH[i++] = gate_dev->f;
	qpH[i++] = gate_dev->b;
	qpH[i++] = gate_dev->g;
	qpH[i++] = gate_dev->xr;
	qpH[i++] = gate_dev->xs1;
	qpH[i++] = gate_dev->xs2;
	qpH[i++] = gate_dev->zdv;
	qpH[i++] = gate_dev->ydv;
	qpH[i++] = gate_dev->nai;
	qpH[i++] = gate_dev->ki;
	qpH[i++] = gate_dev->nsr;
	qpH[i++] = gate_dev->nao;
	qpH[i++] = gate_dev->ko;
	qpH[i++] = gate_dev->cao;

	CudaSafeCall(hipMemcpy((void *)gate_dev->qp, (void*)qpH, sizeof(real *)*gate_h->qpl, hipMemcpyHostToDevice));
	

	CudaCheckError();

	puts("\nFinished initializing device arrays\n");

}

void LRD_sync(int memSize, size_t pitch, gateType* gate_h, gateType* gate_dev) {

	CudaSafeCall(hipMemcpy2D((void *)gate_h->vm, *pitch, (void *)gate_dev->vm,
		memSize, memSize, 1, hipMemcpyDeviceToHost));

	CudaSafeCall(hipMemcpy2D((void *)gate_h->m, *pitch, (void *)gate_dev->m,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->h, *pitch, (void *)gate_dev->h,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->j, *pitch, (void *)gate_dev->j,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->d, *pitch, (void *)gate_dev->d,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->f, *pitch, (void *)gate_dev->f,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->b, *pitch, (void *)gate_dev->b,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->g, *pitch, (void *)gate_dev->g,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->xr, *pitch, (void *)gate_dev->xr,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->xs1, *pitch, (void *)gate_dev->xs1,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->xs2, *pitch, (void *)gate_dev->xs2,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->zdv, *pitch, (void *)gate_dev->zdv,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->ydv, *pitch, (void *)gate_dev->ydv,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->nai, *pitch, (void *)gate_dev->nai,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->ki, *pitch, (void *)gate_dev->ki,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->nsr, *pitch, (void *)gate_dev->nsr,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->nao, *pitch, (void *)gate_dev->nao,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->ko, *pitch, (void *)gate_dev->ko,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->cao, *pitch, (void *)gate_dev->cao,
		memSize, memSize, 1, hipMemcpyDeviceToHost));

	CudaSafeCall(hipMemcpy2D((void *)gate_h->cai, *pitch, (void *)gate_dev->cai,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->jsr, *pitch, (void *)gate_dev->jsr,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->caiont, *pitch, (void *)gate_dev->caiont,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->BOOL, *pitch, (void *)gate_dev->BOOL,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->tcicr, *pitch, (void *)gate_dev->tcicr,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->tjsrol, *pitch, (void *)gate_dev->tjsrol,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->dcaiont, *pitch, (void *)gate_dev->dcaiont,
		memSize, memSize, 1, hipMemcpyDeviceToHost));
}

void LRD_exit(int memSize, size_t pitch, gateType* gate_h, gateType* gate_dev, gateType* gate_devF, sparse* MatrixINT, cudasparse* cudaMatrixINT){
	// Free gate host and device memory
	hipHostFree(gate_h->vm);

	hipHostFree(gate_h->m);
	hipHostFree(gate_h->h);
	hipHostFree(gate_h->j);
	hipHostFree(gate_h->d);
	hipHostFree(gate_h->f);
	hipHostFree(gate_h->b);
	hipHostFree(gate_h->g);
	hipHostFree(gate_h->xr);
	hipHostFree(gate_h->xs1);
	hipHostFree(gate_h->xs2);
	hipHostFree(gate_h->zdv);
	hipHostFree(gate_h->ydv);
	hipHostFree(gate_h->nai);
	hipHostFree(gate_h->ki);
	hipHostFree(gate_h->nsr);
	hipHostFree(gate_h->nao);
	hipHostFree(gate_h->ko);
	hipHostFree(gate_h->cao);

	hipHostFree(gate_h->cai);
	hipHostFree(gate_h->jsr);
	hipHostFree(gate_h->caiont);
	hipHostFree(gate_h->BOOL);
	hipHostFree(gate_h->tcicr);
	hipHostFree(gate_h->tjsrol);
	hipHostFree(gate_h->dcaiont);
	hipHostFree(gate_h->qp);


	hipFree(gate_dev->vm);

	hipFree(gate_dev->m);
	hipFree(gate_dev->h);
	hipFree(gate_dev->j);
	hipFree(gate_dev->d);
	hipFree(gate_dev->f);
	hipFree(gate_dev->b);
	hipFree(gate_dev->g);
	hipFree(gate_dev->xr);
	hipFree(gate_dev->xs1);
	hipFree(gate_dev->xs2);
	hipFree(gate_dev->zdv);
	hipFree(gate_dev->ydv);
	hipFree(gate_dev->nai);
	hipFree(gate_dev->ki);
	hipFree(gate_dev->nsr);
	hipFree(gate_dev->nao);
	hipFree(gate_dev->ko);
	hipFree(gate_dev->cao);

	hipFree(gate_dev->cai);
	hipFree(gate_dev->jsr);
	hipFree(gate_dev->caiont);
	hipFree(gate_dev->BOOL);
	hipFree(gate_dev->tcicr);
	hipFree(gate_dev->tjsrol);
	hipFree(gate_dev->dcaiont);
	hipFree(gate_dev->qp);


	hipFree(gate_devF->vm);

	hipFree(gate_devF->m);
	hipFree(gate_devF->h);
	hipFree(gate_devF->j);
	hipFree(gate_devF->d);
	hipFree(gate_devF->f);
	hipFree(gate_devF->b);
	hipFree(gate_devF->g);
	hipFree(gate_devF->xr);
	hipFree(gate_devF->xs1);
	hipFree(gate_devF->xs2);
	hipFree(gate_devF->zdv);
	hipFree(gate_devF->ydv);
	hipFree(gate_devF->nai);
	hipFree(gate_devF->ki);
	hipFree(gate_devF->nsr);
	hipFree(gate_devF->nao);
	hipFree(gate_devF->ko);
	hipFree(gate_devF->cao);

	hipFree(gate_devF->cai);
	hipFree(gate_devF->jsr);
	hipFree(gate_devF->caiont);
	hipFree(gate_devF->BOOL);
	hipFree(gate_devF->tcicr);
	hipFree(gate_devF->tjsrol);
	hipFree(gate_devF->dcaiont);	
	hipFree(gate_devF->qp);



	hipFree(cudaMatrixINT->type);
	hipFree(cudaMatrixINT->rows);
	hipFree(cudaMatrixINT->maxnz);
	hipFree(cudaMatrixINT->csep);
	hipFree(cudaMatrixINT->jcoef);
	hipFree(cudaMatrixINT->coef);
}


void __device__ GetFDev_LRD(int i2d, int pitch, real beta, real Cm, real t, real dt, int totpoints, real rx, gateType g_dev, gateType g_devF) {

	/*------------------------------------------------------------------------
	* return if outside domain
	*------------------------------------------------------------------------
	*/
	if (i2d >= totpoints) {
		return;
	}

	/* declare variables */ 	                                 
    real vm,BOOL,tcicr,tjsrol,csqn;
    real LRD_RTF;
    real LRD_Gkr, LRD_Gks,LRD_Gki,LRD_Gkatp;
    real LRD_Ena,LRD_Etca,LRD_Ekr,LRD_Eks,LRD_Eki,LRD_Ekp;
    real LRD_Ekna,LRD_Ekatp,LRD_Ekdv,LRD_Ecan,LRD_Enan;
    real m,h,j,am,bm,ah,bh,aj,bj,Ina;  
    real d,f,dss,taud,fss,tauf,Ibarca,Ibarna,Ibark;
    real fca,Ilca,Ilcana,Ilcak,Ilcatot;
    real b,g,bss,taub,gss,taug,Itca;
    real xr,r,xrss,tauxr,Ikr;
    real xs1,xs2,xs1ss,xs2ss,tauxs1,tauxs2,Iks; 
    real aki,bki,kin,Ikti;
    real kp,Ikp;
    real Inaca;
    real sigma,fnak,Inak;
    real Ipca;
    real Icab;
    real Inab;
    real pona,pov,Ikna;
    real patp,gkbaratp,Ikatp;
    real Ibarnsna,Ibarnsk,Insna,Insk;
    real rvdv,Ito;
    real azdv,bzdv,tauzdv,zdvss,zdv;
    real aydv,bydv,tauydv,ydvss,ydv;
    real naiont,kiont,caiont,Itotal;
    /*ions*/
    real nao,ko,cao;
    real dnao,dko,dcao;
 
	real nai,ki;
	real dnai,dki;

    real itr;
    real nsr,kleak,ileak,iup,dnsr;
    /* JSR CICR */
    real dcaiont,caiontold;
    real magrel,on,off,irelcicr;
    real greljsrol,ireljsrol;
    real trpn,cmdn;
    real jsr,bjsr,cjsr,djsr; 
    /* cai update here */
    real cai,catotal,bmyo,cmyo,dmyo,gpig,dcai;
    real vcell,ageo,acap,vmyo,vnsr,vjsr,vcleft;
    LRD_RTF = LRD_R*LRD_temp/LRD_frdy;


	vm        = g_dev.vm[i2d];

	m 		  = g_dev.m[i2d];
	h 		  = g_dev.h[i2d];
	j 		  = g_dev.j[i2d];
	d 		  = g_dev.d[i2d];
	f 		  = g_dev.f[i2d];
	b 		  = g_dev.b[i2d];
	g 	      = g_dev.g[i2d];
	xr 	      = g_dev.xr[i2d];
	xs1 	  = g_dev.xs1[i2d];
	xs2	      = g_dev.xs2[i2d];
	zdv       = g_dev.zdv[i2d];
	ydv	      = g_dev.ydv[i2d];
	nai 	  = g_dev.nai[i2d];
	ki 	      = g_dev.ki[i2d];
	nsr	      = g_dev.nsr[i2d];
	nao	      = g_dev.nao[i2d];
	ko 	      = g_dev.ko[i2d];
	cao       = g_dev.cao[i2d];

	cai       = g_dev.cai[i2d];
	jsr       = g_dev.jsr[i2d];
	caiont    = g_dev.caiont[i2d];
	BOOL      = g_dev.BOOL[i2d];
	tcicr     = g_dev.tcicr[i2d];
	tjsrol    = g_dev.tjsrol[i2d];
	dcaiont   = g_dev.dcaiont[i2d];

	/*------------------------------------------------------------------------
	* setting local variables
	*------------------------------------------------------------------------
	*/

	real fv = g_devF.vm[i2d];

	/* Declare varying G's and E's */
    LRD_Gkr   = 0.02614*sqrt(ko/5.4); 
    LRD_Gks   = 0.433*(1+0.6/(1+pow((0.000038/cai),1.4)));
    LRD_Gki   = 0.75*(sqrt(ko/5.4));
    LRD_Gkatp = 0.000195/nicholsarea;
    
    LRD_Ena   = (LRD_RTF)*log(nao/nai);
    LRD_Etca  = 0.5*(LRD_RTF)*log(cao/cai);  
    LRD_Ekr   = (LRD_RTF)*log(ko/ki);
    LRD_Eks   = (LRD_RTF)*log((ko+prnak*nao)/(ki+prnak*nai));
    LRD_Eki   = (LRD_RTF)*log(ko/ki);
    LRD_Ekp   = LRD_Eki;
    LRD_Ekna  = LRD_Ekr; 
    LRD_Ekatp = LRD_Ekr; 
    LRD_Ekdv  = LRD_Ekr;
    LRD_Ecan  = LRD_Etca;
    LRD_Enan  = LRD_Ena;

	/* Na current [15] */
    am = 0.32*(vm+47.13)/(1-exp(-0.1*(vm+47.13)));
    bm = 0.08*exp(-vm/11);
    if (vm < -40) {
		ah = 0.135*exp((80+vm)/-6.8);
		bh = 3.56*exp(0.079*vm)+310000*exp(0.35*vm);
		aj = (-127140*exp(0.2444*vm)-0.00003474*exp(-0.04391*vm))*((vm+37.78)/(1+exp(0.311*(vm+79.23))));
		bj = (0.1212*exp(-0.01052*vm))/(1+exp(-0.1378*(vm+40.14)));
     } else {
		ah = 0;
		bh = 1/(0.13*(1+exp((vm+10.66)/-11.1)));
		aj = 0;
		bj = (0.3*exp(-0.0000002535*vm))/(1+exp(-0.1*(vm+32)));
    }        

      Ina=LRD_Gna*(m*m*m*h*j)*(vm-LRD_Ena);

	/* L-type Calcium current [14,15] */
	dss  = 1/(1+exp(-(vm+10)/6.24));
	taud = dss*(1-exp(-(vm+10)/6.24))/(0.035*(vm+10));
	fss  = (1/(1+exp((vm+32)/8)))+(0.6/(1+exp((50-vm)/20)));
	tauf = 1/(0.0197*exp(-0.0337*0.0337*(vm+10)*(vm+10))+0.02);

	Ibarca = pca*zca*zca*((vm*LRD_frdy)/(LRD_RTF))*((gacai*cai*exp((zca*vm)/(LRD_RTF))-gacao*cao)/(exp((zca*vm)/(LRD_RTF))-1));
	Ibarna = pna*zna*zna*((vm*LRD_frdy)/(LRD_RTF))*((ganai*nai*exp((zna*vm)/(LRD_RTF))-ganao*nao)/(exp((zna*vm)/(LRD_RTF))-1));
	Ibark  = pk*zk*zk*((vm*LRD_frdy)/(LRD_RTF))*((gaki*ki*exp((zk*vm)/(LRD_RTF))-gako*ko)/(exp((zk*vm)/(LRD_RTF))-1));

	fca = 1/(1+cai/kmca);

	Ilca    = d*f*fca*Ibarca;
	Ilcana  = d*f*fca*Ibarna;
	Ilcak   = d*f*fca*Ibark;	
	Ilcatot = Ilca+Ilcana+Ilcak;

	/* T-type Calcium current [13] */
	bss  = 1/(1+exp(-(vm+14)/10.8));
	taub = 3.7+6.1/(1+exp((vm+25)/4.5));
	gss  = 1/(1+exp((vm+60)/5.6));
	if (vm<=0) {
		taug = -0.875*vm+12;
	} else {
		taug = 12;
	}
	Itca = LRD_Gtca*b*b*g*(vm-LRD_Etca);

	/* K current - Rapid [13] */
	xrss  = 1/(1+exp(-(vm+21.5)/7.5));
	tauxr = 1/(0.00138*(vm+14.2)/(1-exp(-0.123*(vm+14.2)))+0.00061*(vm+38.9)/(exp(0.145*(vm+38.9))-1));

	r = 1/(1+exp((vm+9)/22.4));

	Ikr = LRD_Gkr*xr*r*(vm-LRD_Ekr);

	/* K current - Slow [10,13] */
	xs1ss  = 1/(1+exp(-(vm-1.5)/16.7));
	xs2ss  = xs1ss;
	tauxs1 = 1/(0.0000719*(vm+30)/(1-exp(-0.148*(vm+30)))+0.000131*(vm+30)/(exp(0.0687*(vm+30))-1));
	tauxs2 = 4*tauxs1;

	Iks = LRD_Gks*xs1*xs2*(vm-LRD_Eks);

	/* K current -  Time independent [15] */
	aki = 1.02/(1+exp(0.2385*(vm-LRD_Eki-59.215)));
	bki = (0.49124*exp(0.08032*(vm-LRD_Eki+5.476))+exp(0.06175*(vm-LRD_Eki-594.31)))/(1+exp(-0.5143*(vm-LRD_Eki+4.753)));	

	kin = aki/(aki+bki);

	Ikti = LRD_Gki*kin*(vm-LRD_Eki);

	/* K current - Plateau [15] */
	kp  = 1/(1+exp((7.488-vm)/5.98));	

	Ikp = LRD_Gkp*kp*(vm-LRD_Ekp);
      
	/* Na-Ca exchanger [6,14,15] */
	Inaca = c1*exp((gammas-1)*vm/(LRD_RTF))*((exp(vm/(LRD_RTF))*nai*nai*nai*cao-nao*nao*nao*cai)/(1+c2*exp((gammas-1)*vm/(LRD_RTF))*(exp(vm/(LRD_RTF))*nai*nai*nai*cao+nao*nao*nao*cai)));

	/* Na-K pump [15] */
	sigma = (exp(nao/67.3)-1)/7;
	fnak  = 1/(1+0.1245*exp((-0.1*vm)/(LRD_RTF))+0.0365*sigma*exp((-vm)/(LRD_RTF)));

	Inak = Ibarnak*fnak*(1/(1+kmnai*kmnai/(nai*nai)))*(ko/(ko+kmko));

	/* Sarcolemmal Ca pump [15] */
	Ipca = (Ibarpca*cai)/(kmpca+cai);

	/* Ca background current [15] */
	Icab = LRD_Gcab*(vm-LRD_Ecan);

	/* Na background current [15] */
	Inab = LRD_Gnab*(vm-LRD_Enan);          
      
	/* Na activated K current [6] */
	pona = 0.85/(1+pow((kdkna/nai),2.8));
	pov  = 0.8-(0.65/(1+exp((vm+125)/15)));

	Ikna = LRD_ikna*LRD_Gkna*pona*pov*(vm-LRD_Ekna);

	/* ATP sensitive K current [11] */
	patp = 1/(1+(pow((atpi/katp),hatp)));
	gkbaratp = LRD_Gkatp*patp*(pow((ko/4),natp));

	Ikatp = LRD_ikatp*gkbaratp*(vm-LRD_Ekatp);     

	/* Non-specific Ca-activated current [14,15] */
	Ibarnsna = pnsca*zna*zna*((vm*LRD_frdy)/(LRD_RTF))*((ganai*nai*exp((zna*vm)/(LRD_RTF))-ganao*nao)/(exp((zna*vm)/(LRD_RTF))-1));
	Ibarnsk  = pnsca*zk*zk*((vm*LRD_frdy)/(LRD_RTF))*((gaki*ki*exp((zk*vm)/(LRD_RTF))-gako*ko)/(exp((zk*vm)/(LRD_RTF))-1));

	Insna = LRD_insna*Ibarnsna/(1+kmnsca*kmnsca*kmnsca/(cai*cai*cai)); 
	Insk = LRD_insk*Ibarnsk/(1+kmnsca*kmnsca*kmnsca/(cai*cai*cai));   

	/* Transient outward current */
	rvdv = exp(vm/100);

	azdv = (10*exp((vm-40)/25))/(1+exp((vm-40)/25));
	bzdv = (10*exp(-(vm+90)/25))/(1+exp(-(vm+90)/25));
	tauzdv = 1/(azdv+bzdv);
	zdvss = azdv/(azdv+bzdv);

	aydv = 0.015/(1+exp((vm+60)/5));
	bydv = (0.1*exp((vm+25)/5))/(1+exp((vm+25)/5));
	tauydv = 1/(aydv+bydv);
	ydvss = aydv/(aydv+bydv);

	Ito = LRD_ito*LRD_Gitodv*zdv*zdv*zdv*ydv*rvdv*(vm-LRD_Ekdv);

	/* Summing currents (inactive currents are set to zero with activation variables) */
	naiont = Ina+Inab+Ilcana+3*Inak+3*Inaca+Insna;
	kiont  = Ikr+Iks+Ikti+Ikp+Ilcak+-2*Inak+Insk+Ito+Ikna+Ikatp;  
	caiont = Ilca+Icab+Ipca-2*Inaca+Itca;

	Itotal = LRD_cm*(naiont+kiont+caiont);     /* uA/cm2 */
 
	if (((t-tcicr)>80) && (vm<-30)) {
		BOOL = 0;  
		g_dev.BOOL[i2d] = BOOL;                                                                  
	}

	/* Put voltage update here  */
	fv += -Itotal;
	g_devF.vm[i2d] = fv;


	/* change in cleft concentration */
	dnao = LRD_cleft*((nabm-nao)/taudiff+naiont*acap*LRD_cm/(vcleft*LRD_frdy));
	dko = LRD_cleft*((kbm-ko)/taudiff+kiont*acap*LRD_cm/(vcleft*LRD_frdy));
	dcao = LRD_cleft*((cabm-cao)/taudiff+caiont*acap*LRD_cm/(vcleft*LRD_frdy*2));

	/* change in nai and ki concentration */
	dnai = -LRD_cm*(naiont*acap)/(vmyo*zna*LRD_frdy);     /* dnai/dt */
	dki = -LRD_cm*(kiont*acap)/(vmyo*zk*LRD_frdy);        /* dki/dt */

	/* change in itr [14] */ 
	itr = (nsr-jsr)/tautr;                           

	/* change in nsr [14] */
	kleak = iupbar/nsrbar;
	ileak = kleak*nsr;
	iup   = iupbar*cai/(cai+kmup);

	dnsr = (iup-ileak-itr*vjsr/vnsr);          /* dnsr/dt */

	/* Calcium-induced-calcium-release (CICR) criteia [6] */
	if ((vm>-35) && (((caiont-caiontold)/dt)<dcaiont) && (BOOL==0)){
		BOOL  = 1;
		tcicr = t;
		g_dev.BOOL[i2d] = BOOL;
		g_dev.tcicr[i2d] = tcicr;   /* changes reference time */
	}   

	on  = 1/(1+exp((-(t-tcicr)+4)/.5));
	off = 1-on;
	magrel = 1/(1+exp(((Ilca+Icab+Ipca-2*Inaca+Itca)+5)/0.9));

	irelcicr = gmaxrel*on*off*magrel*(jsr-cai);

	/* JSR Calciium overload [13] */
	greljsrol = grelbarjsrol*(1-exp(-(t-tjsrol)/tauon))*exp(-(t-tjsrol)/tauoff);
	ireljsrol = greljsrol*(jsr-cai);

	csqn = csqnbar*(jsr/(jsr+kmcsqn));

	djsr = dt*(itr-irelcicr-ireljsrol);

	bjsr = csqnbar-csqn-djsr-jsr+kmcsqn;
	cjsr = kmcsqn*(csqn+djsr+jsr);

	jsr =(sqrt(bjsr*bjsr+4*cjsr)-bjsr)/2; 

	/* Calcium buffers in myoplasm [15] */
	trpn = trpnbar*(cai/(cai+kmtrpn));
	cmdn = cmdnbar*(cai/(cai+kmcmdn));
     
	/* change in cai concentration [13] */
	dcai = -dt*(((LRD_cm*caiont*acap)/(vmyo*zca*LRD_frdy))+((iup-ileak)*vnsr/vmyo)-(irelcicr*vjsr/vmyo)-(ireljsrol*vjsr/vmyo));

	catotal = trpn+cmdn+dcai+cai;

	bmyo = cmdnbar+trpnbar-catotal+kmtrpn+kmcmdn;
	cmyo = (kmcmdn*kmtrpn)-(catotal*(kmtrpn+kmcmdn))+(trpnbar*kmcmdn)+(cmdnbar*kmtrpn);
	dmyo = -kmtrpn*kmcmdn*catotal;
	gpig = sqrt(bmyo*bmyo-3*cmyo);

	cai = ((2*gpig/3)*cos(acos((9*bmyo*cmyo-2*bmyo*bmyo*bmyo-27*dmyo)/(2*pow((bmyo*bmyo-3*cmyo),1.5)))/3)-(bmyo/3)); 

	/* Calcium overload criteria [15] */
	if((csqn>=csqnth) && ((t-tjsrol)>50)){
		printf("Spontaneous Release occured at time %lf at node %i\n",t,i);
		tjsrol = t;  	       
		g_dev.tjsrol[i2d] = tjsrol;                /* changes reference time */
	}

    g_devF.m[i2d]    = am*(1.0-m) - bm*m;
    g_devF.h[i2d]    = ah*(1.0-h) - bh*h;
    g_devF.j[i2d]    = aj*(1.0-j) - bj*j;
    g_devF.d[i2d]    = (dss/taud)*(1-d)-(1-dss)*(d/taud);
    g_devF.f[i2d]    = (fss/tauf)*(1-f)-(1-fss)*(f/tauf);
    g_devF.b[i2d]    = (bss/taub)*(1-b)-(1-bss)*(b/taub);
    g_devF.g[i2d]    = (gss/taug)*(1-g)-(1-gss)*(g/taug);
    g_devF.xr[i2d]   = (xrss/tauxr)*(1-xr)-(1-xrss)*(xr/tauxr);
    g_devF.xs1[i2d]  = (xs1ss/tauxs1)*(1-xs1)-(1-xs1ss)*(xs1/tauxs1);
    g_devF.xs2[i2d]  = (xs2ss/tauxs2)*(1-xs2)-(1-xs2ss)*(xs2/tauxs2);
    g_devF.zdv[i2d]  = (zdvss/tauzdv)*(1-zdv)-(1-zdvss)*(zdv/tauzdv);
    g_devF.ydv[i2d]  = (ydvss/tauydv)*(1-ydv)-(1-ydvss)*(ydv/tauydv);
    g_devF.nai[i2d]  = dnai;
    g_devF.ki[i2d]   = dki;
    g_devF.nsr[i2d]  = dnsr;
    g_devF.nao[i2d]  = dnao; 
    g_devF.ko[i2d]   = dko; 
    g_devF.cao[i2d]  = dcao;


  /* assign Temp variables to memory */    
    g_devF.caiont[i2d]  = caiont;
    g_devF.cai[i2d]     = cai; 
    g_devF.jsr[i2d]     = jsr;
    g_devF.dcaiont[i2d] = (caiont-caiontold)/dt;

}