#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stddef.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include "hip/hip_runtime.h"
#include ""

typedef double real;

#include "typedefSparse.h"
#include "sparsePrototypes.cuh"
#include "parseInput.cuh"
#include "VectorOps.cuh"
#include "mainwrapper.cuh"

#include "inputs.cuh"

#include "./Output/outputDump.c"


int memSize, totpoints;

size_t pitch;
int count;

// Physical parameters
real Lx, Ly, Lz, hx, hy, hz, CFL_max, t, tlim;
real dt, diff, rx, ry, rz;

// Voltage and gate arrays
gateType gate_h, gate_dev, gate_devF;

int i, tp, r, c, bwp, bwm, mnz;
real TissueLocalSize;
sparse MatrixINT = _INIT_SPARSE;
cudasparse cudaMatrixINT;

/////// MAIN ///////
static char* int_file = NULL;
static char* out_file = NULL;
real Tspacing, Tnext, beta, Cm;

//Stim
static int numStim = 0;
static int nodes[MAX_STIM];
static real start[MAX_STIM];
static real dur[MAX_STIM];
static real strength[MAX_STIM];

stimulus stim;

int main(int argc, char **argv) {
	int err = AppendResources(inpBuffer, argc, argv);

	int cmd, k;
	int* vi;
	real* vr;
	for (int i = 0; inpBuffer[i] != NULL; i++) {
		cmd = FindCommand(cases, inpBuffer[i]);
		switch (cmd) {
			case 4001:
				int_file = GetStringValue(inpBuffer[i]);
				break;
			case 2002:
				tlim = GetRealValue(inpBuffer[i]);
				break;
			case 2001:
				dt = GetRealValue(inpBuffer[i]);
				break;
			case 3001:
				numStim = GetIntValue(inpBuffer[i]);
				break;
			case 3002:
				k = FindNum(inpBuffer[i]);
				vi = GetIntArray(inpBuffer[i]);
				for(int j = 0; j < numStim; j++) nodes[j+k] = vi[j];
				break;
			case 3003:
				k = FindNum(inpBuffer[i]);
				vr = GetRealArray(inpBuffer[i]);
				for (int j = 0; j < numStim; j++) strength[j + k] = vr[j];
				break;
			case 3004:
				k = FindNum(inpBuffer[i]);
				vr = GetRealArray(inpBuffer[i]);
				for (int j = 0; j < numStim; j++) start[j + k] = vr[j];
				break;
			case 3005:
				k = FindNum(inpBuffer[i]);
				vr = GetRealArray(inpBuffer[i]);
				for (int j = 0; j < numStim; j++) dur[j + k] = vr[j];
				break;
			case 6003:
				out_file = GetStringValue(inpBuffer[i]);
				break;
			case 6001:
				Tspacing = GetRealValue(inpBuffer[i]);
				break;
			case 9010:
				beta = GetRealValue(inpBuffer[i]);
				break;
			case 9011:
				Cm = GetRealValue(inpBuffer[i]);
				break;
		}
	}

	sprreadinfo(int_file, &tp, &r, &c, &bwp, &bwm, &mnz);
	TissueLocalSize = r;
	sprread(int_file, &MatrixINT);
	cudasprinit(&cudaMatrixINT, &MatrixINT);
	
	// Memory size declaration
	memSize = TissueLocalSize*sizeof(real);
	totpoints = (int)TissueLocalSize;
	CFL_max = 0.2; // Courant stability condition
	
	// Physical length (cm)
	Lx = 12.0;
	// Physical spacing between nodes (cm)
	if(MatrixINT.maxnz <= 5) hx = Lx/(sqrt(totpoints)-1.0);
	else hx = Lx / (cbrt(totpoints) - 1.0);
	//dt = (1.0/diff)*(CFL_max-0.1)*(hx*hx*hy*hy)/(hx*hx+hy*hy);
	diff = 0.001;
	rx = dt*diff/(hx*hx);
	t = 0.0;
	Tnext = t;

	printf("\n********Grid dimensions*********\n");
	printf("Total number of nodes: %d\n", totpoints);
	printf("\n********Spatial dimensions*********\n");
	printf("Physical dx %f cm \n", hx);
	printf("\n********Diffusion*********\n");
	printf("Diffusion parallel component: %f cm^2/ms\n", diff);
	printf("\n********Time*********\n");
	printf("Time step: %f ms\n", dt);

	// Array allocation and init

	//CRN_gateinit(memSize, &pitch, &gate_h, &gate_dev, &gate_devF);
	fourV_gateinit(memSize, &pitch, &gate_h, &gate_dev, &gate_devF);

	setupStim(&stim, nodes, start, dur, strength, numStim);
	puts("\nStarting simulation\n");

	static FILE *fp1;
	fp1 = fopen(out_file, "wb+");
	// main loop
	outputDump(totpoints, &gate_h, fp1);
	Tnext += Tspacing;

	while (t < tlim) {

		main_wrapper(pitch, beta, Cm, t, dt, totpoints, rx, gate_dev, gate_devF, stim, cudaMatrixINT);
		t += dt;

		if (t >= Tnext) {

			//CRN_sync(memSize, pitch, &gate_h, &gate_dev);
			fourV_sync(memSize, pitch, &gate_h, &gate_dev);

			outputDump(totpoints, &gate_h, fp1);
			Tnext += Tspacing;
		}
	}

	if (t >= Tnext) {
		outputDump(totpoints, &gate_h, fp1);
		Tnext += Tspacing;
	}
	
	fclose(fp1);
	puts("\nSimulation ended\n");

	//CRN_exit(memSize, pitch, &gate_h, &gate_dev, &gate_devF, &MatrixINT, &cudaMatrixINT);
	fourV_exit(memSize, pitch, &gate_h, &gate_dev, &gate_devF, &MatrixINT, &cudaMatrixINT);

	return 0;
}