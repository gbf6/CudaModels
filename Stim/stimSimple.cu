#include "hip/hip_runtime.h"
#include "typedefstimSimple.h"
#include "stimSimple.cuh"

void __global__ stimSimpleDev(stimulus stim, real t, gateType g_devF) {
	for (int i = 0; i < stim.numStim[0]; i++) {
		if (t >= stim.start[i] && t <= stim.finish[i]) g_devF.vm[stim.node[i]] += stim.strength[i];
	}
}

void setupStim(stimulus* stim, int* nodes, real* start, real* dur, real* strength, int numStim) {
	CudaSafeCall(hipMalloc((void **)&stim->start, sizeof(real)*numStim));
	CudaSafeCall(hipMalloc((void **)&stim->finish, sizeof(real)*numStim));
	CudaSafeCall(hipMalloc((void **)&stim->strength, sizeof(real)*numStim));
	CudaSafeCall(hipMalloc((void **)&stim->node, sizeof(int)*numStim));
	CudaSafeCall(hipMalloc((void **)&stim->numStim, sizeof(int)));
	

	for (int i = 0; i < numStim; i++) {
		dur[i] = dur[i] + start[i];
	}

	CudaSafeCall(hipMemcpy((void*)stim->start, start, sizeof(real)*numStim, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy((void*)stim->finish, dur, sizeof(real)*numStim, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy((void*)stim->strength, strength, sizeof(real)*numStim, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy((void*)stim->node, nodes, sizeof(int)*numStim, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy((void*)stim->numStim, &numStim, sizeof(int), hipMemcpyHostToDevice));
}