#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include "hip/hip_runtime.h"
#include ""

#include "../common/CudaSafeCall.h"

#include "CRNglobalVariables.cuh"
#include "typedefSparse.h"
#include "sparsePrototypes.cuh"
#include "typedefCRN.h"

#include "CRNhostPrototypes.h"
#include "CRNdevicePrototypes.cuh"

void __device__ stimSimple(real t, int i2d, gateType* g_dev, real* fv);

void CRN_gateinit(int memSize, size_t* pitch, gateType* gate_h, gateType* gate_dev) {

	hipHostAlloc((void**)&(gate_h->vm), memSize, 0);
	hipHostAlloc((void**)&(gate_h->vmp), memSize, 0);
	hipHostAlloc((void**)&(gate_h->fv), memSize, 0);
	hipHostAlloc((void**)&(gate_h->m), memSize, 0);
	hipHostAlloc((void**)&(gate_h->h), memSize, 0);
	hipHostAlloc((void**)&(gate_h->j), memSize, 0);
	hipHostAlloc((void**)&(gate_h->sa), memSize, 0);
	hipHostAlloc((void**)&(gate_h->si), memSize, 0);
	hipHostAlloc((void**)&(gate_h->ua), memSize, 0);
	hipHostAlloc((void**)&(gate_h->ui), memSize, 0);
	hipHostAlloc((void**)&(gate_h->xr), memSize, 0);
	hipHostAlloc((void**)&(gate_h->xs), memSize, 0);
	hipHostAlloc((void**)&(gate_h->d), memSize, 0);
	hipHostAlloc((void**)&(gate_h->f), memSize, 0);
	hipHostAlloc((void**)&(gate_h->fca), memSize, 0);
	hipHostAlloc((void**)&(gate_h->u), memSize, 0);
	hipHostAlloc((void**)&(gate_h->v), memSize, 0);
	hipHostAlloc((void**)&(gate_h->w), memSize, 0);
	hipHostAlloc((void**)&(gate_h->Nai), memSize, 0);
	hipHostAlloc((void**)&(gate_h->Ki), memSize, 0);
	hipHostAlloc((void**)&(gate_h->Cai), memSize, 0);
	hipHostAlloc((void**)&(gate_h->Caup), memSize, 0);
	hipHostAlloc((void**)&(gate_h->Carel), memSize, 0);
	hipHostAlloc((void**)&(gate_h->start), memSize, 0);
	hipHostAlloc((void**)&(gate_h->finish), memSize, 0);
	hipHostAlloc((void**)&(gate_h->strength), memSize, 0);



	// Allocate device memory arrays
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->vm, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->vmp, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->fv, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->m, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->h, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->j, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->sa, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->si, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->ua, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->ui, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->xr, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->xs, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->d, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->f, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->fca, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->u, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->v, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->w, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->Nai, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->Ki, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->Cai, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->Caup, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->Carel, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->start, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->finish, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->strength, pitch,
		memSize, 1));

	puts("\nFinished allocating device arrays\n");

	int totpoints = (int)memSize / sizeof(real);

	for (int idx = 0; idx < totpoints; idx++) {
		gate_h->vm[idx]       = -81.2;
		gate_h->vmp[idx]	  = -81.2;
		gate_h->fv[idx]       = 0;
		gate_h->m[idx]        = 2.91e-3;
		gate_h->h[idx]        = 0.965;
		gate_h->j[idx]        = 0.978;
		gate_h->sa[idx]       = 3.04e-2;
		gate_h->si[idx]       = 0.999;
		gate_h->ua[idx]       = 4.96e-3;
		gate_h->ui[idx]       = 0.999;
		gate_h->xr[idx]       = 3.29e-5;
		gate_h->xs[idx]       = 1.87e-2;
		gate_h->d[idx]        = 1.37e-4;
		gate_h->f[idx]        = 0.999;
		gate_h->fca[idx]      = 0.775;
		gate_h->u[idx]        = 0.0;
		gate_h->v[idx]        = 1.0;
		gate_h->w[idx]        = 0.999;
		gate_h->Nai[idx]      = 11.2;
		gate_h->Ki[idx]       = 139.0;
		gate_h->Cai[idx]      = 1.02e-4;
		gate_h->Caup[idx]     = 1.49;
		gate_h->Carel[idx]    = 1.49;
		gate_h->start[idx]    = -1;
		gate_h->finish[idx]   = -1;
		gate_h->strength[idx] = 0;
	}

	CudaSafeCall(hipMemcpy2D((void *)gate_dev->vm, *pitch, (void *)gate_h->vm,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->vmp, *pitch, (void *)gate_h->vmp,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->fv, *pitch, (void *)gate_h->fv,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->m, *pitch, (void *)gate_h->m,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->h, *pitch, (void *)gate_h->h,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->j, *pitch, (void *)gate_h->j,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->sa, *pitch, (void *)gate_h->sa,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->si, *pitch, (void *)gate_h->si,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->ua, *pitch, (void *)gate_h->ua,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->ui, *pitch, (void *)gate_h->ui,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->xr, *pitch, (void *)gate_h->xr,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->xs, *pitch, (void *)gate_h->xs,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->d, *pitch, (void *)gate_h->d,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->f, *pitch, (void *)gate_h->f,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->fca, *pitch, (void *)gate_h->fca,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->u, *pitch, (void *)gate_h->u,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->v, *pitch, (void *)gate_h->v,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->w, *pitch, (void *)gate_h->w,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->Nai, *pitch, (void *)gate_h->Nai,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->Ki, *pitch, (void *)gate_h->Ki,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->Cai, *pitch, (void *)gate_h->Cai,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->Caup, *pitch, (void *)gate_h->Caup,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->Carel, *pitch, (void *)gate_h->Carel,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	
	CudaCheckError();

	puts("\nFinished initializing device arrays\n");

}

void CRN_sync(int memSize, size_t pitch, gateType* gate_h, gateType* gate_dev) {
	CudaSafeCall(hipMemcpy2D((void *)gate_h->vm, memSize, (void *)gate_dev->vm,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->vmp, memSize, (void *)gate_dev->vmp,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->fv, memSize, (void *)gate_dev->fv,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->m, memSize, (void *)gate_dev->m,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->h, memSize, (void *)gate_dev->h,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->j, memSize, (void *)gate_dev->j,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->sa, memSize, (void *)gate_dev->sa,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->si, memSize, (void *)gate_dev->si,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->ua, memSize, (void *)gate_dev->ua,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->ui, memSize, (void *)gate_dev->ui,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->xr, memSize, (void *)gate_dev->xr,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->xs, memSize, (void *)gate_dev->xs,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->d, memSize, (void *)gate_dev->d,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->f, memSize, (void *)gate_dev->f,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->fca, memSize, (void *)gate_dev->fca,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->u, memSize, (void *)gate_dev->u,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->v, memSize, (void *)gate_dev->v,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->w, memSize, (void *)gate_dev->w,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->Nai, memSize, (void *)gate_dev->Nai,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->Ki, memSize, (void *)gate_dev->Ki,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->Cai, memSize, (void *)gate_dev->Cai,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->Caup, memSize, (void *)gate_dev->Caup,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->Carel, memSize, (void *)gate_dev->Carel,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
}

void CRN_exit(int memSize, size_t pitch, gateType* gate_h, gateType* gate_dev, sparse* MatrixINT, cudasparse* cudaMatrixINT) {
	CRN_sync(memSize, pitch, gate_h, gate_dev);

	// Free gate host and device memory
	hipHostFree(gate_h->vm); hipHostFree(gate_h->m); hipHostFree(gate_h->j); hipHostFree(gate_h->sa);
	hipHostFree(gate_h->si); hipHostFree(gate_h->ua); hipHostFree(gate_h->ui); hipHostFree(gate_h->xr);
	hipHostFree(gate_h->xs); hipHostFree(gate_h->d); hipHostFree(gate_h->f); hipHostFree(gate_h->fca);
	hipHostFree(gate_h->u); hipHostFree(gate_h->v); hipHostFree(gate_h->w); hipHostFree(gate_h->Nai);
	hipHostFree(gate_h->Ki); hipHostFree(gate_h->Cai); hipHostFree(gate_h->Caup); hipHostFree(gate_h->Carel);
	hipHostFree(gate_h->start); hipHostFree(gate_h->finish); hipHostFree(gate_h->strength);
	hipHostFree(gate_h->vmp); hipHostFree(gate_h->fv);

	hipFree(gate_dev->vm); hipFree(gate_dev->m); hipFree(gate_dev->j); hipFree(gate_dev->sa);
	hipFree(gate_dev->si); hipFree(gate_dev->ua); hipFree(gate_dev->ui); hipFree(gate_dev->xr);
	hipFree(gate_dev->xs); hipFree(gate_dev->d); hipFree(gate_dev->f); hipFree(gate_dev->fca);
	hipFree(gate_dev->u); hipFree(gate_dev->v); hipFree(gate_dev->w); hipFree(gate_dev->Nai);
	hipFree(gate_dev->Ki); hipFree(gate_dev->Cai); hipFree(gate_dev->Caup); hipFree(gate_h->Carel);
	hipFree(gate_dev->start); hipFree(gate_dev->finish); hipFree(gate_dev->strength);
	hipFree(gate_dev->vmp); hipFree(gate_dev->fv);
	hipFree(cudaMatrixINT->type);
	hipFree(cudaMatrixINT->rows);
	hipFree(cudaMatrixINT->maxnz);
	hipFree(cudaMatrixINT->csep);
	hipFree(cudaMatrixINT->jcoef);
	hipFree(cudaMatrixINT->coef);

}

 void __global__ CRN_Kernel(int pitch, real beta, real Cm, real t, real dt, int totpoints, real rx, gateType g_dev, cudasparse cudaMatrixINT) {

	 const int i2d = blockIdx.x*BLOCK_DIM_1D + threadIdx.x;


	/*------------------------------------------------------------------------
	* return if outside domain
	*------------------------------------------------------------------------
	*/
	if (i2d >= totpoints) {
		//printf("Aborting");
		return;
	}

	/*------------------------------------------------------------------------
	* setting local variables
	*------------------------------------------------------------------------
	*/

	real vm    = g_dev.vm[i2d];
	real vmp   = g_dev.vmp[i2d];
	real fv	   = g_dev.fv[i2d];
	real m     = g_dev.m[i2d];
	real h     = g_dev.h[i2d];
	real j     = g_dev.j[i2d];
	real sa    = g_dev.sa[i2d];
	real si    = g_dev.si[i2d];
	real ua    = g_dev.ua[i2d];
	real ui    = g_dev.ui[i2d];
	real xr    = g_dev.xr[i2d];
	real xs    = g_dev.xs[i2d];
	real d     = g_dev.d[i2d];
	real f     = g_dev.f[i2d];
	real fca   = g_dev.fca[i2d];
	real u     = g_dev.u[i2d];
	real v     = g_dev.v[i2d];
	real w     = g_dev.w[i2d];
	real Nai   = g_dev.Nai[i2d];
	real Ki    = g_dev.Ki[i2d];
	real Cai   = g_dev.Cai[i2d];
	real Caup  = g_dev.Caup[i2d];
	real Carel = g_dev.Carel[i2d];

	/*------------------------------------------------------------------------
	* Laplacian Calculation
	*
	* No flux boundary condition is applied on all boundaries through
	* the Laplacian operator definition
	*------------------------------------------------------------------------
	*/
	//int printv = 10200;

	fv = 0;
	//sprDiff(&fv, cudaMatrixINT, i2d, totpoints, g_dev.vmp, beta);
	for (int i = 0; i < *cudaMatrixINT.maxnz; i++) {
		fv += (1.0/beta)*cudaMatrixINT.coef[i2d + i * totpoints] * g_dev.vmp[cudaMatrixINT.jcoef[i2d + i * totpoints] - 1];
		//if (i2d == printv) printf("%d: %.31f, %.31f, %.31f, %.31f\n", cudaMatrixINT.jcoef[i2d + i * totpoints], cudaMatrixINT.coef[i2d + i * totpoints], g_dev.vmp[cudaMatrixINT.jcoef[i2d + i * totpoints] - 1], (1.0 / beta)*cudaMatrixINT.coef[i2d + i * totpoints] * g_dev.vmp[cudaMatrixINT.jcoef[i2d + i * totpoints] - 1], fv);
	}

	//if (i2d == printv) printf("t: %.31f\ndiff: %.31f\n", t, fv);

	/*------------------------------------------------------------------------
	* Currents
	*------------------------------------------------------------------------
	*/

	real Ena = (R*T / F)*log(Nao / Nai);
	real Ek = (R*T / F)*log(Ko / Ki);
	real Eca = (R*T / (2.0*F))*log(Cao / Cai);
	real gkur = 1.0e-4*(0.005 + 0.05 / (1.0 + exp(-1.0*(vm - 15.0) / 13.0)));
	real sigma = (1.0 / 7.0)*(exp(Nao / 67.3) - 1.0);
	real fnak = 1.0 / (1.0 + 0.1245*exp(-0.1*F*vm / (R*T)) + 0.0365*sigma*exp(-1.0*F*vm / (R*T)));

	real Ina = gna * m*m*m*h*j*(vm - Ena);
	real Ik1 = gk1 * (vm - Ek) / (1.0 + exp(0.07*(vm + 80.0)));
	real Ito = gto * sa*sa*sa*si*(vm - Ek);
	real Ikur = gkur * ua*ua*ua*ui*(vm - Ek);
	real Ikr = gkr * xr*(vm - Ek) / (1.0 + exp((vm + 15.0) / 22.4));
	real Iks = gks * xs*xs*(vm - Ek);
	real Ical = gcal * d*f*fca*(vm - 65.0);
	real Inak = Inakmax * fnak*(1.0 / (1.0 + pow(Kmnai / Nai, 1.5))) * Ko / (Ko + Kmko);
	real Inaca = Inacamax * (exp(lambda*F*vm / (R*T))*Nai*Nai*Nai*Cao -
		exp((lambda - 1)*F*vm / (R*T))*Nao*Nao*Nao*Cai) /
		((Kmna*Kmna*Kmna + Nao * Nao*Nao)*(Kmca + Cao)*(1.0 + ksat * exp((lambda - 1)*F*vm / (R*T))));
	real Ibca = gbca * (vm - Eca);
	real Ibna = gbna * (vm - Ena);
	real Ipca = Ipcamax * Cai / (0.0005 + Cai);

	real Irel = krel * u*u*v*w*(Carel - Cai);
	real Itr = (Caup - Carel) / tautr;
	real Iup = Iupmax / (1.0 + (kup / Cai));
	real Iupleak = Caup * Iupmax / Caupmax;

	real Iion = Ina + Ik1 + Ito + Ikur + Ikr + Iks + Ical + Ipca + Inak + Inaca + Ibna + Ibca;
	Iion /= (pi*cellDiameter*cellLength*1.0e-8);
	
	/*------------------------------------------------------------------------
	* Gating variables
	*------------------------------------------------------------------------
	*/

	real am;
	if (vm == -47.13) {
		am = 3.2;
	}
	else {
		am = 0.32*(vm + 47.13) / (1.0 - exp(-0.1*(vm + 47.13)));
	}
	real bm = 0.08*exp(-vm / 11.0);
	real taum = 1.0 / (am + bm);
	real infm = am * taum;

	real ah, bh, aj, bj;
	if (vm >= -40.0) {
		ah = 0.0;
		bh = 1.0 / (0.13*(1.0 + exp(-1.0*(vm + 10.66) / 11.1)));
		aj = 0.0;
		bj = 0.3*exp((-2.535e-7)*vm) / (1.0 + exp(-0.1*(vm + 32.0)));
	}
	else {
		ah = 0.135*exp(-1.0*(vm + 80.0) / 6.8);
		bh = 3.56*exp(0.079*vm) + (3.1e5)*exp(0.35*vm);
		aj = (-127140.0*exp(0.2444*vm) - (3.474e-5)*exp(-0.04391*vm))*(vm + 37.78) /
			(1.0 + exp(0.311*(vm + 79.23)));
		bj = 0.1212*exp(-0.01052*vm) / (1.0 + exp(-0.1378*(vm + 40.14)));
	}
	real tauh = 1.0 / (ah + bh);
	real infh = ah * tauh;
	real tauj = 1.0 / (aj + bj);
	real infj = aj * tauj;

	real asa = 0.65 / ((exp(-1.0*(vm + 10.0) / 8.5)) + (exp(-1.0*(vm - 30.0) / 59.0)));
	real bsa = 0.65 / (2.5 + exp((vm + 82.0) / 17.0));
	real infsa = 1.0 / (1.0 + exp(-1.0*(vm + 20.47) / 17.54));
	real tausa = (1.0 / (asa + bsa)) / Kq10;

	real asi = 1.0 / (18.53 + exp((vm + 113.7) / 10.95));
	real bsi = 1.0 / (35.56 + exp(-1.0*(vm + 1.26) / 7.44));
	real infsi = 1.0 / (1.0 + exp((vm + 43.1) / 5.3));
	real tausi = (1.0 / (asi + bsi)) / Kq10;

	real aua = 0.65*1.0 / (exp(-1.0*(vm + 10.0) / 8.5) + exp(-1.0*(vm - 30.0) / 59.0));
	real bua = 0.65*1.0 / (2.5 + exp((vm + 82.0) / 17.0));
	real infua = 1.0 / (1.0 + exp(-1.0*(vm + 30.3) / 9.6));
	real tauua = (1.0 / (aua + bua)) / Kq10;

	real aui = 1.0 / (21.0 + exp(-1.0*(vm - 185.0) / 28.0));
	real bui = exp((vm - 158.0) / 16.0);
	real infui = 1.0 / (1.0 + exp((vm - 99.45) / 27.48));
	real tauui = (1.0 / (aui + bui)) / Kq10;

	real axr = 0.0003*(vm + 14.1) / (1.0 - exp(-1.0*(vm + 14.1) / 5.0));
	real bxr = (7.3898e-5)*(vm - 3.3328) / (exp((vm - 3.3328) / 5.1237) - 1.0);
	real infxr = 1.0 / (1.0 + exp(-1.0*(vm + 14.1) / 6.5));
	real tauxr = (1.0 / (axr + bxr));

	real axs = (4.0e-5)*(vm - 19.9) / (1.0 - exp(-1.0*(vm - 19.9) / 17.0));
	real bxs = (3.5e-5)*(vm - 19.9) / (exp((vm - 19.9) / 9.0) - 1.0);
	real tauxs = 0.5 / (axs + bxs);
	real infxs = 1.0 / sqrt(1.0 + exp(-1.0*(vm - 19.9) / 12.7));

	real taud = (1.0 - exp(-1.0*(vm + 10.0) / 6.24)) / (0.035*(vm + 10.0)*(1.0 + exp(-1.0*(vm + 10.0) / 6.24)));
	real infd = 1.0 / (1.0 + exp(-1.0*(vm + 10.0) / 8.0));

	real tauf = 9.0 / (0.0197*exp(-1.0*0.0337*0.0337*(vm + 10.0)*(vm + 10.0)) + 0.02);
	real inff = 1.0 / (1.0 + exp((vm + 28.0) / 6.9));

	real inffca = 1.0 / (1.0 + Cai / 0.00035);

	real Fn = 1.0e-12*Vrel*Irel - (5.0e-7 / F)*(0.5*Ical - 0.2*Inaca);

	real infu = 1.0 / (1.0 + exp(-1.0*(Fn - 3.4175e-13) / 13.67e-16));

	real tauv = 1.91 + 2.09 / (1.0 + exp(-1.0*(Fn - 3.4175e-13) / 13.67e-16));
	real infv = 1.0 - 1.0 / (1.0 + exp(-1.0*(Fn - 6.835e-14) / 13.67e-16));

	real tauw = 6.0*(1.0 - exp(-1.0*(vm - 7.9) / 5.0)) / ((1.0 + 0.3*exp(-1.0*(vm - 7.9) / 5.0))*(vm - 7.9));
	real infw = 1.0 - 1.0 / (1.0 + exp(-1.0*(vm - 40.0) / 17.0));

	/*------------------------------------------------------------------------
	* Update variables
	*------------------------------------------------------------------------
	*/
	fv += -1.0*Iion;
	g_dev.fv[i2d] = fv;
	//if (i2d == printv) printf("eqns: %.31f\n", fv);

	if (t >= g_dev.start[i2d] && t <= g_dev.finish[i2d]) fv += g_dev.strength[i2d];

	g_dev.vm[i2d] += dt / Cm * fv;

	g_dev.Nai[i2d] += dt * ( 1.0e6*((-3.0*Inak - 3.0*Inaca - Ibna - Ina) / (F*Vi)) );
	g_dev.Ki[i2d] += dt * ( 1.0e6*((2.0*Inak - Ik1 - Ito - Ikur - Ikr - Iks) / (F*Vi)) );
	real B1 = 1.0e6*((2.0*Inaca - Ipca - Ical - Ibca) / (2.0*F*Vi)) +
		(Vup*(Iupleak - Iup) + Irel * Vrel) / Vi;
	real B2 = 1.0 + Trpnmax * KmTrpn / ((Cai + KmTrpn)*(Cai + KmTrpn)) + Cmdnmax * KmCmdn /
		((Cai + KmCmdn)*(Cai + KmCmdn));
	g_dev.Cai[i2d] += dt * ( B1 / B2 );
	g_dev.Caup[i2d] += dt * ( Iup - Iupleak - Itr * Vrel / Vup );
	g_dev.Carel[i2d] += dt * ( (Itr - Irel)*(1.0 / (1.0 + (Csqnmax*KmCsqn) /
		((Carel + KmCsqn)*(Carel + KmCsqn)))) );

	g_dev.m[i2d]   += dt * ( (infm - m) / taum );
	g_dev.h[i2d]   += dt * ( (infh - h) / tauh );
	g_dev.j[i2d]   += dt * ( (infj - j) / tauj );
	g_dev.sa[i2d]  += dt * ( (infsa - sa) / tausa );
	g_dev.si[i2d]  += dt * ( (infsi - si) / tausi );
	g_dev.ua[i2d]  += dt * ( (infua - ua) / tauua );
	g_dev.ui[i2d]  += dt * ( (infui - ui) / tauui );
	g_dev.xr[i2d]  += dt * ( (infxr - xr) / tauxr );
	g_dev.xs[i2d]  += dt * ( (infxs - xs) / tauxs );
	g_dev.d[i2d]   += dt * ( (infd - d) / taud );
	g_dev.f[i2d]   += dt * ( (inff - f) / tauf );
	g_dev.fca[i2d] += dt * ( (inffca - fca) / taufca );
	g_dev.u[i2d]   += dt * ( (infu - u) / tauu );
	g_dev.v[i2d]   += dt * ( (infv - v) / tauv );
	g_dev.w[i2d]   += dt * ( (infw - w) / tauw );

}


void __host__ CRN_print(int totpoints, gateType* gate_h, FILE* fp) {
	fwrite(gate_h->vm, sizeof(real), totpoints, fp);
	fflush(fp);	
}