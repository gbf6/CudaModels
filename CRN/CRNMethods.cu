#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include "hip/hip_runtime.h"
#include ""

#include "../common/CudaSafeCall.h"

#include "CRNglobalVariables.cuh"
#include "typedefSparse.h"
#include "sparsePrototypes.cuh"
#include "typedefCRN.h"

#include "CRNhostPrototypes.h"
#include "CRNdevicePrototypes.cuh"

void __device__ stimSimple(real t, int i2d, gateType* g_dev, real* fv);

void CRN_gateinit(int memSize, size_t* pitch, gateType* gate_h, gateType* gate_dev, gateType* gate_devF) {

	hipHostAlloc((void**)&(gate_h->vm), memSize, 0);
	hipHostAlloc((void**)&(gate_h->m), memSize, 0);
	hipHostAlloc((void**)&(gate_h->h), memSize, 0);
	hipHostAlloc((void**)&(gate_h->j), memSize, 0);
	hipHostAlloc((void**)&(gate_h->sa), memSize, 0);
	hipHostAlloc((void**)&(gate_h->si), memSize, 0);
	hipHostAlloc((void**)&(gate_h->ua), memSize, 0);
	hipHostAlloc((void**)&(gate_h->ui), memSize, 0);
	hipHostAlloc((void**)&(gate_h->xr), memSize, 0);
	hipHostAlloc((void**)&(gate_h->xs), memSize, 0);
	hipHostAlloc((void**)&(gate_h->d), memSize, 0);
	hipHostAlloc((void**)&(gate_h->f), memSize, 0);
	hipHostAlloc((void**)&(gate_h->fca), memSize, 0);
	hipHostAlloc((void**)&(gate_h->u), memSize, 0);
	hipHostAlloc((void**)&(gate_h->v), memSize, 0);
	hipHostAlloc((void**)&(gate_h->w), memSize, 0);
	hipHostAlloc((void**)&(gate_h->Nai), memSize, 0);
	hipHostAlloc((void**)&(gate_h->Ki), memSize, 0);
	hipHostAlloc((void**)&(gate_h->Cai), memSize, 0);
	hipHostAlloc((void**)&(gate_h->Caup), memSize, 0);
	hipHostAlloc((void**)&(gate_h->Carel), memSize, 0);



	// Allocate device memory arrays
	CudaSafeCall(hipMalloc((void **)&gate_dev->qp, sizeof(real*)*gate_dev->qpl));

	CudaSafeCall(hipMallocPitch((void **)&gate_dev->vm, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->m, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->h, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->j, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->sa, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->si, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->ua, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->ui, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->xr, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->xs, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->d, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->f, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->fca, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->u, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->v, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->w, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->Nai, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->Ki, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->Cai, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->Caup, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->Carel, pitch,
		memSize, 1));



	// Allocate device forward memory arrays
	CudaSafeCall(hipMalloc((void **)&gate_devF->qp, sizeof(real*)*gate_devF->qpl));

	CudaSafeCall(hipMallocPitch((void **)&gate_devF->vm, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->m, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->h, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->j, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->sa, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->si, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->ua, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->ui, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->xr, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->xs, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->d, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->f, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->fca, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->u, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->v, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->w, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->Nai, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->Ki, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->Cai, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->Caup, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->Carel, pitch,
		memSize, 1));

	puts("\nFinished allocating device arrays\n");

	int totpoints = (int)memSize / sizeof(real);

	for (int idx = 0; idx < totpoints; idx++) {
		gate_h->vm[idx]       = -81.2;
		gate_h->m[idx]        = 2.91e-3;
		gate_h->h[idx]        = 0.965;
		gate_h->j[idx]        = 0.978;
		gate_h->sa[idx]       = 3.04e-2;
		gate_h->si[idx]       = 0.999;
		gate_h->ua[idx]       = 4.96e-3;
		gate_h->ui[idx]       = 0.999;
		gate_h->xr[idx]       = 3.29e-5;
		gate_h->xs[idx]       = 1.87e-2;
		gate_h->d[idx]        = 1.37e-4;
		gate_h->f[idx]        = 0.999;
		gate_h->fca[idx]      = 0.775;
		gate_h->u[idx]        = 0.0;
		gate_h->v[idx]        = 1.0;
		gate_h->w[idx]        = 0.999;
		gate_h->Nai[idx]      = 11.2;
		gate_h->Ki[idx]       = 139.0;
		gate_h->Cai[idx]      = 1.02e-4;
		gate_h->Caup[idx]     = 1.49;
		gate_h->Carel[idx]    = 1.49;
	}

	CudaSafeCall(hipMemcpy2D((void *)gate_dev->vm, *pitch, (void *)gate_h->vm,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->m, *pitch, (void *)gate_h->m,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->h, *pitch, (void *)gate_h->h,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->j, *pitch, (void *)gate_h->j,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->sa, *pitch, (void *)gate_h->sa,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->si, *pitch, (void *)gate_h->si,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->ua, *pitch, (void *)gate_h->ua,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->ui, *pitch, (void *)gate_h->ui,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->xr, *pitch, (void *)gate_h->xr,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->xs, *pitch, (void *)gate_h->xs,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->d, *pitch, (void *)gate_h->d,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->f, *pitch, (void *)gate_h->f,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->fca, *pitch, (void *)gate_h->fca,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->u, *pitch, (void *)gate_h->u,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->v, *pitch, (void *)gate_h->v,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->w, *pitch, (void *)gate_h->w,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->Nai, *pitch, (void *)gate_h->Nai,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->Ki, *pitch, (void *)gate_h->Ki,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->Cai, *pitch, (void *)gate_h->Cai,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->Caup, *pitch, (void *)gate_h->Caup,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->Carel, *pitch, (void *)gate_h->Carel,
		memSize, memSize, 1, hipMemcpyHostToDevice));

	real** qpH = (real**)malloc(sizeof(real *)*gate_h->qpl);
	int i = 0;
	qpH[i++] = gate_devF->m;
	qpH[i++] = gate_devF->h;
	qpH[i++] = gate_devF->j;
	qpH[i++] = gate_devF->sa;
	qpH[i++] = gate_devF->si;
	qpH[i++] = gate_devF->ua;
	qpH[i++] = gate_devF->ui;
	qpH[i++] = gate_devF->xr;
	qpH[i++] = gate_devF->xs;
	qpH[i++] = gate_devF->d;
	qpH[i++] = gate_devF->f;
	qpH[i++] = gate_devF->fca;
	qpH[i++] = gate_devF->u;
	qpH[i++] = gate_devF->v;
	qpH[i++] = gate_devF->w;
	qpH[i++] = gate_devF->Nai;
	qpH[i++] = gate_devF->Ki;
	qpH[i++] = gate_devF->Cai;
	qpH[i++] = gate_devF->Caup;
	qpH[i++] = gate_devF->Carel;

	CudaSafeCall(hipMemcpy((void *)gate_devF->qp, (void*)qpH, sizeof(real *)*gate_h->qpl, hipMemcpyHostToDevice));
	
	i = 0;
	qpH[i++] = gate_dev->m;
	qpH[i++] = gate_dev->h;
	qpH[i++] = gate_dev->j;
	qpH[i++] = gate_dev->sa;
	qpH[i++] = gate_dev->si;
	qpH[i++] = gate_dev->ua;
	qpH[i++] = gate_dev->ui;
	qpH[i++] = gate_dev->xr;
	qpH[i++] = gate_dev->xs;
	qpH[i++] = gate_dev->d;
	qpH[i++] = gate_dev->f;
	qpH[i++] = gate_dev->fca;
	qpH[i++] = gate_dev->u;
	qpH[i++] = gate_dev->v;
	qpH[i++] = gate_dev->w;
	qpH[i++] = gate_dev->Nai;
	qpH[i++] = gate_dev->Ki;
	qpH[i++] = gate_dev->Cai;
	qpH[i++] = gate_dev->Caup;
	qpH[i++] = gate_dev->Carel;

	CudaSafeCall(hipMemcpy((void *)gate_dev->qp, (void*)qpH, sizeof(real *)*gate_h->qpl, hipMemcpyHostToDevice));



	puts("\nFinished initializing device arrays\n");

}

void CRN_sync(int memSize, size_t pitch, gateType* gate_h, gateType* gate_dev) {
	CudaSafeCall(hipMemcpy2D((void *)gate_h->vm, memSize, (void *)gate_dev->vm,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->m, memSize, (void *)gate_dev->m,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->h, memSize, (void *)gate_dev->h,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->j, memSize, (void *)gate_dev->j,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->sa, memSize, (void *)gate_dev->sa,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->si, memSize, (void *)gate_dev->si,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->ua, memSize, (void *)gate_dev->ua,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->ui, memSize, (void *)gate_dev->ui,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->xr, memSize, (void *)gate_dev->xr,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->xs, memSize, (void *)gate_dev->xs,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->d, memSize, (void *)gate_dev->d,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->f, memSize, (void *)gate_dev->f,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->fca, memSize, (void *)gate_dev->fca,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->u, memSize, (void *)gate_dev->u,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->v, memSize, (void *)gate_dev->v,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->w, memSize, (void *)gate_dev->w,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->Nai, memSize, (void *)gate_dev->Nai,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->Ki, memSize, (void *)gate_dev->Ki,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->Cai, memSize, (void *)gate_dev->Cai,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->Caup, memSize, (void *)gate_dev->Caup,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->Carel, memSize, (void *)gate_dev->Carel,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
	CudaCheckError();
}

void CRN_exit(int memSize, size_t pitch, gateType* gate_h, gateType* gate_dev, gateType* gate_devF, sparse* MatrixINT, cudasparse* cudaMatrixINT) {
	// Free gate host and device memory
	hipHostFree(gate_h->vm); hipHostFree(gate_h->m); hipHostFree(gate_h->j); hipHostFree(gate_h->sa);
	hipHostFree(gate_h->si); hipHostFree(gate_h->ua); hipHostFree(gate_h->ui); hipHostFree(gate_h->xr);
	hipHostFree(gate_h->xs); hipHostFree(gate_h->d); hipHostFree(gate_h->f); hipHostFree(gate_h->fca);
	hipHostFree(gate_h->u); hipHostFree(gate_h->v); hipHostFree(gate_h->w); hipHostFree(gate_h->Nai);
	hipHostFree(gate_h->Ki); hipHostFree(gate_h->Cai); hipHostFree(gate_h->Caup); hipHostFree(gate_h->Carel);

	hipFree(gate_dev->vm); hipFree(gate_dev->m); hipFree(gate_dev->j); hipFree(gate_dev->sa);
	hipFree(gate_dev->si); hipFree(gate_dev->ua); hipFree(gate_dev->ui); hipFree(gate_dev->xr);
	hipFree(gate_dev->xs); hipFree(gate_dev->d); hipFree(gate_dev->f); hipFree(gate_dev->fca);
	hipFree(gate_dev->u); hipFree(gate_dev->v); hipFree(gate_dev->w); hipFree(gate_dev->Nai);
	hipFree(gate_dev->Ki); hipFree(gate_dev->Cai); hipFree(gate_dev->Caup); hipFree(gate_h->Carel);
	hipFree(gate_h->qp);

	hipFree(gate_devF->vm); hipFree(gate_devF->m); hipFree(gate_devF->j); hipFree(gate_devF->sa);
	hipFree(gate_devF->si); hipFree(gate_devF->ua); hipFree(gate_devF->ui); hipFree(gate_devF->xr);
	hipFree(gate_devF->xs); hipFree(gate_devF->d); hipFree(gate_devF->f); hipFree(gate_devF->fca);
	hipFree(gate_devF->u); hipFree(gate_devF->v); hipFree(gate_devF->w); hipFree(gate_devF->Nai);
	hipFree(gate_devF->Ki); hipFree(gate_devF->Cai); hipFree(gate_devF->Caup); hipFree(gate_h->Carel);
	hipFree(gate_h->qp);

	hipFree(cudaMatrixINT->type);
	hipFree(cudaMatrixINT->rows);
	hipFree(cudaMatrixINT->maxnz);
	hipFree(cudaMatrixINT->csep);
	hipFree(cudaMatrixINT->jcoef);
	hipFree(cudaMatrixINT->coef);

}

 void __device__ GetFDev_CRN(int i2d, int pitch, real beta, real Cm, real t, real dt, int totpoints, real rx, gateType g_dev, gateType g_devF) {

	/*------------------------------------------------------------------------
	* return if outside domain
	*------------------------------------------------------------------------
	*/
	if (i2d >= totpoints) {
		//printf("Aborting");
		return;
	}

	/*------------------------------------------------------------------------
	* setting local variables
	*------------------------------------------------------------------------
	*/

	real vm    = g_dev.vm[i2d];
	real m     = g_dev.m[i2d];
	real h     = g_dev.h[i2d];
	real j     = g_dev.j[i2d];
	real sa    = g_dev.sa[i2d];
	real si    = g_dev.si[i2d];
	real ua    = g_dev.ua[i2d];
	real ui    = g_dev.ui[i2d];
	real xr    = g_dev.xr[i2d];
	real xs    = g_dev.xs[i2d];
	real d     = g_dev.d[i2d];
	real f     = g_dev.f[i2d];
	real fca   = g_dev.fca[i2d];
	real u     = g_dev.u[i2d];
	real v     = g_dev.v[i2d];
	real w     = g_dev.w[i2d];
	real Nai   = g_dev.Nai[i2d];
	real Ki    = g_dev.Ki[i2d];
	real Cai   = g_dev.Cai[i2d];
	real Caup  = g_dev.Caup[i2d];
	real Carel = g_dev.Carel[i2d];


	real fv = g_devF.vm[i2d];
	
	//if (i2d == printv) printf("t: %.31f\ndiff: %.31f\n", t, fv);

	/*------------------------------------------------------------------------
	* Currents
	*------------------------------------------------------------------------
	*/

	real Ena = (R*T / F)*log(Nao / Nai);
	real Ek = (R*T / F)*log(Ko / Ki);
	real Eca = (R*T / (2.0*F))*log(Cao / Cai);
	real gkur = 1.0e-4*(0.005 + 0.05 / (1.0 + exp(-1.0*(vm - 15.0) / 13.0)));
	real sigma = (1.0 / 7.0)*(exp(Nao / 67.3) - 1.0);
	real fnak = 1.0 / (1.0 + 0.1245*exp(-0.1*F*vm / (R*T)) + 0.0365*sigma*exp(-1.0*F*vm / (R*T)));

	real Ina = gna * m*m*m*h*j*(vm - Ena);
	real Ik1 = gk1 * (vm - Ek) / (1.0 + exp(0.07*(vm + 80.0)));
	real Ito = gto * sa*sa*sa*si*(vm - Ek);
	real Ikur = gkur * ua*ua*ua*ui*(vm - Ek);
	real Ikr = gkr * xr*(vm - Ek) / (1.0 + exp((vm + 15.0) / 22.4));
	real Iks = gks * xs*xs*(vm - Ek);
	real Ical = gcal * d*f*fca*(vm - 65.0);
	real Inak = Inakmax * fnak*(1.0 / (1.0 + pow(Kmnai / Nai, 1.5))) * Ko / (Ko + Kmko);
	real Inaca = Inacamax * (exp(lambda*F*vm / (R*T))*Nai*Nai*Nai*Cao -
		exp((lambda - 1)*F*vm / (R*T))*Nao*Nao*Nao*Cai) /
		((Kmna*Kmna*Kmna + Nao * Nao*Nao)*(Kmca + Cao)*(1.0 + ksat * exp((lambda - 1)*F*vm / (R*T))));
	real Ibca = gbca * (vm - Eca);
	real Ibna = gbna * (vm - Ena);
	real Ipca = Ipcamax * Cai / (0.0005 + Cai);

	real Irel = krel * u*u*v*w*(Carel - Cai);
	real Itr = (Caup - Carel) / tautr;
	real Iup = Iupmax / (1.0 + (kup / Cai));
	real Iupleak = Caup * Iupmax / Caupmax;

	real Iion = Ina + Ik1 + Ito + Ikur + Ikr + Iks + Ical + Ipca + Inak + Inaca + Ibna + Ibca;
	Iion /= (pi*cellDiameter*cellLength*1.0e-8);
	
	/*------------------------------------------------------------------------
	* Gating variables
	*------------------------------------------------------------------------
	*/

	real am;
	if (vm == -47.13) {
		am = 3.2;
	}
	else {
		am = 0.32*(vm + 47.13) / (1.0 - exp(-0.1*(vm + 47.13)));
	}
	real bm = 0.08*exp(-vm / 11.0);
	real taum = 1.0 / (am + bm);
	real infm = am * taum;

	real ah, bh, aj, bj;
	if (vm >= -40.0) {
		ah = 0.0;
		bh = 1.0 / (0.13*(1.0 + exp(-1.0*(vm + 10.66) / 11.1)));
		aj = 0.0;
		bj = 0.3*exp((-2.535e-7)*vm) / (1.0 + exp(-0.1*(vm + 32.0)));
	}
	else {
		ah = 0.135*exp(-1.0*(vm + 80.0) / 6.8);
		bh = 3.56*exp(0.079*vm) + (3.1e5)*exp(0.35*vm);
		aj = (-127140.0*exp(0.2444*vm) - (3.474e-5)*exp(-0.04391*vm))*(vm + 37.78) /
			(1.0 + exp(0.311*(vm + 79.23)));
		bj = 0.1212*exp(-0.01052*vm) / (1.0 + exp(-0.1378*(vm + 40.14)));
	}
	real tauh = 1.0 / (ah + bh);
	real infh = ah * tauh;
	real tauj = 1.0 / (aj + bj);
	real infj = aj * tauj;

	real asa = 0.65 / ((exp(-1.0*(vm + 10.0) / 8.5)) + (exp(-1.0*(vm - 30.0) / 59.0)));
	real bsa = 0.65 / (2.5 + exp((vm + 82.0) / 17.0));
	real infsa = 1.0 / (1.0 + exp(-1.0*(vm + 20.47) / 17.54));
	real tausa = (1.0 / (asa + bsa)) / Kq10;

	real asi = 1.0 / (18.53 + exp((vm + 113.7) / 10.95));
	real bsi = 1.0 / (35.56 + exp(-1.0*(vm + 1.26) / 7.44));
	real infsi = 1.0 / (1.0 + exp((vm + 43.1) / 5.3));
	real tausi = (1.0 / (asi + bsi)) / Kq10;

	real aua = 0.65*1.0 / (exp(-1.0*(vm + 10.0) / 8.5) + exp(-1.0*(vm - 30.0) / 59.0));
	real bua = 0.65*1.0 / (2.5 + exp((vm + 82.0) / 17.0));
	real infua = 1.0 / (1.0 + exp(-1.0*(vm + 30.3) / 9.6));
	real tauua = (1.0 / (aua + bua)) / Kq10;

	real aui = 1.0 / (21.0 + exp(-1.0*(vm - 185.0) / 28.0));
	real bui = exp((vm - 158.0) / 16.0);
	real infui = 1.0 / (1.0 + exp((vm - 99.45) / 27.48));
	real tauui = (1.0 / (aui + bui)) / Kq10;

	real axr = 0.0003*(vm + 14.1) / (1.0 - exp(-1.0*(vm + 14.1) / 5.0));
	real bxr = (7.3898e-5)*(vm - 3.3328) / (exp((vm - 3.3328) / 5.1237) - 1.0);
	real infxr = 1.0 / (1.0 + exp(-1.0*(vm + 14.1) / 6.5));
	real tauxr = (1.0 / (axr + bxr));

	real axs = (4.0e-5)*(vm - 19.9) / (1.0 - exp(-1.0*(vm - 19.9) / 17.0));
	real bxs = (3.5e-5)*(vm - 19.9) / (exp((vm - 19.9) / 9.0) - 1.0);
	real tauxs = 0.5 / (axs + bxs);
	real infxs = 1.0 / sqrt(1.0 + exp(-1.0*(vm - 19.9) / 12.7));

	real taud = (1.0 - exp(-1.0*(vm + 10.0) / 6.24)) / (0.035*(vm + 10.0)*(1.0 + exp(-1.0*(vm + 10.0) / 6.24)));
	real infd = 1.0 / (1.0 + exp(-1.0*(vm + 10.0) / 8.0));

	real tauf = 9.0 / (0.0197*exp(-1.0*0.0337*0.0337*(vm + 10.0)*(vm + 10.0)) + 0.02);
	real inff = 1.0 / (1.0 + exp((vm + 28.0) / 6.9));

	real inffca = 1.0 / (1.0 + Cai / 0.00035);

	real Fn = 1.0e-12*Vrel*Irel - (5.0e-7 / F)*(0.5*Ical - 0.2*Inaca);

	real infu = 1.0 / (1.0 + exp(-1.0*(Fn - 3.4175e-13) / 13.67e-16));

	real tauv = 1.91 + 2.09 / (1.0 + exp(-1.0*(Fn - 3.4175e-13) / 13.67e-16));
	real infv = 1.0 - 1.0 / (1.0 + exp(-1.0*(Fn - 6.835e-14) / 13.67e-16));

	real tauw = 6.0*(1.0 - exp(-1.0*(vm - 7.9) / 5.0)) / ((1.0 + 0.3*exp(-1.0*(vm - 7.9) / 5.0))*(vm - 7.9));
	real infw = 1.0 - 1.0 / (1.0 + exp(-1.0*(vm - 40.0) / 17.0));

	/*------------------------------------------------------------------------
	* Update variables
	*------------------------------------------------------------------------
	*/
	fv += -1.0*Iion;
	g_devF.vm[i2d] = fv;
	//if (i2d == printv) printf("eqns: %.31f\n", fv);

	g_devF.Nai[i2d] = 1.0e6*((-3.0*Inak - 3.0*Inaca - Ibna - Ina) / (F*Vi)) ;
	g_devF.Ki[i2d] = 1.0e6*((2.0*Inak - Ik1 - Ito - Ikur - Ikr - Iks) / (F*Vi)) ;
	real B1 = 1.0e6*((2.0*Inaca - Ipca - Ical - Ibca) / (2.0*F*Vi)) +
		(Vup*(Iupleak - Iup) + Irel * Vrel) / Vi;
	real B2 = 1.0 + Trpnmax * KmTrpn / ((Cai + KmTrpn)*(Cai + KmTrpn)) + Cmdnmax * KmCmdn /
		((Cai + KmCmdn)*(Cai + KmCmdn));
	g_devF.Cai[i2d] = B1 / B2 ;
	g_devF.Caup[i2d] = Iup - Iupleak - Itr * Vrel / Vup ;
	g_devF.Carel[i2d] = (Itr - Irel)*(1.0 / (1.0 + (Csqnmax*KmCsqn) /
		((Carel + KmCsqn)*(Carel + KmCsqn)))) ;

	g_devF.m[i2d]   = (infm - m) / taum ;
	g_devF.h[i2d]   = (infh - h) / tauh ;
	g_devF.j[i2d]   = (infj - j) / tauj ;
	g_devF.sa[i2d]  = (infsa - sa) / tausa ;
	g_devF.si[i2d]  = (infsi - si) / tausi ;
	g_devF.ua[i2d]  = (infua - ua) / tauua ;
	g_devF.ui[i2d]  = (infui - ui) / tauui ;
	g_devF.xr[i2d]  = (infxr - xr) / tauxr ;
	g_devF.xs[i2d]  = (infxs - xs) / tauxs ;
	g_devF.d[i2d]   = (infd - d) / taud ;
	g_devF.f[i2d]   = (inff - f) / tauf ;
	g_devF.fca[i2d] = (inffca - fca) / taufca ;
	g_devF.u[i2d]   = (infu - u) / tauu ;
	g_devF.v[i2d]   = (infv - v) / tauv ;
	g_devF.w[i2d]   = (infw - w) / tauw ;

}

