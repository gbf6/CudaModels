#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include "hip/hip_runtime.h"
#include ""


#include "../common/CudaSafeCall.h"

#include "fourVglobalVariables.cuh"
#include "typedefSparse.h"
#include "sparsePrototypes.cuh"
#include "typedeffourV.h"

#include "fourVhostPrototypes.h"
#include "fourVdevicePrototypes.cuh"

void fourV_gateinit(int memSize, size_t* pitch, gateType* gate_h, gateType* gate_dev, gateType* gate_devF) {

	hipHostAlloc((void**)&(gate_h->u), memSize, 0);
	hipHostAlloc((void**)&(gate_h->v), memSize, 0);
	hipHostAlloc((void**)&(gate_h->w), memSize, 0);
	hipHostAlloc((void**)&(gate_h->s), memSize, 0);

	// Allocate device memory arrays
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->u, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->v, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->w, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->s, pitch,
		memSize, 1));

	// Allocate device forward memory arrays
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->u, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->v, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->w, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->s, pitch,
		memSize, 1));

	puts("\nFinished allocating device arrays\n");

	int totpoints = (int)memSize / sizeof(real);

	for (int idx = 0; idx < totpoints; idx++) {
		gate_h->u[idx] = 0.0;
		gate_h->v[idx] = 1.0;
		gate_h->w[idx] = 1.0;
		gate_h->s[idx] = 0.0;
	}

	CudaSafeCall(hipMemcpy2D((void *)gate_dev->u, *pitch, (void *)gate_h->u,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->v, *pitch, (void *)gate_h->v,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->w, *pitch, (void *)gate_h->w,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->s, *pitch, (void *)gate_h->s,
		memSize, memSize, 1, hipMemcpyHostToDevice));

	CudaSafeCall(hipMemcpy2D((void *)gate_devF->u, *pitch, (void *)gate_h->u,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->v, *pitch, (void *)gate_h->v,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->w, *pitch, (void *)gate_h->w,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->s, *pitch, (void *)gate_h->s,
		memSize, memSize, 1, hipMemcpyHostToDevice));

	real** qpH = (real**)malloc(sizeof(real *)*gate_h->qpl);
	int i = 0;
	qpH[i++] = gate_devF->v;
	qpH[i++] = gate_devF->w;
	qpH[i++] = gate_devF->s;

	CudaSafeCall(hipMemcpy((void *)gate_devF->qp, (void*)qpH, sizeof(real *)*gate_h->qpl, hipMemcpyHostToDevice));

	i = 0;
	qpH[i++] = gate_dev->v;
	qpH[i++] = gate_dev->w;
	qpH[i++] = gate_dev->s;

	CudaSafeCall(hipMemcpy((void *)gate_dev->qp, (void*)qpH, sizeof(real *)*gate_h->qpl, hipMemcpyHostToDevice));

	gate_h->vm = (real*)malloc(sizeof(real*));
	gate_h->vm = gate_h->u;

	real* point;

	CudaSafeCall(hipMalloc((void **)&gate_dev->vm, sizeof(gate_dev->v)));
	point = gate_dev->v;
	CudaSafeCall(hipMemcpy((void *)gate_dev->vm, (void *)point, sizeof(real *), hipMemcpyHostToDevice));

	CudaSafeCall(hipMalloc((void **)&gate_devF->vm, sizeof(gate_devF->v)));
	point = gate_devF->v;
	CudaSafeCall(hipMemcpy((void *)gate_devF->vm, (void *)point, sizeof(real *), hipMemcpyHostToDevice));
	

	CudaCheckError();

	puts("\nFinished initializing device arrays\n");

}

void fourV_sync(int memSize, size_t pitch, gateType* gate_h, gateType* gate_dev) {
	CudaSafeCall(hipMemcpy2D((void *)gate_h->u, memSize, (void *)gate_dev->u,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->v, memSize, (void *)gate_dev->v,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->w, memSize, (void *)gate_dev->w,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->s, memSize, (void *)gate_dev->s,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
}

void fourV_exit(int memSize, size_t pitch, gateType* gate_h, gateType* gate_dev, gateType* gate_devF, sparse* MatrixINT, cudasparse* cudaMatrixINT){
	// Free gate host and device memory
	hipHostFree(gate_h->u); hipHostFree(gate_h->v); hipHostFree(gate_h->w); hipHostFree(gate_h->s);
	hipHostFree(gate_h->vm);
	
	hipFree(gate_dev->u); hipFree(gate_dev->v); hipFree(gate_dev->w); hipFree(gate_dev->s);
	hipFree(gate_dev->vm); hipFree(gate_h->qp);

	hipFree(gate_devF->u); hipFree(gate_devF->v); hipFree(gate_devF->w); hipFree(gate_devF->s);
	hipFree(gate_devF->vm); hipFree(gate_h->qp);

	hipFree(cudaMatrixINT->type);
	hipFree(cudaMatrixINT->rows);
	hipFree(cudaMatrixINT->maxnz);
	hipFree(cudaMatrixINT->csep);
	hipFree(cudaMatrixINT->jcoef);
	hipFree(cudaMatrixINT->coef);
}

/*========================================================================
* 2D : 4-Variable Model Time Integrator
*========================================================================
*/

void __device__ GetFDev_fourV(int i2d, int pitch, real beta, real Cm, real t, real dt, int totpoints, real rx, gateType g_dev, gateType g_devF) {

	/*------------------------------------------------------------------------
	* return if outside domain
	*------------------------------------------------------------------------
	*/
	if (i2d >= totpoints) {
		return;
	}

	/*------------------------------------------------------------------------
	* setting local variables
	*------------------------------------------------------------------------
	*/
	real u = g_dev.u[i2d];
	real v = g_dev.v[i2d];
	real w = g_dev.w[i2d];
	real s = g_dev.s[i2d];

	real fu = g_devF.u[i2d];

	/*------------------------------------------------------------------------
	* Additional heaviside functions
	*------------------------------------------------------------------------
	*/
	real  H_theta_v = (u > theta_v) ? 1.0 : 0.0;
	real  H_theta_m_v = (u > theta_m_v) ? 1.0 : 0.0;
	real  H_theta_w = (u > theta_w) ? 1.0 : 0.0;
	real  H_theta_so = (u > theta_so) ? 1.0 : 0.0;
	real  H_theta_si = (u > theta_si) ? 1.0 : 0.0;
	real  H_theta_s = (u > theta_s) ? 1.0 : 0.0;
	real  H_theta_o = (u > theta_o) ? 1.0 : 0.0;
	real  H_theta_vinf = (u > theta_vinf) ? 1.0 : 0.0;
	real  H_theta_winf = (u > theta_winf) ? 1.0 : 0.0;
	/*------------------------------------------------------------------------
	* Calculating dependant tau's
	*------------------------------------------------------------------------
	*/
	real tau_m_v = (1.0 - H_theta_m_v)*tau_m_v1
		+ H_theta_m_v * tau_m_v2;
	real tau_m_w = tau_m_w1
		+ (tau_m_w2 - tau_m_w1)*(1. + tanh(k_m_w*(u - u_m_w)))*0.5;
	real  tau_p_w = tau_p_w1
		+ (tau_p_w2 - tau_p_w1)*(1. + tanh(k_p_w*(
			delta_w*(w - w_p_c) + (1. - delta_w)*(u - u_p_w))))*0.5;
	real tau_s = (1. - H_theta_s)*tau_s1 + H_theta_s * tau_s2;
	real tau_o = (1. - H_theta_o)*tau_o1 + H_theta_o * tau_o2;
	real tau_so = tau_so1
		+ (tau_so2 - tau_so1)*(1. + tanh(k_so*(u - u_so)))*0.5;
	real  tau_si = tau_si1
		+ (tau_si2 - tau_si1)*(1. + tanh(k_si*(s - s_c)))*0.5;
	real  tau_p_si = alpha_si * (1. + exp(k_si1*(u - theta_p_si))) /
		(1. - tanh(k_si2*(u - theta_p_si)));

	real v_inf = 1. - H_theta_vinf;
	real w_inf = (1. - H_theta_winf)*(1. - u / tau_winf)
		+ H_theta_winf * w_sinf;

	/*------------------------------------------------------------------------
	* v
	*------------------------------------------------------------------------
	*/
	real  dv2dt = (1. - H_theta_v)*(v_inf - v) / tau_m_v
		- H_theta_v * v / tau_p_v;
	v += dv2dt * dt;
	g_devF.v[i2d] = v;

	/*------------------------------------------------------------------------
	* w
	*------------------------------------------------------------------------
	*/
	real  wx = (2. - alpha_w)*(3. - alpha_w)*(4. - alpha_w)*w / 6.0
		+ (alpha_w - 1.)*(3. - alpha_w)*(4. - alpha_w)*0.5*w*w
		+ (alpha_w - 1.)*(alpha_w - 2.)*(4. - alpha_w)*0.5*w*w*w
		+ (alpha_w - 1.)*(alpha_w - 2.)*(alpha_w - 3.)*w*w*w*w / 6.;

	real dw2dt = (1. - H_theta_w)*(w_inf - wx) / tau_m_w
		- H_theta_w * w / tau_p_w;
	w += dw2dt * dt;
	g_devF.w[i2d] = w;

	/*------------------------------------------------------------------------
	* s
	*------------------------------------------------------------------------
	*/
	real   ds2dt = ((1. + tanh(k_s*(u - u_s)))*0.5 - s) / tau_s;

	s += ds2dt * dt;
	g_devF.s[i2d] = s;

	/*------------------------------------------------------------------------
	* I_sum
	*------------------------------------------------------------------------
	*/
	real  J_fi = -v * H_theta_v*(u - theta_p_v)*(u_u - u) / tau_fi;
	real  J_so = (u - u_o)*(1. - H_theta_so)*(1. - beta_v * v) / tau_o
		+ H_theta_so / tau_so;
	real  J_si;
	if (gamma_si > 0.5)
		J_si = -H_theta_si * w*s / tau_si;
	else
		J_si = -(1. + tanh(k_si_c*(u - theta_si_c)))*w / tau_p_si;

	real  I_sum = J_fi + J_so + J_si;


	/*------------------------------------------------------------------------
	* Time integration
	*------------------------------------------------------------------------
	*/
	fu -= dt * I_sum / Cm;

	g_devF.u[i2d] = u;
}