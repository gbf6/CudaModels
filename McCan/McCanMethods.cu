#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include "hip/hip_runtime.h"
#include ""


#include "../common/CudaSafeCall.h"

#include "McCanglobalVariables.cuh"
#include "typedefSparse.h"
#include "sparsePrototypes.cuh"
#include "typedefMcCan.h"

#include "parseInput.h"

#include "McCanhostPrototypes.h"
#include "McCandevicePrototypes.cuh"

__device__ real gkv = gkv_0;
__device__ real shiftrs = shiftrs_0;
__device__ real GbNa = GbNa_0;
real McCan_RestVoltage = 

void McCan_init(char** res) {

	 rword resources[] = {
	{ "McCan_Node",    1100 },
	{ "McCan_Nodetype",1100 },
	{ "McCan_Type",    1100 },
	{ "McCan_patch",   1102 },
	{ "McCan_Vr",	1007 },
	{ "McCan_Vrest",	1007 },
	{ "McCan_gkv",	1008 },
	{ "McCan_shiftrs",	1009 },
	{ "McCan_gbna",	1010 },
	{ NULL, 0 }
	};

	int i, j, c;
	int cmd;
	real temp;

	i = 0;
	while (res[i] != NULL) {
		cmd = FindCommand(resources, res[i]);
		switch (cmd) {
		case 1007:
			McCan_RestVoltage = GetRealValue(res[i]);
			break;
		case 1008:
			temp = GetRealValue(res[i]);
			hipMemcpyToSymbol(HIP_SYMBOL(gkv), (void *)&temp, sizeof(real), 0, hipMemcpyHostToDevice);
			break;
		case 1009:
			temp = GetRealValue(res[i]);
			hipMemcpyToSymbol(HIP_SYMBOL(shiftrs), (void *)&temp, sizeof(real), 0, hipMemcpyHostToDevice);
			break;
		case 1010:
			temp = GetRealValue(res[i]);
			hipMemcpyToSymbol(HIP_SYMBOL(GbNa), (void *)&temp, sizeof(real), 0, hipMemcpyHostToDevice);
			break;
		case 1100:
			//McCan_NodeType = GetByteValue(res[i]);
			break;
		case 1102:
			/*iv = GetRealArray(res[i]);
			p = (real*)(&McCan_RestPatch);
			c = GetNumValues(res[i]);
			if (c > McCan_PatchSize) {
				c = McCan_PatchSize;
			}
			for (j = 0; j<c; j++) {
				p[j] = iv[j];
			}*/
			break;
		}
		i++;
	}
}

void McCan_gateinit(int memSize, size_t* pitch, gateType* gate_h, gateType* gate_dev, gateType* gate_devF) {

	hipHostAlloc((void**)&(gate_h->vm), memSize, 0);
	hipHostAlloc((void**)&(gate_h->r), memSize, 0);
	hipHostAlloc((void**)&(gate_h->s), memSize, 0);
	

	// Allocate device memory arrays
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->vm, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->r, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->x, pitch,
		memSize, 1));

	// Allocate device forward memory arrays
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->vm, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->r, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->s, pitch,
		memSize, 1));

	puts("\nFinished allocating device arrays\n");

	int totpoints = (int)memSize / sizeof(real);

	for (int idx = 0; idx < totpoints; idx++) {
		gate_h->vm[idx] = McCan_RestVoltage;
		gate_h->r[idx] = 6.434895e-02;
		gate_h->s[idx] = 9.776443e-01;
	}

	CudaSafeCall(hipMemcpy2D((void *)gate_dev->vm, *pitch, (void *)gate_h->vm,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->r, *pitch, (void *)gate_h->r,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->s, *pitch, (void *)gate_h->s,
		memSize, memSize, 1, hipMemcpyHostToDevice));


	CudaSafeCall(hipMemcpy2D((void *)gate_devF->vm, *pitch, (void *)gate_h->vm,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->r, *pitch, (void *)gate_h->r,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->s, *pitch, (void *)gate_h->s,
		memSize, memSize, 1, hipMemcpyHostToDevice));

	real** qpH = (real**)malloc(sizeof(real *)*gate_h->qpl);
	int i = 0;
	qpH[i++] = gate_devF->r;
	qpH[i++] = gate_devF->s;


	CudaSafeCall(hipMemcpy((void *)gate_devF->qp, (void*)qpH, sizeof(real *)*gate_h->qpl, hipMemcpyHostToDevice));

	i = 0;
	qpH[i++] = gate_dev->r;
	qpH[i++] = gate_dev->s;

	CudaSafeCall(hipMemcpy((void *)gate_dev->qp, (void*)qpH, sizeof(real *)*gate_h->qpl, hipMemcpyHostToDevice));
	

	CudaCheckError();

	puts("\nFinished initializing device arrays\n");

}

void McCan_sync(int memSize, size_t pitch, gateType* gate_h, gateType* gate_dev) {
	CudaSafeCall(hipMemcpy2D((void *)gate_h->vm, memSize, (void *)gate_dev->vm,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->r, memSize, (void *)gate_dev->r,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->s, memSize, (void *)gate_dev->s,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
}

void McCan_exit(int memSize, size_t pitch, gateType* gate_h, gateType* gate_dev, gateType* gate_devF, sparse* MatrixINT, cudasparse* cudaMatrixINT){
	// Free gate host and device memory
	hipHostFree(gate_h->vm); hipHostFree(gate_h->r); hipHostFree(gate_h->s); hipFree(gate_dev->qp); 

	hipFree(gate_dev->vm); hipFree(gate_dev->r); hipFree(gate_dev->s); hipFree(gate_dev->qp); 

	hipFree(gate_devF->vm); hipFree(gate_devF->r); hipFree(gate_devF->s); hipFree(gate_devF->qp); 

	hipFree(cudaMatrixINT->type);
	hipFree(cudaMatrixINT->rows);
	hipFree(cudaMatrixINT->maxnz);
	hipFree(cudaMatrixINT->csep);
	hipFree(cudaMatrixINT->jcoef);
	hipFree(cudaMatrixINT->coef);
}


void __device__ GetFDev_McCan(int i2d, int pitch, real beta, real Cm, real t, real dt, int totpoints, real rx, gateType g_dev, gateType g_devF) {

	/*------------------------------------------------------------------------
	* return if outside domain
	*------------------------------------------------------------------------
	*/
	if (i2d >= totpoints) {
		return;
	}

	real   vm;
	real   r, s;
	real   r_inf, s_inf, tau_r, tau_s, aK1, bK1, ENa;
	real   Ikv, Ik1, Inak, Ibna, Iion;

	real vm = g_dev.vm[i2d];
	real r  = g_dev.r[i2d];
	real s  = g_dev.s[i2d];

	/*------------------------------------------------------------------------
	* setting local variables
	*------------------------------------------------------------------------
	*/

	real fv = g_devF.vm[i2d];

	/* gating variables */
	r_inf = 1/(1+exp(-(vm+20-shiftrs)/11));
	tau_r = 20.3 + 138 * exp( -sqr((vm+20-shiftrs)/25.9) );
	s_inf = 1/(1+exp((vm+23-shiftrs)/7));
	tau_s = 1574 + 5268 * exp( -sqr((vm+23-shiftrs)/22.7) );
	
	/* I_Kv */
	Ikv = gkv * r * s * (vm-EK);
	
	/* I_K1 */
	aK1 = 0.1/(1+exp(0.06*(vm-EK-200)));
	bK1 = ( 3*exp(0.0002*(vm-EK+100)) + exp(0.1*(vm-EK-10)) )
	    / ( 1+exp(-0.5*(vm-EK)) );
	Ik1 = gK1 * aK1/(aK1+bK1) * (vm-EK);
	
	/* I_NaK */
	Inak = INaKbar / (1+KmK/Ko) / (1+pow((KmNa/Nai),1.5)) * (vm-Vrev) / (vm-B);
	
	/* I_bNa */
	ENa =  R*T/F * log(Nao/Nai);
	Ibna = GbNa * (vm-ENa);

	/* I_ion */
	Iion = Ikv + Ik1 + Inak + Ibna; 
	
	/* differential equations */
	fv -= Cm*Iion;
	g_devF.r[i2d]  = (r_inf - r) / tau_r;
	g_devF.s[i2d]  = (s_inf - s) / tau_s;


	g_devF.vm[i2d] = fv;
}