#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include "hip/hip_runtime.h"
#include ""


#include "../common/CudaSafeCall.h"

#include "globalVariables.cuh"
#include "typedefSparse.h"
#include "sparsePrototypes.cuh"
#include "typedef.h"

#include "hostPrototypes.h"
#include "devicePrototypes.cuh"

void _gateinit(int memSize, size_t* pitch, gateType* gate_h, gateType* gate_dev, gateType* gate_devF) {

	hipHostAlloc((void**)&(gate_h->), memSize, 0);
	

	// Allocate device memory arrays
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->, pitch,
		memSize, 1));

	// Allocate device forward memory arrays
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->, pitch,
		memSize, 1));

	puts("\nFinished allocating device arrays\n");

	int totpoints = (int)memSize / sizeof(real);

	for (int idx = 0; idx < totpoints; idx++) {
		gate_h->[idx] = 0.0;
	}

	CudaSafeCall(hipMemcpy2D((void *)gate_dev->, *pitch, (void *)gate_h->,
		memSize, memSize, 1, hipMemcpyHostToDevice));

	CudaSafeCall(hipMemcpy2D((void *)gate_devF->, *pitch, (void *)gate_h->,
		memSize, memSize, 1, hipMemcpyHostToDevice));

	real** qpH = (real**)malloc(sizeof(real *)*gate_h->qpl);
	int i = 0;
	qpH[i++] = gate_devF->;

	CudaSafeCall(hipMemcpy((void *)gate_devF->qp, (void*)qpH, sizeof(real *)*gate_h->qpl, hipMemcpyHostToDevice));

	i = 0;
	qpH[i++] = gate_dev->;

	CudaSafeCall(hipMemcpy((void *)gate_dev->qp, (void*)qpH, sizeof(real *)*gate_h->qpl, hipMemcpyHostToDevice));
	

	CudaCheckError();

	puts("\nFinished initializing device arrays\n");

}

void _sync(int memSize, size_t pitch, gateType* gate_h, gateType* gate_dev) {
	CudaSafeCall(hipMemcpy2D((void *)gate_h->, memSize, (void *)gate_dev->,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
}

void _exit(int memSize, size_t pitch, gateType* gate_h, gateType* gate_dev, gateType* gate_devF, sparse* MatrixINT, cudasparse* cudaMatrixINT){
	// Free gate host and device memory
	hipHostFree(gate_h->); 

	hipFree(gate_dev->); 

	hipFree(gate_devF->); 

	hipFree(cudaMatrixINT->type);
	hipFree(cudaMatrixINT->rows);
	hipFree(cudaMatrixINT->maxnz);
	hipFree(cudaMatrixINT->csep);
	hipFree(cudaMatrixINT->jcoef);
	hipFree(cudaMatrixINT->coef);
}


void __device__ GetFDev_(int i2d, int pitch, real beta, real Cm, real t, real dt, int totpoints, real rx, gateType g_dev, gateType g_devF) {

	/*------------------------------------------------------------------------
	* return if outside domain
	*------------------------------------------------------------------------
	*/
	if (i2d >= totpoints) {
		return;
	}

	/*------------------------------------------------------------------------
	* setting local variables
	*------------------------------------------------------------------------
	*/
	real  = g_dev.[i2d];

	real f = g_devF.[i2d];



	f -= dt * I_sum / Cm;

	g_devF.[i2d] = ;
}