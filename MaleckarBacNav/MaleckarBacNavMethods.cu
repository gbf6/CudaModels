#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include "hip/hip_runtime.h"
#include ""


#include "../common/CudaSafeCall.h"

#include "MaleckarBacNavglobalVariables.cuh"
#include "typedefSparse.h"
#include "sparsePrototypes.cuh"
#include "typedefMaleckar.h"

#include "parseInput.h"

#include "MaleckarBacNavhostPrototypes.h"
#include "MaleckarBacNavdevicePrototypes.cuh"

real MaleckarBacNav_RestVoltage = MaleckarBacNav_RestVoltage_0;
__device__ real gkv = gkv_0;
__device__ real shiftrs = shiftrs_0;
__device__ real GbNa = GbNa_0;
__device__ real GNA = GNA_0;


void Maleckar_init(char** res) {
	rword resources[] = {
	{ "MaleckarBacNav_Node",    1100 },
	{ "MaleckarBacNav_Nodetype",1100 },
	{ "MaleckarBacNav_Type",    1100 },
	{ "MaleckarBacNav_patch",   1102 },
	{ "MaleckarBacNav_Vr",	1007 },
	{ "MaleckarBacNav_Vrest",	1007 },
	{ "MaleckarBacNav_gkv",	1008 },
	{ "MaleckarBacNav_shiftrs",	1009 },
	{ "MaleckarBacNav_gbna",	1010 },
	{ "MaleckarBacNav_gna",         1011 },
	{ NULL, 0 }
	};

	int i, j, c;
	int cmd;
	real temp;

	i = 0;
	while( res[i] != NULL ) {
		cmd = FindCommand( resources, res[i] );
		switch( cmd ) {
			case 1007:
				MaleckarBacNav_RestVoltage = GetRealValue( res[i] );
				break;
			case 1008:
				temp = GetRealValue(res[i]);
				hipMemcpyToSymbol(HIP_SYMBOL(gkv), (void *)&temp, sizeof(real), 0, hipMemcpyHostToDevice);
				break;
			case 1009:
				temp = GetRealValue(res[i]);
				hipMemcpyToSymbol(HIP_SYMBOL(shiftrs), (void *)&temp, sizeof(real), 0, hipMemcpyHostToDevice);
				break;
			case 1010:
				temp = GetRealValue(res[i]);
				hipMemcpyToSymbol(HIP_SYMBOL(GbNa), (void *)&temp, sizeof(real), 0, hipMemcpyHostToDevice);
				break;
			case 1011:
				temp = GetRealValue(res[i]);
				hipMemcpyToSymbol(HIP_SYMBOL(GNA), (void *)&temp, sizeof(real), 0, hipMemcpyHostToDevice);
				break;
			case 1100:
				//MaleckarBacNav_NodeType = GetByteValue( res[i] );
				break;
			case 1102:
				/*iv = GetRealArray( res[i] );
				p  = (real*)(&MaleckarBacNav_RestPatch);
				c  = GetNumValues( res[i] );
				if( c > MaleckarBacNav_PatchSize ) {
					c = MaleckarBacNav_PatchSize;
				}
				for(j=0;j<c;j++) {
					p[j] = iv[j];
				}
				break;*/
		}
		i++;
	}
}

void Maleckar_gateinit(int memSize, size_t* pitch, gateType* gate_h, gateType* gate_dev, gateType* gate_devF) {

	hipHostAlloc((void**)&(gate_h->vm), memSize, 0);
	hipHostAlloc((void**)&(gate_h->r), memSize, 0);
	hipHostAlloc((void**)&(gate_h->s), memSize, 0);
	hipHostAlloc((void**)&(gate_h->m), memSize, 0);
	hipHostAlloc((void**)&(gate_h->h), memSize, 0);
	

	// Allocate device memory arrays
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->vm, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->r, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->s, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->m, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_dev->h, pitch,
		memSize, 1));

	// Allocate device forward memory arrays
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->vm, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->r, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->s, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->m, pitch,
		memSize, 1));
	CudaSafeCall(hipMallocPitch((void **)&gate_devF->h, pitch,
		memSize, 1));

	puts("\nFinished allocating device arrays\n");

	int totpoints = (int)memSize / sizeof(real);

	for (int idx = 0; idx < totpoints; idx++) {
		gate_h->vm[idx] = MaleckarBacNav_RestVoltage;
		gate_h->r[idx] = 5.454e-02;
		gate_h->s[idx] = 9.814e-01;
		gate_h->m[idx] = 0.00012037;
		gate_h->h[idx] = .4769;
	}

	CudaSafeCall(hipMemcpy2D((void *)gate_dev->vm, *pitch, (void *)gate_h->vm,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->r, *pitch, (void *)gate_h->r,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->s, *pitch, (void *)gate_h->s,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->m, *pitch, (void *)gate_h->m,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_dev->h, *pitch, (void *)gate_h->h,
		memSize, memSize, 1, hipMemcpyHostToDevice));


	CudaSafeCall(hipMemcpy2D((void *)gate_devF->vm, *pitch, (void *)gate_h->vm,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->r, *pitch, (void *)gate_h->r,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->s, *pitch, (void *)gate_h->s,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->m, *pitch, (void *)gate_h->m,
		memSize, memSize, 1, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy2D((void *)gate_devF->h, *pitch, (void *)gate_h->h,
		memSize, memSize, 1, hipMemcpyHostToDevice));

	real** qpH = (real**)malloc(sizeof(real *)*gate_h->qpl);
	int i = 0;
	qpH[i++] = gate_devF->r;
	qpH[i++] = gate_devF->s;
	qpH[i++] = gate_devF->m;
	qpH[i++] = gate_devF->h;


	CudaSafeCall(hipMemcpy((void *)gate_devF->qp, (void*)qpH, sizeof(real *)*gate_h->qpl, hipMemcpyHostToDevice));

	i = 0;
	qpH[i++] = gate_dev->r;
	qpH[i++] = gate_dev->s;
	qpH[i++] = gate_dev->m;
	qpH[i++] = gate_dev->h;

	CudaSafeCall(hipMemcpy((void *)gate_dev->qp, (void*)qpH, sizeof(real *)*gate_h->qpl, hipMemcpyHostToDevice));
	

	CudaCheckError();

	puts("\nFinished initializing device arrays\n");

}

void Maleckar_sync(int memSize, size_t pitch, gateType* gate_h, gateType* gate_dev) {
	CudaSafeCall(hipMemcpy2D((void *)gate_h->vm, memSize, (void *)gate_dev->vm,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->r, memSize, (void *)gate_dev->r,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->s, memSize, (void *)gate_dev->s,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->m, memSize, (void *)gate_dev->m,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
	CudaSafeCall(hipMemcpy2D((void *)gate_h->h, memSize, (void *)gate_dev->h,
		pitch, memSize, 1, hipMemcpyDeviceToHost));
}

void Maleckar_exit(int memSize, size_t pitch, gateType* gate_h, gateType* gate_dev, gateType* gate_devF, sparse* MatrixINT, cudasparse* cudaMatrixINT){
	// Free gate host and device memory
	hipHostFree(gate_h->vm); hipHostFree(gate_h->r); hipHostFree(gate_h->s); hipHostFree(gate_h->m); hipHostFree(gate_h->h); hipFree(gate_dev->qp); 

	hipFree(gate_dev->vm); hipFree(gate_dev->r); hipFree(gate_dev->s); hipFree(gate_dev->m); hipFree(gate_dev->h); hipFree(gate_dev->qp); 

	hipFree(gate_devF->vm); hipFree(gate_devF->r); hipFree(gate_devF->s); hipFree(gate_devF->m); hipFree(gate_devF->h); hipFree(gate_devF->qp); 

	hipFree(cudaMatrixINT->type);
	hipFree(cudaMatrixINT->rows);
	hipFree(cudaMatrixINT->maxnz);
	hipFree(cudaMatrixINT->csep);
	hipFree(cudaMatrixINT->jcoef);
	hipFree(cudaMatrixINT->coef);
}


void __device__ GetFDev_Maleckar(int i2d, int pitch, real beta, real Cm, real t, real dt, int totpoints, real rx, gateType g_dev, gateType g_devF) {

	/*------------------------------------------------------------------------
	* return if outside domain
	*------------------------------------------------------------------------
	*/
	if (i2d >= totpoints) {
		return;
	}

	real   vm;
	real   r, s, m, h;
	real   r_inf, s_inf, tau_r, tau_s, minf, hinf, taum, tauh, aK1, bK1, ENa, EK;
	real   Ikv, Ik1, Inak, Ibna, Iion, INaBacNav;

	real vm = g_dev.vm[i2d];
	real r  = g_dev.r[i2d];
	real s  = g_dev.s[i2d];
	real m  = g_dev.m[i2d];
	real h  = g_dev.h[i2d];

	/*------------------------------------------------------------------------
	* setting local variables
	*------------------------------------------------------------------------
	*/
	
	real fv = g_devF.vm[i2d];

	/* gating variables */
	r_inf = 1/(1+exp(-(vm)/11));
	tau_r = 20.3 + 138 * exp( -sqr((vm+20)/25.9) );
	s_inf = 1/(1+exp((vm+3)/7));
	tau_s = 1574 + 5268 * exp( -sqr((vm+23)/22.7) );
    minf = (1.0/(1.0+exp((vm+28.34)/(-5.33))));
	hinf = (1.0-1.0/(1.0+exp((-77.21-vm)/8.32)));
	taum = (96.37/(exp((vm+82.74)/17.64) + exp(-(vm+6.008)/3.337)) + .4844);
	tauh = (96.17 - (96.17 - 10.45)/(1.0+exp((-23.26-vm)/2.529)));

	/* I_Kv */
	EK = R*T/F * log(Ko/Ki);
	Ikv = gkv * r * s * (vm-EK);
	
	/* I_K1 */
	aK1 = 0.1/(1+exp(0.06*(vm-EK-200)));
	bK1 = ( 3*exp(0.0002*(vm-EK+100)) + exp(0.1*(vm-EK-10)) )
	    / ( 1+exp(-0.5*(vm-EK)) );
	Ik1 = gK1 * aK1/(aK1+bK1) * (vm-EK);
	
	/* I_NaK */
	Inak = INaKbar / (1+KmK/Ko) / (1+pow((KmNa/Nai),1.5)) * (vm-Vrev) / (vm-B);

	/* I_bNa */
	ENa =  R*T/F * log(Nao/Nai);
	Ibna = GbNa * (vm-ENa);
	
	/* INaBacNav */
	INaBacNav = GNA*m*m*m*h*(vm-ENa);	

	/* I_ion */
	Iion = Ikv + Ik1 + Inak + INaBacNav + Ibna; 
	
	/* differential equations */
	fv -= Cm*Iion;
	g_devF.r[i2d]  = (r_inf - r) / tau_r;
	g_devF.s[i2d]  = (s_inf - s) / tau_s;
	g_devF.m[i2d]  = (m_inf - m) / tau_m;
	g_devF.h[i2d]  = (h_inf - h) / tau_h;


	g_devF.vm[i2d] = fv;
}